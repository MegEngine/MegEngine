#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017-2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are
 *permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this
 *list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this
 *list of conditions and the following disclaimer in the documentation and/or other
 *materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors
 *may be used to endorse or promote products derived from this software without specific
 *prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
 *EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
 *OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
 *SHALL NVIDIA CORPORATION BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 *EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 *SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
 *HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 *OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 *SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/**
 * \file dnn/src/cuda/matrix_mul/uint4x4x32_wmma/preprocess_quantize_sum.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 */

#include "./preprocess_quantize_sum.cuh"

#include <stdio.h>
#include <limits>

#include "src/cuda/cub/util_ptx.cuh"
#include "src/cuda/utils.cuh"

namespace {

using namespace megdnn::cuda;

template <int block_size_log2, int max_nr_threads_per_row>
__global__ void reduce_column_with_scale_u4(
        const uint8_t* src, int32_t scale, int rows, int cols_int32, int ld_in_bytes,
        int nr_thread_per_row_log2, int sm_width_in_bytes, int32_t* dst) {
    constexpr int warp_size = 32;
    extern __shared__ uint8_t sub_block_raw[];

    uint32_t nr_row_per_block = 1 << (block_size_log2 - nr_thread_per_row_log2),
             nr_threads_per_row = 1 << nr_thread_per_row_log2,
             row_num = threadIdx.x >> nr_thread_per_row_log2,
             tid = threadIdx.x - (row_num << nr_thread_per_row_log2),
             row_idx = blockIdx.x * nr_row_per_block + row_num;
    if (row_idx >= rows)
        return;

    volatile int32_t* row = (int32_t*)(sub_block_raw + row_num * sm_width_in_bytes);
    const int32_t* sptr = (const int32_t*)(src + row_idx * ld_in_bytes);
    sptr += tid;
    int32_t local = 0;
    for (int i = tid; i < cols_int32; i += nr_threads_per_row) {
        int32_t val = (*sptr);
#pragma unroll
        for (int j = 0; j < 8; j++) {
            local += (val & 0xF);
            val = (val >> 4);
        }
        sptr += nr_threads_per_row;
    }
    row[tid] = local;

#pragma unroll
    for (int i = max_nr_threads_per_row / 2; i; i >>= 1) {
        bool cond = nr_threads_per_row >= (i * 2) && tid < i;
        if (i >= warp_size) {
            __syncthreads();
        } else {
            hipcub::WARP_SYNC(0xffffffff);
        }
        if (cond) {
            row[tid] += row[tid + i];
        }
    }
    if (!tid) {
        int32_t* dptr = dst + row_idx;
        *dptr = row[0] * scale;
    }
}

template <size_t TX, size_t TY, size_t BX, size_t BY>
__global__ void span_qsum(
        const int32_t* qSumA, const uint32_t M, const int32_t* qSumB, const uint32_t N,
        int32_t* dst, const uint32_t strd, const int32_t scaler_bias) {
    constexpr size_t mm = (BY + TY - 1) / TY;
    constexpr size_t nn = (BX + TX - 1) / TX;

#pragma unroll
    for (int i = 0; i < mm; ++i) {
#pragma unroll
        for (int j = 0; j < nn; ++j) {
            int gtidx = threadIdx.x + TX * j + blockIdx.x * BX;
            int gtidy = threadIdx.y + TY * i + blockIdx.y * BY;
            if (gtidx < N && gtidy < M) {
                dst[gtidy * strd + gtidx] += qSumA[gtidy] + qSumB[gtidx] + scaler_bias;
            }
        }
    }
}

template <int block_size_log2, int max_nr_threads_per_row>
void _do_dispatch_reduce_column_with_scale_u4(
        const uint8_t* src, int32_t scale, int rows, int cols_int32, int ld_in_bytes,
        int32_t* dst, hipStream_t stream) {
    constexpr int warp_size = 32;
    int block_size = 1 << block_size_log2;
    int nr_thread_per_row = 1, nr_thread_per_row_log2 = 0;

    while (nr_thread_per_row < max_nr_threads_per_row &&
           nr_thread_per_row * 2 < cols_int32) {
        ++nr_thread_per_row_log2;
        nr_thread_per_row *= 2;
    }
    // now: nr_thread_per_row <= B < nr_thread_per_row * 2

    if (cols_int32 <= max_nr_threads_per_row * 4) {
        // find nr_thread_per_row with minimal wasted threads
        int min_cost = std::numeric_limits<int>::max(), min_cost_th = 0;
        for (int i = warp_size; i <= nr_thread_per_row; i *= 2) {
            int cost = (i - cols_int32 % i) % i;
            if (cost < min_cost) {
                min_cost = cost;
                min_cost_th = i;
            }
        }
        if (min_cost_th) {
            nr_thread_per_row = min_cost_th;
            while ((1 << nr_thread_per_row_log2) != nr_thread_per_row)
                --nr_thread_per_row_log2;
        }
    }

    int nr_row_per_block = block_size / nr_thread_per_row,
        nr_blk = DIVUP(rows, nr_row_per_block), sm_width_word32 = nr_thread_per_row;

    // gcd(sm_width_word32, BANKS) should be 1 to avoid bank confliction
    // iff sm_width_word32 is odd
    sm_width_word32 += !(sm_width_word32 % 2);
    int sm_width_in_bytes = sm_width_word32 * 4,
        sm_size = nr_row_per_block * sm_width_in_bytes;

    void (*kptr)(
            const uint8_t* src, int32_t scale, int rows, int cols_int32,
            int ld_in_bytes, int nr_thread_per_row_log2, int sm_width_in_bytes,
            int32_t* dst);
    if (nr_thread_per_row <= max_nr_threads_per_row / 4) {
        kptr = reduce_column_with_scale_u4<block_size_log2, max_nr_threads_per_row / 4>;
    } else if (nr_thread_per_row <= max_nr_threads_per_row / 2) {
        kptr = reduce_column_with_scale_u4<block_size_log2, max_nr_threads_per_row / 2>;
    } else {
        kptr = reduce_column_with_scale_u4<block_size_log2, max_nr_threads_per_row>;
    }
    kptr<<<nr_blk, block_size, sm_size, stream>>>(
            src, scale, rows, cols_int32, ld_in_bytes, nr_thread_per_row_log2,
            sm_width_in_bytes, dst);
    after_kernel_launch();
}

}  // namespace

void megdnn::cuda::exec_reduce_sum_with_scale_uint4(
        const uint8_t* A, int32_t scale, uint32_t M, uint32_t K, uint32_t ldA_in_byte,
        int32_t* dst, hipStream_t stream) {
    _do_dispatch_reduce_column_with_scale_u4<7, 64>(
            A, scale, M, K / 8, ldA_in_byte, dst, stream);
}

void megdnn::cuda::exec_span_qsum(
        const int32_t* qSumA, const uint32_t M, const int32_t* qSumB, const uint32_t N,
        int32_t* dst, const uint32_t strd, const int32_t scaler_bias,
        hipStream_t stream) {
    constexpr uint32_t TX = 32, TY = 32, BX = 32, BY = 32;
    dim3 nthreads{TX, TY};
    dim3 nblocks{DIVUP(N, BX), DIVUP(M, BY)};
    span_qsum<TX, TY, BX, BY><<<nblocks, nthreads, 0, stream>>>(
            qSumA, M, qSumB, N, dst, strd, scaler_bias);
    after_kernel_launch();
}

// vim: ft=cpp syntax=cuda.doxygen
