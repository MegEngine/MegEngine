/**
 * \file dnn/src/cuda/matrix_mul/cutlass_matrix_mul_wrapper_batched_gemv_strided.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or
 * implied.
 */
// ignore warning of cutlass
#include "hip/hip_runtime.h"
#if __CUDACC_VER_MAJOR__ > 9 || (__CUDACC_VER_MAJOR__ == 9 && __CUDACC_VER_MINOR__ >= 2)
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wunused-parameter"
#pragma GCC diagnostic ignored "-Wstrict-aliasing"

#include "cutlass/gemm/device/gemm.h"
#include "cutlass/gemm/device/gemm_splitk_parallel.h"
#include "cutlass/gemm/kernel/default_gemv.h"
#include "src/common/opr_param_defs_enumv.cuh"
#include "src/cuda/matrix_mul/cutlass_matrix_mul_wrapper.cuh"
#pragma GCC diagnostic pop

using namespace megdnn;
using namespace cuda;
using namespace cutlass_wrapper;

/* ============ cutlass kernel wrapper for f32 vector-matrix mul batched strided
 * ===========
 */
#define DISPATCH(cb)                                                          \
    cb(128, 4, 4);                                                            \
    cb(128, 4, 2);                                                            \
    cb(128, 4, 1);                                                            \
    cb(128, 2, 4);                                                            \
    cb(128, 1, 4);                                                            \
    cb(128, 2, 2);                                                            \
    cb(128, 1, 2);                                                            \
    cb(128, 2, 1);                                                            \
    cb(128, 1, 1);                                                            \
    cb(64, 4, 4);                                                             \
    cb(64, 4, 2);                                                             \
    cb(64, 4, 1);                                                             \
    cb(64, 2, 4);                                                             \
    cb(64, 1, 4);                                                             \
    cb(64, 2, 2);                                                             \
    cb(64, 1, 2);                                                             \
    cb(64, 2, 1);                                                             \
    cb(64, 1, 1);                                                             \
    cb(32, 4, 4);                                                             \
    cb(32, 4, 2);                                                             \
    cb(32, 4, 1);                                                             \
    cb(32, 2, 4);                                                             \
    cb(32, 1, 4);                                                             \
    cb(32, 2, 2);                                                             \
    cb(32, 1, 2);                                                             \
    cb(32, 2, 1);                                                             \
    cb(32, 1, 1);                                                             \
    megdnn_assert(                                                            \
            false, "unsupported gemv batched strided A=%dX%dX%d, B=%dX%dX%d", \
            problem_size.batch(), problem_size.m(), problem_size.k(),         \
            problem_size.batch(), problem_size.k(), problem_size.n());

void megdnn::cuda::cutlass_wrapper::
        cutlass_matrix_mul_float32_simt_gemv_batched_strided(
                const float* d_A, size_t lda, size_t batch_stride_a, const float* d_B,
                size_t ldb, size_t batch_stride_b, float* d_C, size_t ldc,
                size_t batch_stride_c, BatchedGemmCoord const& problem_size,
                int threadblock_n, cudaStream_t stream) {
    int LDG_K, LDG_N;
    if (lda % 4 == 0)
        LDG_K = 4;
    else if (lda % 2 == 0)
        LDG_K = 2;
    else
        LDG_K = 1;

    if (ldb % 4 == 0)
        LDG_N = 4;
    else if (ldb % 2 == 0)
        LDG_N = 2;
    else
        LDG_N = 1;
#define cb(threadblock_n_, LDG_K_, LDG_N_)                                             \
    if (threadblock_n == threadblock_n_ && LDG_K == LDG_K_ && LDG_N == LDG_N_) {       \
        using ThreadBlockShape = cutlass::gemm::GemmShape<                             \
                1, threadblock_n_, (256 * LDG_K_) / (threadblock_n_ / LDG_N_)>;        \
        using ThreadShape = cutlass::gemm::GemmShape<1, LDG_N_, LDG_K_>;               \
        using GemvKernel = cutlass::gemm::kernel::DefaultGemv<                         \
                ThreadBlockShape, ThreadShape, float, cutlass::layout::RowMajor,       \
                float, cutlass::layout::RowMajor, float, cutlass::layout::RowMajor>;   \
        return cutlass_vector_matrix_mul_batched_strided_wrapper<GemvKernel>(          \
                problem_size, d_A, lda, batch_stride_a, d_B, ldb, batch_stride_b, d_C, \
                ldc, batch_stride_c, stream);                                          \
    }
    DISPATCH(cb)
#undef cb
}
#undef DISPATCH

#endif

// vim: syntax=cuda.doxygen
