#include "hip/hip_runtime.h"
/**
 * \file dnn/src/cuda/batched_matrix_mul/helper.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 */
#include "src/cuda/batched_matrix_mul/helper.cuh"

namespace {

template <typename T>
__global__ void kernel(T* Xs, T start, uint32_t step, uint32_t n) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        Xs[i] = start + i * step;
    }
}

}  // anonymous namespace

namespace megdnn {
namespace cuda {
namespace batched_matrix_mul {

template <typename T>
void arange(T* Xs, T start, uint32_t step, uint32_t n, hipStream_t stream) {
    uint32_t threads = NR_THREADS;
    uint32_t blocks = DIVUP(n, threads);
    kernel<T><<<blocks, threads, 0, stream>>>(Xs, start, step, n);
    after_kernel_launch();
}

template void arange<uintptr_t>(
        uintptr_t*, uintptr_t, uint32_t, uint32_t, hipStream_t);

}  // namespace batched_matrix_mul
}  // namespace cuda
}  // namespace megdnn

// vim: syntax=cpp.doxygen
