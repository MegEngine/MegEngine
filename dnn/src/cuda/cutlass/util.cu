
#include <hip/hip_runtime.h>
/***************************************************************************************************
 * Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 *modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice,
 *this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *notice, this list of conditions and the following disclaimer in the
 *documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its
 *contributors may be used to endorse or promote products derived from this
 *software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 *AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 *IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 *DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY DIRECT,
 *INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 *DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 *OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TOR (INCLUDING
 *NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 *EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/**
 * \file dnn/src/cuda/cutlass/util.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or
 * implied.
 */

#if __CUDACC_VER_MAJOR__ > 9 || (__CUDACC_VER_MAJOR__ == 9 && __CUDACC_VER_MINOR__ >= 2)

#include <complex>
#include <iosfwd>

#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wreorder"
#pragma GCC diagnostic ignored "-Wstrict-aliasing"
#pragma GCC diagnostic ignored "-Wunused-parameter"

#include "cutlass/complex.h"
#include "cutlass/cutlass.h"
#include "cutlass/layout/matrix.h"
#include "cutlass/numeric_types.h"

#include "src/cuda/cutlass/util.h"

#pragma GCC diagnostic pop

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace cutlass {
namespace library {

/////////////////////////////////////////////////////////////////////////////////////////////////

static struct {
    char const* text;
    char const* pretty;
    Provider enumerant;
} Provider_enumerants[] = {
        {"none", "None", Provider::kNone},
        {"cutlass", "CUTLASS", Provider::kCUTLASS},
        {"host", "reference_host", Provider::kReferenceHost},
        {"device", "reference_device", Provider::kReferenceDevice},
        {"cublas", "cuBLAS", Provider::kCUBLAS},
        {"cudnn", "cuDNN", Provider::kCUDNN},
};

/// Converts a Provider enumerant to a string
char const* to_string(Provider provider, bool pretty) {
    for (auto const& possible : Provider_enumerants) {
        if (provider == possible.enumerant) {
            if (pretty) {
                return possible.pretty;
            } else {
                return possible.text;
            }
        }
    }

    return pretty ? "Invalid" : "invalid";
}

/// Parses a Provider enumerant from a string
template <>
Provider from_string<Provider>(std::string const& str) {
    for (auto const& possible : Provider_enumerants) {
        if ((str.compare(possible.text) == 0) || (str.compare(possible.pretty) == 0)) {
            return possible.enumerant;
        }
    }

    return Provider::kInvalid;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

static struct {
    char const* text;
    char const* pretty;
    GemmKind enumerant;
} GemmKind_enumerants[] = {
        {"gemm", "<Gemm>", GemmKind::kGemm},
        {"spgemm", "<Sparse>", GemmKind::kSparse},
        {"universal", "<Universal>", GemmKind::kUniversal},
        {"planar_complex", "<PlanarComplex>", GemmKind::kPlanarComplex},
        {"planar_complex_array", "<PlanarComplexArray>", GemmKind::kPlanarComplexArray},
};

/// Converts a GemmKind enumerant to a string
char const* to_string(GemmKind type, bool pretty) {
    for (auto const& possible : GemmKind_enumerants) {
        if (type == possible.enumerant) {
            if (pretty) {
                return possible.pretty;
            } else {
                return possible.text;
            }
        }
    }

    return pretty ? "Invalid" : "invalid";
}

///////////////////////////////////////////////////////////////////////////////////////////////////

static struct {
    char const* text;
    char const* pretty;
    OperationKind enumerant;
} OperationKind_enumerants[] = {
        {"eq_gemm", "EqGemm", OperationKind::kEqGemm},
        {"gemm", "Gemm", OperationKind::kGemm},
        {"conv2d", "Conv2d", OperationKind::kConv2d},
        {"conv3d", "Conv3d", OperationKind::kConv3d},
        {"spgemm", "SparseGemm", OperationKind::kSparseGemm},
};

/// Converts a Status enumerant to a string
char const* to_string(OperationKind enumerant, bool pretty) {
    for (auto const& possible : OperationKind_enumerants) {
        if (enumerant == possible.enumerant) {
            if (pretty) {
                return possible.pretty;
            } else {
                return possible.text;
            }
        }
    }

    return pretty ? "Invalid" : "invalid";
}

/// Converts a Status enumerant from a string
template <>
OperationKind from_string<OperationKind>(std::string const& str) {
    for (auto const& possible : OperationKind_enumerants) {
        if ((str.compare(possible.text) == 0) || (str.compare(possible.pretty) == 0)) {
            return possible.enumerant;
        }
    }

    return OperationKind::kInvalid;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

static struct {
    char const* text;
    char const* pretty;
    Status enumerant;
} Status_enumerants[] = {
        {"success", "Success", Status::kSuccess},
        {"misaligned_operand", "Error: misaligned operand",
         Status::kErrorMisalignedOperand},
        {"invalid_problem", "Error: invalid problem", Status::kErrorInvalidProblem},
        {"not_supported", "Error: not supported", Status::kErrorNotSupported},
        {"internal", "Error: internal", Status::kErrorInternal}};

/// Converts a Status enumerant to a string
char const* to_string(Status status, bool pretty) {
    for (auto const& possible : Status_enumerants) {
        if (status == possible.enumerant) {
            if (pretty) {
                return possible.pretty;
            } else {
                return possible.text;
            }
        }
    }

    return pretty ? "Invalid" : "invalid";
}

/// Converts a Status enumerant from a string
template <>
Status from_string<Status>(std::string const& str) {
    for (auto const& possible : Status_enumerants) {
        if ((str.compare(possible.text) == 0) || (str.compare(possible.pretty) == 0)) {
            return possible.enumerant;
        }
    }

    return Status::kInvalid;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

static struct {
    char const* text;
    char const* pretty;
    NumericTypeID enumerant;
} NumericTypeID_enumerants[] = {
        {"unknown", "<unkown>", NumericTypeID::kUnknown},
        {"void", "Void", NumericTypeID::kVoid},
        {"b1", "B1", NumericTypeID::kB1},
        {"u2", "U2", NumericTypeID::kU2},
        {"u4", "U4", NumericTypeID::kU4},
        {"u8", "U8", NumericTypeID::kU8},
        {"u16", "U16", NumericTypeID::kU16},
        {"u32", "U32", NumericTypeID::kU32},
        {"u64", "U64", NumericTypeID::kU64},
        {"s2", "S2", NumericTypeID::kS2},
        {"s4", "S4", NumericTypeID::kS4},
        {"s8", "S8", NumericTypeID::kS8},
        {"s16", "S16", NumericTypeID::kS16},
        {"s32", "S32", NumericTypeID::kS32},
        {"s64", "S64", NumericTypeID::kS64},
        {"f16", "F16", NumericTypeID::kF16},
        {"bf16", "BF16", NumericTypeID::kBF16},
        {"f32", "F32", NumericTypeID::kF32},
        {"tf32", "TF32", NumericTypeID::kTF32},
        {"f64", "F64", NumericTypeID::kF64},
        {"cf16", "CF16", NumericTypeID::kCF16},
        {"cbf16", "CBF16", NumericTypeID::kCBF16},
        {"cf32", "CF32", NumericTypeID::kCF32},
        {"ctf32", "CTF32", NumericTypeID::kCTF32},
        {"cf64", "CF64", NumericTypeID::kCF64},
        {"cu2", "CU2", NumericTypeID::kCU2},
        {"cu4", "CU4", NumericTypeID::kCU4},
        {"cu8", "CU8", NumericTypeID::kCU8},
        {"cu16", "CU16", NumericTypeID::kCU16},
        {"cu32", "CU32", NumericTypeID::kCU32},
        {"cu64", "CU64", NumericTypeID::kCU64},
        {"cs2", "CS2", NumericTypeID::kCS2},
        {"cs4", "CS4", NumericTypeID::kCS4},
        {"cs8", "CS8", NumericTypeID::kCS8},
        {"cs16", "CS16", NumericTypeID::kCS16},
        {"cs32", "CS32", NumericTypeID::kCS32},
        {"cs64", "CS64", NumericTypeID::kCS64},
        {"*", "<unkown/enumerate all>", NumericTypeID::kUnknown}};

/// Converts a NumericTypeID enumerant to a string
char const* to_string(NumericTypeID type, bool pretty) {
    for (auto const& possible : NumericTypeID_enumerants) {
        if (type == possible.enumerant) {
            if (pretty) {
                return possible.pretty;
            } else {
                return possible.text;
            }
        }
    }

    return pretty ? "Invalid" : "invalid";
}

/// Parses a NumericTypeID enumerant from a string
template <>
NumericTypeID from_string<NumericTypeID>(std::string const& str) {
    for (auto const& possible : NumericTypeID_enumerants) {
        if ((str.compare(possible.text) == 0) || (str.compare(possible.pretty) == 0)) {
            return possible.enumerant;
        }
    }

    return NumericTypeID::kInvalid;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Returns the size of a data type in bits
int sizeof_bits(NumericTypeID type) {
    switch (type) {
        case NumericTypeID::kF16:
            return 16;
        case NumericTypeID::kBF16:
            return 16;
        case NumericTypeID::kTF32:
            return 32;
        case NumericTypeID::kF32:
            return 32;
        case NumericTypeID::kF64:
            return 64;
        case NumericTypeID::kCF16:
            return 32;
        case NumericTypeID::kCBF16:
            return 32;
        case NumericTypeID::kCF32:
            return 64;
        case NumericTypeID::kCTF32:
            return 64;
        case NumericTypeID::kCF64:
            return 128;
        case NumericTypeID::kS2:
            return 2;
        case NumericTypeID::kS4:
            return 4;
        case NumericTypeID::kS8:
            return 8;
        case NumericTypeID::kS16:
            return 16;
        case NumericTypeID::kS32:
            return 32;
        case NumericTypeID::kS64:
            return 64;
        case NumericTypeID::kU2:
            return 2;
        case NumericTypeID::kU4:
            return 4;
        case NumericTypeID::kU8:
            return 8;
        case NumericTypeID::kU16:
            return 16;
        case NumericTypeID::kU32:
            return 32;
        case NumericTypeID::kU64:
            return 64;
        case NumericTypeID::kB1:
            return 1;
        default:
            break;
    }
    return 0;
}

/// Returns true if the numeric type is a complex data type or false if
/// real-valued.
bool is_complex_type(NumericTypeID type) {
    switch (type) {
        case NumericTypeID::kCF16:
            return true;
        case NumericTypeID::kCF32:
            return true;
        case NumericTypeID::kCF64:
            return true;
        case NumericTypeID::kCBF16:
            return true;
        case NumericTypeID::kCTF32:
            return true;
        default:
            break;
    }
    return false;
}

/// Returns the field underlying a complex valued type
NumericTypeID get_real_type(NumericTypeID type) {
    switch (type) {
        case NumericTypeID::kCF16:
            return NumericTypeID::kF16;
        case NumericTypeID::kCF32:
            return NumericTypeID::kF32;
        case NumericTypeID::kCF64:
            return NumericTypeID::kF64;
        case NumericTypeID::kCBF16:
            return NumericTypeID::kBF16;
        case NumericTypeID::kCTF32:
            return NumericTypeID::kTF32;
        default:
            break;
    }
    return type;
}

/// Returns true if numeric type is integer
bool is_integer_type(NumericTypeID type) {
    switch (type) {
        case NumericTypeID::kS2:
            return true;
        case NumericTypeID::kS4:
            return true;
        case NumericTypeID::kS8:
            return true;
        case NumericTypeID::kS16:
            return true;
        case NumericTypeID::kS32:
            return true;
        case NumericTypeID::kS64:
            return true;
        case NumericTypeID::kU2:
            return true;
        case NumericTypeID::kU4:
            return true;
        case NumericTypeID::kU8:
            return true;
        case NumericTypeID::kU16:
            return true;
        case NumericTypeID::kU32:
            return true;
        case NumericTypeID::kU64:
            return true;
        default:
            break;
    }
    return false;
}

/// Returns true if numeric type is signed
bool is_signed_type(NumericTypeID type) {
    switch (type) {
        case NumericTypeID::kF16:
            return true;
        case NumericTypeID::kBF16:
            return true;
        case NumericTypeID::kTF32:
            return true;
        case NumericTypeID::kF32:
            return true;
        case NumericTypeID::kF64:
            return true;
        case NumericTypeID::kS2:
            return true;
        case NumericTypeID::kS4:
            return true;
        case NumericTypeID::kS8:
            return true;
        case NumericTypeID::kS16:
            return true;
        case NumericTypeID::kS32:
            return true;
        case NumericTypeID::kS64:
            return true;
        default:
            break;
    }
    return false;
}

/// Returns true if numeric type is a signed integer
bool is_signed_integer(NumericTypeID type) {
    return is_integer_type(type) && is_signed_type(type);
}

/// returns true if numeric type is an unsigned integer
bool is_unsigned_integer(NumericTypeID type) {
    return is_integer_type(type) && !is_signed_type(type);
}

/// Returns true if numeric type is floating-point type
bool is_float_type(NumericTypeID type) {
    switch (type) {
        case NumericTypeID::kF16:
            return true;
        case NumericTypeID::kBF16:
            return true;
        case NumericTypeID::kTF32:
            return true;
        case NumericTypeID::kF32:
            return true;
        case NumericTypeID::kF64:
            return true;
        case NumericTypeID::kCF16:
            return true;
        case NumericTypeID::kCBF16:
            return true;
        case NumericTypeID::kCTF32:
            return true;
        case NumericTypeID::kCF32:
            return true;
        case NumericTypeID::kCF64:
            return true;
        default:
            break;
    }
    return false;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

static struct {
    LayoutTypeID layout;
    char const* alias;
} layout_aliases[] = {
        {LayoutTypeID::kUnknown, "unknown"},
        {LayoutTypeID::kRowMajor, "row"},
        {LayoutTypeID::kRowMajor, "t"},
        {LayoutTypeID::kColumnMajor, "column"},
        {LayoutTypeID::kColumnMajor, "col"},
        {LayoutTypeID::kColumnMajor, "n"},

        {LayoutTypeID::kColumnMajorInterleavedK2, "nk2"},
        {LayoutTypeID::kRowMajorInterleavedK2, "tk2"},

        {LayoutTypeID::kColumnMajorInterleavedK4, "nk4"},
        {LayoutTypeID::kRowMajorInterleavedK4, "tk4"},

        {LayoutTypeID::kColumnMajorInterleavedK16, "nk16"},
        {LayoutTypeID::kRowMajorInterleavedK16, "tk16"},

        {LayoutTypeID::kColumnMajorInterleavedK32, "nk32"},
        {LayoutTypeID::kRowMajorInterleavedK32, "tk32"},

        {LayoutTypeID::kColumnMajorInterleavedK64, "nk64"},
        {LayoutTypeID::kRowMajorInterleavedK64, "tk64"},

        {LayoutTypeID::kTensorNCHW, "nchw"},
        {LayoutTypeID::kTensorNCDHW, "ncdhw"},
        {LayoutTypeID::kTensorNHWC, "nhwc"},
        {LayoutTypeID::kTensorNDHWC, "ndhwc"},
        {LayoutTypeID::kTensorNC4HW4, "nc4hw4"},
        {LayoutTypeID::kTensorNC8HW8, "nc8hw8"},
        {LayoutTypeID::kTensorNC16HW16, "nc16hw16"},
        {LayoutTypeID::kTensorNC32HW32, "nc32hw32"},
        {LayoutTypeID::kTensorNC64HW64, "nc64hw64"},
        {LayoutTypeID::kTensorC4RSK4, "c4rsk4"},
        {LayoutTypeID::kTensorC8RSK8, "c8rsk8"},
        {LayoutTypeID::kTensorC16RSK16, "c16rsk16"},
        {LayoutTypeID::kTensorC32RSK32, "c32rsk32"},
        {LayoutTypeID::kTensorC64RSK64, "c64rsk64"},
        {LayoutTypeID::kTensorK4RSC4, "k4rsc4"},
        {LayoutTypeID::kTensorCK4RS4, "ck4rs4"},
        {LayoutTypeID::kTensorCK8RS8, "ck8rs8"},
        {LayoutTypeID::kTensorCK16RS16, "ck16rs16"},
        {LayoutTypeID::kUnknown, "*"},
        {LayoutTypeID::kInvalid, nullptr}};

/// Converts a LayoutTypeID enumerant to a string
char const* to_string(LayoutTypeID layout, bool pretty) {
    for (auto const& alias : layout_aliases) {
        if (alias.layout == layout) {
            return alias.alias;
        }
    }
    return pretty ? "Invalid" : "invalid";
}

/// Parses a LayoutTypeID enumerant from a string
template <>
LayoutTypeID from_string<LayoutTypeID>(std::string const& str) {
    for (auto const& alias : layout_aliases) {
        if (str.compare(alias.alias) == 0) {
            return alias.layout;
        }
    }
    return LayoutTypeID::kInvalid;
}

/// Gets stride rank for the layout_id (static function)
int get_layout_stride_rank(LayoutTypeID layout_id) {
    switch (layout_id) {
        case LayoutTypeID::kColumnMajor:
            return cutlass::layout::ColumnMajor::kStrideRank;
        case LayoutTypeID::kRowMajor:
            return cutlass::layout::RowMajor::kStrideRank;
        case LayoutTypeID::kColumnMajorInterleavedK2:
            return cutlass::layout::ColumnMajorInterleaved<2>::kStrideRank;
        case LayoutTypeID::kRowMajorInterleavedK2:
            return cutlass::layout::RowMajorInterleaved<2>::kStrideRank;
        case LayoutTypeID::kColumnMajorInterleavedK4:
            return cutlass::layout::ColumnMajorInterleaved<4>::kStrideRank;
        case LayoutTypeID::kRowMajorInterleavedK4:
            return cutlass::layout::RowMajorInterleaved<4>::kStrideRank;
        case LayoutTypeID::kColumnMajorInterleavedK16:
            return cutlass::layout::ColumnMajorInterleaved<16>::kStrideRank;
        case LayoutTypeID::kRowMajorInterleavedK16:
            return cutlass::layout::RowMajorInterleaved<16>::kStrideRank;
        case LayoutTypeID::kColumnMajorInterleavedK32:
            return cutlass::layout::ColumnMajorInterleaved<32>::kStrideRank;
        case LayoutTypeID::kRowMajorInterleavedK32:
            return cutlass::layout::RowMajorInterleaved<32>::kStrideRank;
        case LayoutTypeID::kColumnMajorInterleavedK64:
            return cutlass::layout::ColumnMajorInterleaved<64>::kStrideRank;
        case LayoutTypeID::kRowMajorInterleavedK64:
            return cutlass::layout::RowMajorInterleaved<64>::kStrideRank;
        case LayoutTypeID::kTensorNCHW:
            return cutlass::layout::TensorNCHW::kStrideRank;
        case LayoutTypeID::kTensorNHWC:
            return cutlass::layout::TensorNHWC::kStrideRank;
        case LayoutTypeID::kTensorNDHWC:
            return cutlass::layout::TensorNDHWC::kStrideRank;
        case LayoutTypeID::kTensorNC32HW32:
            return cutlass::layout::TensorNCxHWx<32>::kStrideRank;
        case LayoutTypeID::kTensorNC64HW64:
            return cutlass::layout::TensorNCxHWx<64>::kStrideRank;
        case LayoutTypeID::kTensorC32RSK32:
            return cutlass::layout::TensorCxRSKx<32>::kStrideRank;
        case LayoutTypeID::kTensorC64RSK64:
            return cutlass::layout::TensorCxRSKx<64>::kStrideRank;
        default:
            throw std::runtime_error(
                    "Unsupported LayoutTypeID in LayoutType::get_stride_rank");
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////

static struct {
    char const* text;
    char const* pretty;
    OpcodeClassID enumerant;
} OpcodeClassID_enumerants[] = {
        {"simt", "<simt>", OpcodeClassID::kSimt},
        {"tensorop", "<tensorop>", OpcodeClassID::kTensorOp},
        {"wmmatensorop", "<wmmatensorop>", OpcodeClassID::kWmmaTensorOp},
        {"wmma", "<wmma>", OpcodeClassID::kWmmaTensorOp},
};

/// Converts a OpcodeClassID enumerant to a string
char const* to_string(OpcodeClassID type, bool pretty) {
    for (auto const& possible : OpcodeClassID_enumerants) {
        if (type == possible.enumerant) {
            if (pretty) {
                return possible.pretty;
            } else {
                return possible.text;
            }
        }
    }

    return pretty ? "Invalid" : "invalid";
}

/// Converts a OpcodeClassID enumerant from a string
template <>
OpcodeClassID from_string<OpcodeClassID>(std::string const& str) {
    for (auto const& possible : OpcodeClassID_enumerants) {
        if ((str.compare(possible.text) == 0) || (str.compare(possible.pretty) == 0)) {
            return possible.enumerant;
        }
    }

    return OpcodeClassID::kInvalid;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

static struct {
    char const* text;
    char const* pretty;
    ComplexTransform enumerant;
} ComplexTransform_enumerants[] = {
        {"n", "none", ComplexTransform::kNone},
        {"c", "conj", ComplexTransform::kConjugate}};

/// Converts a ComplexTransform enumerant to a string
char const* to_string(ComplexTransform type, bool pretty) {
    for (auto const& possible : ComplexTransform_enumerants) {
        if (type == possible.enumerant) {
            if (pretty) {
                return possible.pretty;
            } else {
                return possible.text;
            }
        }
    }

    return pretty ? "Invalid" : "invalid";
}

/// Converts a ComplexTransform enumerant from a string
template <>
ComplexTransform from_string<ComplexTransform>(std::string const& str) {
    for (auto const& possible : ComplexTransform_enumerants) {
        if ((str.compare(possible.text) == 0) || (str.compare(possible.pretty) == 0)) {
            return possible.enumerant;
        }
    }

    return ComplexTransform::kInvalid;
}

static struct {
    char const* text;
    char const* pretty;
    SplitKMode enumerant;
} SplitKMode_enumerants[] = {
        {"none", "<none>", SplitKMode::kNone},
        {"serial", "<serial>", SplitKMode::kSerial},
        {"parallel", "<parallel>", SplitKMode::kParallel},
};

/// Converts a SplitKMode enumerant to a string
char const* to_string(SplitKMode type, bool pretty) {
    for (auto const& possible : SplitKMode_enumerants) {
        if (type == possible.enumerant) {
            if (pretty) {
                return possible.pretty;
            } else {
                return possible.text;
            }
        }
    }

    return pretty ? "Invalid" : "invalid";
}

/// Converts a SplitKMode enumerant from a string
template <>
SplitKMode from_string<SplitKMode>(std::string const& str) {
    for (auto const& possible : SplitKMode_enumerants) {
        if ((str.compare(possible.text) == 0) || (str.compare(possible.pretty) == 0)) {
            return possible.enumerant;
        }
    }

    return SplitKMode::kInvalid;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

static struct {
    char const* text;
    char const* pretty;
    ConvModeID enumerant;
} ConvModeID_enumerants[] = {
        {"cross", "<cross>", ConvModeID::kCrossCorrelation},
        {"conv", "<conv>", ConvModeID::kConvolution},
};

/// Converts a ConvModeID enumerant to a string
char const* to_string(ConvModeID type, bool pretty) {
    for (auto const& possible : ConvModeID_enumerants) {
        if (type == possible.enumerant) {
            if (pretty) {
                return possible.pretty;
            } else {
                return possible.text;
            }
        }
    }

    return pretty ? "Invalid" : "invalid";
}

/// Converts a ConvModeID enumerant from a string
template <>
ConvModeID from_string<ConvModeID>(std::string const& str) {
    for (auto const& possible : ConvModeID_enumerants) {
        if ((str.compare(possible.text) == 0) || (str.compare(possible.pretty) == 0)) {
            return possible.enumerant;
        }
    }

    return ConvModeID::kInvalid;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

static struct {
    char const* text;
    char const* pretty;
    IteratorAlgorithmID enumerant;
} IteratorAlgorithmID_enumerants[] = {
        {"none", "<none>", IteratorAlgorithmID::kNone},
        {"analytic", "<analytic>", IteratorAlgorithmID::kAnalytic},
        {"optimized", "<optimized>", IteratorAlgorithmID::kOptimized},
};

/// Converts a ConvModeID enumerant to a string
char const* to_string(IteratorAlgorithmID type, bool pretty) {
    for (auto const& possible : IteratorAlgorithmID_enumerants) {
        if (type == possible.enumerant) {
            if (pretty) {
                return possible.pretty;
            } else {
                return possible.text;
            }
        }
    }

    return pretty ? "Invalid" : "invalid";
}

/// Converts a ConvModeID enumerant from a string
template <>
IteratorAlgorithmID from_string<IteratorAlgorithmID>(std::string const& str) {
    for (auto const& possible : IteratorAlgorithmID_enumerants) {
        if ((str.compare(possible.text) == 0) || (str.compare(possible.pretty) == 0)) {
            return possible.enumerant;
        }
    }

    return IteratorAlgorithmID::kInvalid;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

static struct {
    char const* text;
    char const* pretty;
    ConvKind enumerant;
} ConvKind_enumerants[] = {
        {"unknown", "<unkown>", ConvKind::kUnknown},
        {"fprop", "<fprop>", ConvKind::kFprop},
        {"dgrad", "<dgrad>", ConvKind::kDgrad},
        {"wgrad", "<wgrad>", ConvKind::kWgrad},
};

/// Converts a ConvKind enumerant to a string
char const* to_string(ConvKind type, bool pretty) {
    for (auto const& possible : ConvKind_enumerants) {
        if (type == possible.enumerant) {
            if (pretty) {
                return possible.pretty;
            } else {
                return possible.text;
            }
        }
    }

    return pretty ? "Invalid" : "invalid";
}

/// Converts a ConvKind enumerant from a string
template <>
ConvKind from_string<ConvKind>(std::string const& str) {
    for (auto const& possible : ConvKind_enumerants) {
        if ((str.compare(possible.text) == 0) || (str.compare(possible.pretty) == 0)) {
            return possible.enumerant;
        }
    }

    return ConvKind::kInvalid;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Lexical cast a string to a byte array. Returns true if cast is successful or
/// false if invalid.
bool lexical_cast(
        std::vector<uint8_t>& bytes, NumericTypeID type, std::string const& str) {
    int size_bytes = sizeof_bits(type) / 8;
    if (!size_bytes) {
        return false;
    }

    bytes.resize(size_bytes, 0);

    std::stringstream ss;
    ss << str;

    switch (type) {
        case NumericTypeID::kU8: {
            ss >> *reinterpret_cast<uint8_t*>(bytes.data());
        } break;
        case NumericTypeID::kU16: {
            ss >> *reinterpret_cast<uint16_t*>(bytes.data());
        } break;
        case NumericTypeID::kU32: {
            ss >> *reinterpret_cast<uint32_t*>(bytes.data());
        } break;
        case NumericTypeID::kU64: {
            ss >> *reinterpret_cast<uint64_t*>(bytes.data());
        } break;
        case NumericTypeID::kS8: {
            ss >> *reinterpret_cast<int8_t*>(bytes.data());
        } break;
        case NumericTypeID::kS16: {
            ss >> *reinterpret_cast<int16_t*>(bytes.data());
        } break;
        case NumericTypeID::kS32: {
            ss >> *reinterpret_cast<int32_t*>(bytes.data());
        } break;
        case NumericTypeID::kS64: {
            ss >> *reinterpret_cast<int64_t*>(bytes.data());
        } break;
        case NumericTypeID::kF16: {
            float tmp;
            ss >> tmp;
            *reinterpret_cast<half_t*>(bytes.data()) = static_cast<half_t>(tmp);
        } break;
        case NumericTypeID::kBF16: {
            float tmp;
            ss >> tmp;
            *reinterpret_cast<bfloat16_t*>(bytes.data()) = static_cast<bfloat16_t>(tmp);
        } break;
        case NumericTypeID::kTF32: {
            float tmp;
            ss >> tmp;
            *reinterpret_cast<tfloat32_t*>(bytes.data()) = static_cast<tfloat32_t>(tmp);
        } break;
        case NumericTypeID::kF32: {
            ss >> *reinterpret_cast<float*>(bytes.data());
        } break;
        case NumericTypeID::kF64: {
            ss >> *reinterpret_cast<double*>(bytes.data());
        } break;
        case NumericTypeID::kCF16: {
            std::complex<float> tmp;
            ss >> tmp;
            cutlass::complex<cutlass::half_t>* x =
                    reinterpret_cast<cutlass::complex<half_t>*>(bytes.data());
            x->real() = static_cast<half_t>(std::real(tmp));
            x->imag() = static_cast<half_t>(std::imag(tmp));
        } break;
        case NumericTypeID::kCBF16: {
            std::complex<float> tmp;
            ss >> tmp;
            cutlass::complex<cutlass::bfloat16_t>* x =
                    reinterpret_cast<cutlass::complex<bfloat16_t>*>(bytes.data());
            x->real() = static_cast<bfloat16_t>(std::real(tmp));
            x->imag() = static_cast<bfloat16_t>(std::imag(tmp));
        } break;
        case NumericTypeID::kCF32: {
            ss >> *reinterpret_cast<std::complex<float>*>(bytes.data());
        } break;
        case NumericTypeID::kCTF32: {
            std::complex<float> tmp;
            ss >> tmp;
            cutlass::complex<cutlass::tfloat32_t>* x =
                    reinterpret_cast<cutlass::complex<tfloat32_t>*>(bytes.data());
            x->real() = static_cast<tfloat32_t>(std::real(tmp));
            x->imag() = static_cast<tfloat32_t>(std::imag(tmp));
        } break;
        case NumericTypeID::kCF64: {
            ss >> *reinterpret_cast<std::complex<double>*>(bytes.data());
        } break;
        default:
            return false;
    }

    return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

std::string lexical_cast(int64_t int_value) {
    std::stringstream ss;
    ss << int_value;
    return ss.str();
}

/// Lexical cast TO a string FROM a byte array. Returns true if cast is
/// successful or false if invalid.
std::string lexical_cast(std::vector<uint8_t>& bytes, NumericTypeID type) {
    size_t size_bytes = sizeof_bits(type) / 8;

    if (!size_bytes || size_bytes != bytes.size()) {
        return "<invalid>";
    }

    bytes.resize(size_bytes, 0);

    std::stringstream ss;

    switch (type) {
        case NumericTypeID::kU8: {
            ss << *reinterpret_cast<uint8_t*>(bytes.data());
        } break;
        case NumericTypeID::kU16: {
            ss << *reinterpret_cast<uint16_t*>(bytes.data());
        } break;
        case NumericTypeID::kU32: {
            ss << *reinterpret_cast<uint32_t*>(bytes.data());
        } break;
        case NumericTypeID::kU64: {
            ss << *reinterpret_cast<uint64_t*>(bytes.data());
        } break;
        case NumericTypeID::kS8: {
            ss << *reinterpret_cast<int8_t*>(bytes.data());
        } break;
        case NumericTypeID::kS16: {
            ss << *reinterpret_cast<int16_t*>(bytes.data());
        } break;
        case NumericTypeID::kS32: {
            ss << *reinterpret_cast<int32_t*>(bytes.data());
        } break;
        case NumericTypeID::kS64: {
            ss << *reinterpret_cast<int64_t*>(bytes.data());
        } break;
        case NumericTypeID::kF16: {
            float tmp = *reinterpret_cast<half_t*>(bytes.data());
            ss << tmp;
        } break;
        case NumericTypeID::kBF16: {
            float tmp = *reinterpret_cast<bfloat16_t*>(bytes.data());
            ;
            ss << tmp;
        } break;
        case NumericTypeID::kTF32: {
            float tmp = *reinterpret_cast<tfloat32_t*>(bytes.data());
            ;
            ss << tmp;
        } break;
        case NumericTypeID::kF32: {
            ss << *reinterpret_cast<float*>(bytes.data());
        } break;
        case NumericTypeID::kF64: {
            ss << *reinterpret_cast<double*>(bytes.data());
        } break;
        case NumericTypeID::kCF16: {
            cutlass::complex<half_t> const* x =
                    reinterpret_cast<cutlass::complex<half_t> const*>(bytes.data());

            ss << float(x->real());

            if (x->imag() != cutlass::half_t()) {
                ss << "+i" << float(x->imag());
            }
        } break;
        case NumericTypeID::kCBF16: {
            cutlass::complex<bfloat16_t> const* x =
                    reinterpret_cast<cutlass::complex<bfloat16_t> const*>(bytes.data());

            ss << float(x->real());

            if (x->imag() != cutlass::bfloat16_t()) {
                ss << "+i" << float(x->imag());
            }
        } break;
        case NumericTypeID::kCF32: {
            cutlass::complex<float> const* x =
                    reinterpret_cast<cutlass::complex<float> const*>(bytes.data());

            ss << x->real();

            if (x->imag() != float()) {
                ss << "+i" << x->imag();
            }
        } break;
        case NumericTypeID::kCTF32: {
            cutlass::complex<tfloat32_t> const* x =
                    reinterpret_cast<cutlass::complex<tfloat32_t> const*>(bytes.data());

            ss << float(x->real());

            if (x->imag() != tfloat32_t()) {
                ss << "+i" << float(x->imag());
            }
        } break;
        case NumericTypeID::kCF64: {
            cutlass::complex<double> const* x =
                    reinterpret_cast<cutlass::complex<double> const*>(bytes.data());

            ss << x->real();

            if (x->imag() != double()) {
                ss << "+i" << x->imag();
            }
        } break;
        default:
            return "<unknown>";
    }

    return ss.str();
}

/// Casts from a signed int64 to the destination type. Returns true if
/// successful.
bool cast_from_int64(std::vector<uint8_t>& bytes, NumericTypeID type, int64_t src) {
    int size_bytes = sizeof_bits(type) / 8;
    if (!size_bytes) {
        return false;
    }

    bytes.resize(size_bytes, 0);

    switch (type) {
        case NumericTypeID::kU8: {
            *reinterpret_cast<uint8_t*>(bytes.data()) = static_cast<uint8_t>(src);
        } break;
        case NumericTypeID::kU16: {
            *reinterpret_cast<uint16_t*>(bytes.data()) = static_cast<uint16_t>(src);
        } break;
        case NumericTypeID::kU32: {
            *reinterpret_cast<uint32_t*>(bytes.data()) = static_cast<uint32_t>(src);
        } break;
        case NumericTypeID::kU64: {
            *reinterpret_cast<uint64_t*>(bytes.data()) = static_cast<uint64_t>(src);
        } break;
        case NumericTypeID::kS8: {
            *reinterpret_cast<int8_t*>(bytes.data()) = static_cast<int8_t>(src);
        } break;
        case NumericTypeID::kS16: {
            *reinterpret_cast<int16_t*>(bytes.data()) = static_cast<int16_t>(src);
        } break;
        case NumericTypeID::kS32: {
            *reinterpret_cast<int32_t*>(bytes.data()) = static_cast<int32_t>(src);
        } break;
        case NumericTypeID::kS64: {
            *reinterpret_cast<int64_t*>(bytes.data()) = static_cast<int64_t>(src);
        } break;
        case NumericTypeID::kF16: {
            *reinterpret_cast<half_t*>(bytes.data()) = static_cast<half_t>(float(src));
        } break;
        case NumericTypeID::kBF16: {
            *reinterpret_cast<bfloat16_t*>(bytes.data()) =
                    static_cast<bfloat16_t>(float(src));
        } break;
        case NumericTypeID::kTF32: {
            *reinterpret_cast<tfloat32_t*>(bytes.data()) =
                    static_cast<tfloat32_t>(float(src));
        } break;
        case NumericTypeID::kF32: {
            *reinterpret_cast<float*>(bytes.data()) = static_cast<float>(src);
        } break;
        case NumericTypeID::kF64: {
            *reinterpret_cast<double*>(bytes.data()) = double(src);
        } break;
        case NumericTypeID::kCF16: {
            cutlass::complex<cutlass::half_t>* x =
                    reinterpret_cast<cutlass::complex<half_t>*>(bytes.data());
            x->real() = static_cast<half_t>(float(src));
            x->imag() = static_cast<half_t>(float(0));
        } break;
        case NumericTypeID::kCF32: {
            *reinterpret_cast<cutlass::complex<float>*>(bytes.data()) =
                    cutlass::complex<float>(float(src), float(0));
        } break;
        case NumericTypeID::kCF64: {
            *reinterpret_cast<cutlass::complex<double>*>(bytes.data()) =
                    cutlass::complex<double>(double(src), double(0));
        } break;
        default:
            return false;
    }

    return true;
}

/// Casts from an unsigned int64 to the destination type. Returns true if
/// successful.
bool cast_from_uint64(std::vector<uint8_t>& bytes, NumericTypeID type, uint64_t src) {
    int size_bytes = sizeof_bits(type) / 8;
    if (!size_bytes) {
        return false;
    }

    bytes.resize(size_bytes, 0);

    switch (type) {
        case NumericTypeID::kU8: {
            *reinterpret_cast<uint8_t*>(bytes.data()) = static_cast<uint8_t>(src);
        } break;
        case NumericTypeID::kU16: {
            *reinterpret_cast<uint16_t*>(bytes.data()) = static_cast<uint16_t>(src);
        } break;
        case NumericTypeID::kU32: {
            *reinterpret_cast<uint32_t*>(bytes.data()) = static_cast<uint32_t>(src);
        } break;
        case NumericTypeID::kU64: {
            *reinterpret_cast<uint64_t*>(bytes.data()) = static_cast<uint64_t>(src);
        } break;
        case NumericTypeID::kS8: {
            *reinterpret_cast<int8_t*>(bytes.data()) = static_cast<int8_t>(src);
        } break;
        case NumericTypeID::kS16: {
            *reinterpret_cast<int16_t*>(bytes.data()) = static_cast<int16_t>(src);
        } break;
        case NumericTypeID::kS32: {
            *reinterpret_cast<int32_t*>(bytes.data()) = static_cast<int32_t>(src);
        } break;
        case NumericTypeID::kS64: {
            *reinterpret_cast<int64_t*>(bytes.data()) = static_cast<int64_t>(src);
        } break;
        case NumericTypeID::kF16: {
            *reinterpret_cast<half_t*>(bytes.data()) = static_cast<half_t>(float(src));
        } break;
        case NumericTypeID::kBF16: {
            *reinterpret_cast<bfloat16_t*>(bytes.data()) =
                    static_cast<bfloat16_t>(float(src));
        } break;
        case NumericTypeID::kTF32: {
            *reinterpret_cast<tfloat32_t*>(bytes.data()) =
                    static_cast<tfloat32_t>(float(src));
        } break;
        case NumericTypeID::kF32: {
            *reinterpret_cast<float*>(bytes.data()) = static_cast<float>(src);
        } break;
        case NumericTypeID::kF64: {
            *reinterpret_cast<double*>(bytes.data()) = double(src);
        } break;
        case NumericTypeID::kCF16: {
            cutlass::complex<cutlass::half_t>* x =
                    reinterpret_cast<cutlass::complex<half_t>*>(bytes.data());
            x->real() = static_cast<half_t>(float(src));
            x->imag() = static_cast<half_t>(float(0));
        } break;
        case NumericTypeID::kCF32: {
            *reinterpret_cast<std::complex<float>*>(bytes.data()) =
                    std::complex<float>(float(src), float(0));
        } break;
        case NumericTypeID::kCF64: {
            *reinterpret_cast<std::complex<double>*>(bytes.data()) =
                    std::complex<double>(double(src), double(0));
        } break;
        default:
            return false;
    }

    return true;
}

/// Lexical cast a string to a byte array. Returns true if cast is successful or
/// false if invalid.
bool cast_from_double(std::vector<uint8_t>& bytes, NumericTypeID type, double src) {
    int size_bytes = sizeof_bits(type) / 8;
    if (!size_bytes) {
        return false;
    }

    bytes.resize(size_bytes, 0);

    switch (type) {
        case NumericTypeID::kU8: {
            *reinterpret_cast<uint8_t*>(bytes.data()) = static_cast<uint8_t>(src);
        } break;
        case NumericTypeID::kU16: {
            *reinterpret_cast<uint16_t*>(bytes.data()) = static_cast<uint16_t>(src);
        } break;
        case NumericTypeID::kU32: {
            *reinterpret_cast<uint32_t*>(bytes.data()) = static_cast<uint32_t>(src);
        } break;
        case NumericTypeID::kU64: {
            *reinterpret_cast<uint64_t*>(bytes.data()) = static_cast<uint64_t>(src);
        } break;
        case NumericTypeID::kS8: {
            *reinterpret_cast<int8_t*>(bytes.data()) = static_cast<int8_t>(src);
        } break;
        case NumericTypeID::kS16: {
            *reinterpret_cast<int16_t*>(bytes.data()) = static_cast<int16_t>(src);
        } break;
        case NumericTypeID::kS32: {
            *reinterpret_cast<int32_t*>(bytes.data()) = static_cast<int32_t>(src);
        } break;
        case NumericTypeID::kS64: {
            *reinterpret_cast<int64_t*>(bytes.data()) = static_cast<int64_t>(src);
        } break;
        case NumericTypeID::kF16: {
            *reinterpret_cast<half_t*>(bytes.data()) = static_cast<half_t>(float(src));
        } break;
        case NumericTypeID::kBF16: {
            *reinterpret_cast<bfloat16_t*>(bytes.data()) =
                    static_cast<bfloat16_t>(float(src));
        } break;
        case NumericTypeID::kTF32: {
            *reinterpret_cast<tfloat32_t*>(bytes.data()) =
                    static_cast<tfloat32_t>(float(src));
        } break;
        case NumericTypeID::kF32: {
            *reinterpret_cast<float*>(bytes.data()) = static_cast<float>(src);
        } break;
        case NumericTypeID::kF64: {
            *reinterpret_cast<double*>(bytes.data()) = src;
        } break;
        case NumericTypeID::kCF16: {
            cutlass::complex<cutlass::half_t>* x =
                    reinterpret_cast<cutlass::complex<half_t>*>(bytes.data());
            x->real() = static_cast<half_t>(float(src));
            x->imag() = static_cast<half_t>(float(0));
        } break;
        case NumericTypeID::kCBF16: {
            cutlass::complex<cutlass::bfloat16_t>* x =
                    reinterpret_cast<cutlass::complex<bfloat16_t>*>(bytes.data());
            x->real() = static_cast<bfloat16_t>(bfloat16_t(src));
            x->imag() = static_cast<bfloat16_t>(bfloat16_t(0));
        } break;
        case NumericTypeID::kCF32: {
            *reinterpret_cast<cutlass::complex<float>*>(bytes.data()) =
                    cutlass::complex<float>(float(src), float());
        } break;
        case NumericTypeID::kCTF32: {
            *reinterpret_cast<cutlass::complex<tfloat32_t>*>(bytes.data()) =
                    cutlass::complex<tfloat32_t>(tfloat32_t(src), tfloat32_t());
        } break;
        case NumericTypeID::kCF64: {
            *reinterpret_cast<cutlass::complex<double>*>(bytes.data()) =
                    cutlass::complex<double>(src, double());
        } break;
        default:
            return false;
    }

    return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

static struct {
    char const* text;
    char const* pretty;
    conv::Operator enumerant;
} ConvOperator_enumerants[] = {
        {"fprop", "Fprop", conv::Operator::kFprop},
        {"dgrad", "Dgrad", conv::Operator::kDgrad},
        {"wgrad", "Wgrad", conv::Operator::kWgrad},
};

/// Converts a conv::Operator enumerant to a string
char const* to_string(conv::Operator conv_op, bool pretty) {
    for (auto const& possible : ConvOperator_enumerants) {
        if (conv_op == possible.enumerant) {
            if (pretty) {
                return possible.pretty;
            } else {
                return possible.text;
            }
        }
    }

    return pretty ? "Invalid" : "invalid";
}

///////////////////////////////////////////////////////////////////////////////////////////////////

static struct {
    char const* text;
    char const* pretty;
    conv::ConvType enumerant;
} ConvType_enumerants[] = {
        {"convolution", "Convolution", conv::ConvType::kConvolution},
        {"batch_convolution", "BatchConvolution", conv::ConvType::kBatchConvolution},
        {"local", "Local", conv::ConvType::kLocal},
        {"local_share", "LocalShare", conv::ConvType::kLocalShare},
};

/// Converts a ConvType enumerant to a string
char const* to_string(conv::ConvType type, bool pretty) {
    for (auto const& possible : ConvType_enumerants) {
        if (type == possible.enumerant) {
            if (pretty) {
                return possible.pretty;
            } else {
                return possible.text;
            }
        }
    }

    return pretty ? "Invalid" : "invalid";
}

///////////////////////////////////////////////////////////////////////////////////////////////////

static struct {
    char const* text;
    char const* pretty;
    ArchTagID enumerant;
} ArchTagID_enumerants[] = {
        {"sm_50", "Sm50", ArchTagID::kSm50}, {"sm_60", "Sm60", ArchTagID::kSm60},
        {"sm_61", "Sm61", ArchTagID::kSm61}, {"sm_70", "Sm70", ArchTagID::kSm70},
        {"sm_72", "Sm72", ArchTagID::kSm72}, {"sm_75", "Sm75", ArchTagID::kSm75},
        {"sm_80", "Sm80", ArchTagID::kSm80}, {"sm_86", "Sm86", ArchTagID::kSm86},
};

/// Converts an ArchTagID enumerant to a string
char const* to_string(ArchTagID tag, bool pretty) {
    for (auto const& possible : ArchTagID_enumerants) {
        if (tag == possible.enumerant) {
            if (pretty) {
                return possible.pretty;
            } else {
                return possible.text;
            }
        }
    }

    return pretty ? "Invalid" : "invalid";
}

///////////////////////////////////////////////////////////////////////////////////////////////////

static struct {
    char const* text;
    char const* pretty;
    epilogue::EpilogueType enumerant;
} EpilogueType_enumerants[] = {
        {"bias_add_linear_combination", "BiasAddLinearCombination",
         epilogue::EpilogueType::kBiasAddLinearCombination},
        {"bias_add_linear_combination_clamp", "BiasAddLinearCombinationClamp",
         epilogue::EpilogueType::kBiasAddLinearCombinationClamp},
        {"bias_add_linear_combination_hswish", "BiasAddLinearCombinationHSwish",
         epilogue::EpilogueType::kBiasAddLinearCombinationHSwish},
        {"bias_add_linear_combination_hswish_clamp",
         "BiasAddLinearCombinationHSwishClamp",
         epilogue::EpilogueType::kBiasAddLinearCombinationHSwishClamp},
        {"bias_add_linear_combination_relu", "BiasAddLinearCombinationRelu",
         epilogue::EpilogueType::kBiasAddLinearCombinationRelu},
        {"bias_add_linear_combination_relu_clamp", "BiasAddLinearCombinationReluClamp",
         epilogue::EpilogueType::kBiasAddLinearCombinationReluClamp},
        {"conversion", "Conversion", epilogue::EpilogueType::kConversion},
        {"linear_combination", "LinearCombination",
         epilogue::EpilogueType::kLinearCombination},
        {"linear_combination_clamp", "LinearCombination_clamp",
         epilogue::EpilogueType::kLinearCombinationClamp},
        {"linear_combination_planar_complex", "LinearCombinationPlanarComplex",
         epilogue::EpilogueType::kLinearCombinationPlanarComplex},
        {"linear_combination_relu", "LinearCombinationRelu",
         epilogue::EpilogueType::kLinearCombinationRelu},
        {"linear_combination_sigmoid", "LinearCombinationSigmoid",
         epilogue::EpilogueType::kLinearCombinationSigmoid},
};

/// Converts an EpilogueType enumerant to a string
char const* to_string(epilogue::EpilogueType type, bool pretty) {
    for (auto const& possible : EpilogueType_enumerants) {
        if (type == possible.enumerant) {
            if (pretty) {
                return possible.pretty;
            } else {
                return possible.text;
            }
        }
    }

    return pretty ? "Invalid" : "invalid";
}

///////////////////////////////////////////////////////////////////////////////////////////////////

static struct {
    char const* text;
    char const* pretty;
    ThreadblockSwizzleID enumerant;
} ThreadblockSwizzleID_enumerants[] = {
        {"gemm_identity", "GemmIdentityThreadblockSwizzle",
         ThreadblockSwizzleID::kGemmIdentity},
        {"gemm_horizontal", "GemmHorizontalThreadblockSwizzle",
         ThreadblockSwizzleID::kGemmHorizontal},
        {"gemm_batched_identity", "GemmBatchedIdentityThreadblockSwizzle",
         ThreadblockSwizzleID::kGemmBatchedIdentity},
        {"gemm_split_k_identity", "GemmSplitKIdentityThreadblockSwizzle",
         ThreadblockSwizzleID::kGemmSplitKIdentity},
        {"gemm_split_k_horizontal", "GemmSplitKHorizontalThreadblockSwizzle",
         ThreadblockSwizzleID::kGemmSplitKHorizontal},
        {"gemv_batched_strided_default", "GemvBatchedStridedThreadblockDefaultSwizzle",
         ThreadblockSwizzleID::kGemvBatchedStridedDefault},
        {"gemv_batched_strided_reduction",
         "GemvBatchedStridedThreadblockReductionSwizzle",
         ThreadblockSwizzleID::kGemvBatchedStridedReduction},
        {"convolution_fprop_cxrskx", "ConvolutionFpropCxRSKxThreadblockSwizzle",
         ThreadblockSwizzleID::kConvolutionFpropCxRSKx},
        {"convolution_dgrad_cxrskx", "ConvolutionDgradCxRSKxThreadblockSwizzle",
         ThreadblockSwizzleID::kConvolutionDgradCxRSKx},
        {"convolution_fprop_ncxhwx", "ConvolutionFpropNCxHWxThreadblockSwizzle",
         ThreadblockSwizzleID::kConvolutionFpropNCxHWx},
        {"convolution_fprop_nhwc", "ConvolutionFpropTransThreadblockSwizzle",
         ThreadblockSwizzleID::kConvolutionFpropTrans},
        {"convolution_dgrad_ncxhwx", "ConvolutionDgradNCxHWxThreadblockSwizzle",
         ThreadblockSwizzleID::kConvolutionDgradNCxHWx},
        {"convolution_dgrad_ncxhwx", "ConvolutionDgradTransThreadblockSwizzle",
         ThreadblockSwizzleID::kConvolutionDgradTrans},
};

/// Converts a ThreadblockSwizzleID enumerant to a string
char const* to_string(ThreadblockSwizzleID threadblock_swizzle, bool pretty) {
    for (auto const& possible : ThreadblockSwizzleID_enumerants) {
        if (threadblock_swizzle == possible.enumerant) {
            if (pretty) {
                return possible.pretty;
            } else {
                return possible.text;
            }
        }
    }

    return pretty ? "Invalid" : "invalid";
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Converts a bool value to a string
char const* to_string(bool val, bool pretty) {
    if (val) {
        return pretty ? "True" : "true";
    } else {
        return pretty ? "False" : "false";
    }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

static struct {
    char const* text;
    char const* pretty;
    MathOperationID enumerant;
} MathOperationID_enumerants[] = {
        {"add", "Add", MathOperationID::kAdd},
        {"multiply_add", "MultiplyAdd", MathOperationID::kMultiplyAdd},
        {"multiply_add_saturate", "MultiplyAddSaturate",
         MathOperationID::kMultiplyAddSaturate},
        {"multiply_add_fast_bf16", "MultiplyAddFastBF16",
         MathOperationID::kMultiplyAddFastBF16},
        {"multiply_add_fast_f16", "MultiplyAddFastF16",
         MathOperationID::kMultiplyAddFastF16},
        {"multiply_add_complex", "MultiplyAddComplex",
         MathOperationID::kMultiplyAddComplex},
        {"multiply_add_gaussian_complex", "MultiplyAddGaussianComplex",
         MathOperationID::kMultiplyAddGaussianComplex},
        {"xor_popc", "XorPopc", MathOperationID::kXorPopc},
};

/// Converts a MathOperationID enumerant to a string
char const* to_string(MathOperationID math_op, bool pretty) {
    for (auto const& possible : MathOperationID_enumerants) {
        if (math_op == possible.enumerant) {
            if (pretty) {
                return possible.pretty;
            } else {
                return possible.text;
            }
        }
    }

    return pretty ? "Invalid" : "invalid";
}

///////////////////////////////////////////////////////////////////////////////////////////////////

static struct {
    char const* text;
    char const* pretty;
    conv::SpecialOptimizeDesc enumerant;
} SpecialOptimizeDesc_enumerants[] = {
        {"none_special_opt", "NoneSpecialOpt", conv::SpecialOptimizeDesc::NONE},
        {"conv_filter_unity", "ConvFilterUnity",
         conv::SpecialOptimizeDesc::CONV_FILTER_UNITY},
        {"deconv_double_upsampling", "DeconvDoubleUpsampling",
         conv::SpecialOptimizeDesc::DECONV_DOUBLE_UPSAMPLING},
};

/// Converts an SpecialOptimizeDesc enumerant to a string
char const* to_string(conv::SpecialOptimizeDesc special_opt, bool pretty) {
    for (auto const& possible : SpecialOptimizeDesc_enumerants) {
        if (special_opt == possible.enumerant) {
            if (pretty) {
                return possible.pretty;
            } else {
                return possible.text;
            }
        }
    }

    return pretty ? "Invalid" : "invalid";
}

///////////////////////////////////////////////////////////////////////////////////////////////////

static struct {
    char const* text;
    char const* pretty;
    conv::ImplicitGemmMode enumerant;
} ImplicitGemmMode_enumerants[] = {
        {"gemm_nt", "GemmNT", conv::ImplicitGemmMode::GEMM_NT},
        {"gemm_tn", "GemmTN", conv::ImplicitGemmMode::GEMM_TN},
};

/// Converts an ImplicitGemmMode enumerant to a string
char const* to_string(conv::ImplicitGemmMode mode, bool pretty) {
    for (auto const& possible : ImplicitGemmMode_enumerants) {
        if (mode == possible.enumerant) {
            if (pretty) {
                return possible.pretty;
            } else {
                return possible.text;
            }
        }
    }

    return pretty ? "Invalid" : "invalid";
}

///////////////////////////////////////////////////////////////////////////////////////////////////

}  // namespace library
}  // namespace cutlass

///////////////////////////////////////////////////////////////////////////////////////////////////

#endif
