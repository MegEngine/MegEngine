/***************************************************************************************************
 * Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 *modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice,
 *this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *notice, this list of conditions and the following disclaimer in the
 *documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its
 *contributors may be used to endorse or promote products derived from this
 *software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 *AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 *IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 *DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY DIRECT,
 *INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 *DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 *OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TOR (INCLUDING
 *NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 *EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/**
 * \file dnn/src/cuda/cutlass/initialize_all.cpp
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or
 * implied.
 */

#include "src/cuda/cutlass/manifest.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace cutlass {
namespace library {

/////////////////////////////////////////////////////////////////////////////////////////////////
#if ((__CUDACC_VER_MAJOR__ > 10) || \
     (__CUDACC_VER_MAJOR__ == 10 && __CUDACC_VER_MINOR__ >= 2))
#define CUTLASS_ARCH_MMA_SM75_SUPPORTED 1
#endif

#if __CUDACC_VER_MAJOR__ > 9 || (__CUDACC_VER_MAJOR__ == 9 && __CUDACC_VER_MINOR__ >= 2)

void initialize_all_gemm_simt_operations(Manifest& manifest);
void initialize_all_conv2d_simt_operations(Manifest& manifest);
void initialize_all_deconv_simt_operations(Manifest& manifest);
#if defined(CUTLASS_ARCH_MMA_SM75_SUPPORTED) && CUTLASS_ARCH_MMA_SM75_SUPPORTED
void initialize_all_gemm_tensorop884_operations(Manifest& manifest);
void initialize_all_gemm_tensorop1688_operations(Manifest& manifest);
void initialize_all_conv2d_tensorop8816_operations(Manifest& manifest);
void initialize_all_conv2d_tensorop8832_operations(Manifest& manifest);
void initialize_all_deconv_tensorop8816_operations(Manifest& manifest);
#endif

void initialize_all(Manifest& manifest) {
    initialize_all_gemm_simt_operations(manifest);
    initialize_all_conv2d_simt_operations(manifest);
    initialize_all_deconv_simt_operations(manifest);
#if defined(CUTLASS_ARCH_MMA_SM75_SUPPORTED) && CUTLASS_ARCH_MMA_SM75_SUPPORTED
    initialize_all_gemm_tensorop884_operations(manifest);
    initialize_all_gemm_tensorop1688_operations(manifest);
    initialize_all_conv2d_tensorop8816_operations(manifest);
    initialize_all_conv2d_tensorop8832_operations(manifest);
    initialize_all_deconv_tensorop8816_operations(manifest);
#endif
}

#else

void initialize_all(Manifest& manifest) {}

#endif

/////////////////////////////////////////////////////////////////////////////////////////////////

}  // namespace library
}  // namespace cutlass

/////////////////////////////////////////////////////////////////////////////////////////////////
