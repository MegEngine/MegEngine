#include "hip/hip_runtime.h"
/**
 * \file dnn/src/cuda/split/split.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 */
#include "src/cuda/split/split.cuh"

#include "megdnn/dtype.h"
#include "src/cuda/utils.cuh"

namespace megdnn {
namespace cuda {
namespace split {

template <typename T>
__global__ void forward_kernel(
        const T* src, T** dsts, size_t nr_dsts, size_t A, size_t B, size_t C,
        const size_t* Bv, const size_t* table_outer, const size_t* table_inner) {
    size_t addr = threadIdx.x + blockIdx.x * blockDim.x;
    if (addr < A * B * C) {
        size_t c = addr % C;
        size_t b = addr / C % B;
        size_t a = addr / (B * C);
        size_t i = table_outer[b];
        size_t B_dst = Bv[i];
        size_t b_dst = table_inner[b];
        size_t addr_dst = (a * B_dst + b_dst) * C + c;
        dsts[i][addr_dst] = src[addr];
    }
}

template <typename T>
void forward_proxy(
        const T* src, T** dsts, size_t nr_dsts, size_t A, size_t B, size_t C,
        const size_t* Bv, const size_t* table_outer, const size_t* table_inner,
        hipStream_t stream) {
    size_t total_nr_elem = A * B * C;
    size_t NR_BLOCKS = DIVUP(total_nr_elem, NR_THREADS);
    forward_kernel<<<NR_BLOCKS, NR_THREADS, 0, stream>>>(
            src, dsts, nr_dsts, A, B, C, Bv, table_outer, table_inner);
    after_kernel_launch();
}

#define INST(T)                                                           \
    template void forward_proxy<T>(                                       \
            const T*, T**, size_t, size_t, size_t, size_t, const size_t*, \
            const size_t*, const size_t*, hipStream_t);
#define cb(DType) INST(typename DTypeTrait<DType>::ctype)

MEGDNN_FOREACH_COMPUTING_DTYPE(cb)

#undef cb
#undef INST

}  // namespace split
}  // namespace cuda
}  // namespace megdnn

// vim: syntax=cpp.doxygen
