#include "hip/hip_runtime.h"
/**
 * \file dnn/src/cuda/eye/eye.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 */
#include "megdnn/dtype.h"
#include "src/cuda/eye/eye.cuh"
#include "src/cuda/utils.cuh"

namespace {

template <typename T>
__global__ void kernel(T* dst, uint32_t m, uint32_t n, int k) {
    int32_t i = threadIdx.x + blockIdx.x * blockDim.x;
    int32_t x = i % n;
    int32_t y = i / n;
    if (i < m * n) {
        dst[i] = (y + k == x);
    }
}

}  // anonymous namespace

namespace megdnn {
namespace cuda {
namespace eye {

template <typename T>
void exec_internal(T* dst, size_t m, size_t n, int k, hipStream_t stream) {
    kernel<T><<<DIVUP(m * n, NR_THREADS), NR_THREADS, 0, stream>>>(dst, m, n, k);
    after_kernel_launch();
}

#define INST(T)   template void exec_internal<T>(T*, size_t, size_t, int, hipStream_t);
#define cb(DType) INST(typename DTypeTrait<DType>::ctype)
MEGDNN_FOREACH_COMPUTING_DTYPE(cb)

}  // namespace eye
}  // namespace cuda
}  // namespace megdnn
// vim: syntax=cpp.doxygen foldmethod=marker foldmarker=f{{{,f}}}
