#include "hip/hip_runtime.h"
/**
 * \file dnn/src/cuda/images2neibs/kernel.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 */
#include "src/cuda/images2neibs/kernel.cuh"

#include <cstdio>
#include "megdnn/dtype.h"
#include "src/cuda/utils.cuh"

namespace megdnn {
namespace cuda {
namespace images2neibs {

#define grid_y_max 512

template <typename T>
__global__ void forward_kernel(
        const T* src, T* dst, int N, int C, int IH, int IW, int OH, int OW, int ph,
        int pw, int sh, int sw, int dh, int dw, int WH, int WW) {
    int NC = N * C;
    int WP = WH * WW;
    for (int wp = threadIdx.x; wp < WP; wp += blockDim.x) {
        int nc = blockIdx.y;
        while (nc < NC) {
            int wh = wp / WW;
            int ww = wp % WW;
            int op = threadIdx.y + blockIdx.x * blockDim.y;
            if (op < OH * OW) {
                int oh = op / OW;
                int ow = op % OW;
                int ih = -ph + sh * oh + wh * dh;
                int iw = -pw + sw * ow + ww * dw;
                int dst_pos = nc * OH * OW * WH * WW + op * WH * WW + wp;
                int src_pos = nc * IH * IW + ih * IW + iw;
                dst[dst_pos] = (ih >= 0 && ih < IH && iw >= 0 && iw < IW) ? src[src_pos]
                                                                          : 0.0f;
            }
            nc += grid_y_max;
        }
    }
}

template <typename T>
void forward(
        const T* src, T* dst, int N, int C, int IH, int IW, int OH, int OW, int ph,
        int pw, int sh, int sw, int dh, int dw, int wh, int ww, hipStream_t stream) {
    int spatial_size = OH * OW;
    int kernel_size = wh * ww;
    int tx = min(NR_THREADS, kernel_size);
    int ty = NR_THREADS / tx;
    megdnn_assert(ty > 0);
    int bx = DIVUP(spatial_size, ty);
    int by = N * C;

    forward_kernel<<<dim3(bx, std::min(grid_y_max, by)), dim3(tx, ty), 0, stream>>>(
            src, dst, N, C, IH, IW, OH, OW, ph, pw, sh, sw, dh, dw, wh, ww);
    after_kernel_launch();
}

#undef grid_y_max

template <typename T>
__global__ void backward_kernel(
        const T* diff, T* grad, int N, int C, int IH, int IW, int OH, int OW, int ph,
        int pw, int sh, int sw, int dh, int dw, int WH, int WW) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < N * C * IH * IW) {
        int nc = id / (IH * IW);
        int ih = id % (IH * IW) / IW;
        int iw = id % (IH * IW) % IW;
        grad[nc * IH * IW + ih * IW + iw] = 0.0f;
        int oh_max = min((ih + ph) / sh, OH - 1);
        int oh_min = max((ih + ph - (WH - 1) * dh + sh - 1) / sh, 0);
        int ow_max = min((iw + pw) / sw, OW - 1);
        int ow_min = max((iw + pw - (WW - 1) * dw + sw - 1) / sw, 0);
        for (int oh = oh_min; oh <= oh_max; ++oh)
            for (int ow = ow_min; ow <= ow_max; ++ow) {
                if ((ih + ph - sh * oh) % dh == 0 && (iw + pw - sw * ow) % dw == 0) {
                    int wh = ih + ph - sh * oh - (ih + ph - sh * oh) / dh * (dh - 1);
                    int ww = iw + pw - sw * ow - (iw + pw - sw * ow) / dw * (dw - 1);
                    grad[nc * IH * IW + ih * IW + iw] +=
                            diff[nc * OH * OW * WH * WW + oh * OW * WH * WW +
                                 ow * WH * WW + wh * WW + ww];
                }
            }
    }
}

template <typename T>
void backward(
        const T* diff, T* grad, int N, int C, int IH, int IW, int OH, int OW, int ph,
        int pw, int sh, int sw, int dh, int dw, int wh, int ww, hipStream_t stream) {
    int threads = NR_THREADS;
    int blocks = DIVUP(N * C * IH * IW, threads);
    backward_kernel<<<blocks, threads, 0, stream>>>(
            diff, grad, N, C, IH, IW, OH, OW, ph, pw, sh, sw, dh, dw, wh, ww);
    after_kernel_launch();
}

#define INST(T)                                                                       \
    template void forward<T>(                                                         \
            const T*, T*, int, int, int, int, int, int, int, int, int, int, int, int, \
            int, int, hipStream_t);                                                  \
    template void backward<T>(                                                        \
            const T*, T*, int, int, int, int, int, int, int, int, int, int, int, int, \
            int, int, hipStream_t);
#define cb(DType) INST(DTypeTrait<DType>::ctype)

MEGDNN_FOREACH_COMPUTING_DTYPE(cb)

}  // namespace images2neibs
}  // namespace cuda
}  // namespace megdnn

// vim: syntax=cpp.doxygen
