#include "hip/hip_runtime.h"
/**
 * By downloading, copying, installing or using the software you agree to this license.
 * If you do not agree to this license, do not download, install,
 * copy or use the software.
 *
 *
 *                           License Agreement
 *                For Open Source Computer Vision Library
 *                        (3-clause BSD License)
 *
 * Copyright (C) 2000-2020, Intel Corporation, all rights reserved.
 * Copyright (C) 2009-2011, Willow Garage Inc., all rights reserved.
 * Copyright (C) 2009-2016, NVIDIA Corporation, all rights reserved.
 * Copyright (C) 2010-2013, Advanced Micro Devices, Inc., all rights reserved.
 * Copyright (C) 2015-2016, OpenCV Foundation, all rights reserved.
 * Copyright (C) 2015-2016, Itseez Inc., all rights reserved.
 * Copyright (C) 2019-2020, Xperience AI, all rights reserved.
 * Third party copyrights are property of their respective owners.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 *   * Redistributions of source code must retain the above copyright notice,
 *     this list of conditions and the following disclaimer.
 *
 *   * Redistributions in binary form must reproduce the above copyright notice,
 *     this list of conditions and the following disclaimer in the documentation
 *     and/or other materials provided with the distribution.
 *
 *   * Neither the names of the copyright holders nor the names of the contributors
 *     may be used to endorse or promote products derived from this software
 *     without specific prior written permission.
 *
 * This software is provided by the copyright holders and contributors "as is" and
 * any express or implied warranties, including, but not limited to, the implied
 * warranties of merchantability and fitness for a particular purpose are disclaimed.
 * In no event shall copyright holders or contributors be liable for any direct,
 * indirect, incidental, special, exemplary, or consequential damages
 * (including, but not limited to, procurement of substitute goods or services;
 * loss of use, data, or profits; or business interruption) however caused
 * and on any theory of liability, whether in contract, strict liability,
 * or tort (including negligence or otherwise) arising in any way out of
 * the use of this software, even if advised of the possibility of such damage.
 *
 * ---------------------------------------------------------------------------
 * \file dnn/src/cuda/resize/resize_cv.cu
 *
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *
 * This file has been modified by Megvii ("Megvii Modifications").
 * All Megvii Modifications are Copyright (C) 2014-2021 Megvii Inc. All rights reserved.
 *
 * ---------------------------------------------------------------------------
 */
#include "src/common/resize.cuh"
#include "src/cuda/cv/kernel_common.cuh"
#include "src/cuda/resize/resize_cv.cuh"
#include "src/cuda/utils.cuh"

using namespace megdnn;
using namespace cuda;
using namespace megcv;
using megdnn::resize::interpolate_cubic;

namespace {

#define SCALE           11
#define at(A, r, c, ch) A[(r)*A##_step + (c)*CH + (ch)]
#define ONE             (1 << SCALE)

#define ELEMENTS_PER_THREADS 8
#define THREADS_X            32
#define THREADS_Y            16

__global__ void precompute_lanczos4_coef_f32(float* dst, float scale, size_t size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= size)
        return;

    float fr = (tid + 0.5) * scale - 0.5;
    int* sr = (int*)(dst + size * 8);
    sr[tid] = (int)(floorf(fr));

    fr -= sr[tid];
    float coef[8];
    interpolate_lanczos4_coefs(fr, coef);
#pragma unroll
    for (int j = 0, index = 0; j < 8; j++, index += size) {
        dst[tid + index] = coef[j];
    }
}

__global__ void precompute_lanczos4_coef_u8(short* dst, float scale, size_t size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= size)
        return;

    float fr = (tid + 0.5) * scale - 0.5;
    int* sr = (int*)(dst + size * 8);
    sr[tid] = (int)(floorf(fr));

    fr -= sr[tid];
    float coef[8];
    interpolate_lanczos4_coefs(fr, coef);
#pragma unroll
    for (int j = 0, index = 0; j < 8; j++, index += size) {
        dst[tid + index] = (short)(coef[j] * ONE);
    }
}

__global__ void precompute_cubic_coef_f32(float* dst, float scale, size_t size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= size)
        return;

    float fr = (tid + 0.5) * scale - 0.5;
    int* sr = (int*)(dst + size * 4);
    sr[tid] = (int)(floorf(fr));

    fr -= sr[tid];
    float coef[4];
    interpolate_cubic(fr, coef);
#pragma unroll
    for (int j = 0, index = 0; j < 4; j++, index += size) {
        dst[tid + index] = coef[j];
    }
}

__global__ void precompute_cubic_coef_u8(short* dst, float scale, size_t size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= size)
        return;

    float fr = (tid + 0.5) * scale - 0.5;
    int* sr = (int*)(dst + size * 4);
    sr[tid] = (int)(floorf(fr));

    fr -= sr[tid];
    float coef[4];
    interpolate_cubic(fr, coef);
#pragma unroll
    for (int j = 0, index = 0; j < 4; j++, index += size) {
        dst[tid + index] = (short)(coef[j] * ONE);
    }
}

template <typename T, size_t CH>
__global__ void resize_nearest_vector_kernel(
        const T* src, T* dst, const size_t dst_rows, const size_t dst_cols,
        const size_t src_step, const size_t dst_step, const float row_scale,
        const float col_scale) {
    size_t dc = blockIdx.x * blockDim.x + threadIdx.x;
    size_t dr = blockIdx.y * blockDim.y * ELEMENTS_PER_THREADS + threadIdx.y;

    if (dr < dst_rows && dc < dst_cols) {
        int dst_address_incress = dr * dst_step + dc * CH;
        size_t sc = dc * col_scale;
        src += sc * CH;

        for (int i = 0; i < ELEMENTS_PER_THREADS; i++) {
            if (dr >= dst_rows)
                return;

            size_t sr = dr * row_scale;
            int src_address_incress = sr * src_step;
            for (size_t ch = 0; ch < CH; ch++)
                dst[dst_address_incress + ch] = src[src_address_incress + ch];

            dr += blockDim.y;
            dst_address_incress += blockDim.y * dst_step;
        }
    }
}

template <typename T, size_t CH>
__global__ void resize_nearest_kernel(
        const T* __restrict__ src, T* dst, const size_t dst_rows, const size_t dst_cols,
        const size_t src_step, const size_t dst_step, const float row_scale,
        const float col_scale) {
    size_t dc = blockIdx.x * blockDim.x + threadIdx.x;
    size_t dr = blockIdx.y * blockDim.y + threadIdx.y;
    if (dr < dst_rows && dc < dst_cols) {
        size_t sr = dr * row_scale;
        size_t sc = dc * col_scale;
        src += sr * src_step + sc * CH;
        dst += dr * dst_step + dc * CH;
#pragma unroll
        for (size_t ch = 0; ch < CH; ++ch)
            dst[ch] = src[ch];
    }
}

template <typename T, size_t CH>
void resize_nearest_proxy(
        const T* src, T* dst, const size_t src_rows, const size_t src_cols,
        const size_t dst_rows, const size_t dst_cols, const size_t src_step,
        const size_t dst_step, void* workspace, hipStream_t stream) {
    MEGDNN_MARK_USED_VAR(workspace);
    float row_scale = (float)src_rows / dst_rows;
    float col_scale = (float)src_cols / dst_cols;

    if (CH == 3 && sizeof(T) == 4 && (dst_cols * dst_rows <= src_cols * src_rows)) {
        dim3 THREADS(32, 8, 1);
        dim3 BLOCKS(DIVUP(dst_cols, THREADS.x), DIVUP(dst_rows, THREADS.y));

        hipDeviceSetCacheConfig(hipFuncCachePreferL1);

        resize_nearest_kernel<T, CH><<<BLOCKS, THREADS, 0, stream>>>(
                src, dst, dst_rows, dst_cols, src_step, dst_step, row_scale, col_scale);

    } else {
        dim3 THREADS(32, 8, 1);
        dim3 BLOCKS(
                DIVUP(dst_cols, THREADS.x),
                DIVUP(dst_rows, THREADS.y * ELEMENTS_PER_THREADS));

        if (CH == 3 && sizeof(T) == 1)
            hipDeviceSetCacheConfig(hipFuncCachePreferL1);

        resize_nearest_vector_kernel<T, CH><<<BLOCKS, THREADS, 0, stream>>>(
                src, dst, dst_rows, dst_cols, src_step, dst_step, row_scale, col_scale);
    }
}

template <typename T, size_t CH>
__global__ void resize_linear_Restric_kernel(
        const T* __restrict__ src, T* dst, const size_t src_rows, const size_t src_cols,
        const size_t dst_rows, const size_t dst_cols, const size_t src_step,
        const size_t dst_step, const float row_scale, const float col_scale,
        const float inverse_row_scale, const float inverse_col_scale) {
    size_t dc = blockIdx.x * blockDim.x + threadIdx.x;
    size_t dr = blockIdx.y * blockDim.y + threadIdx.y;

    if (dr < dst_rows && dc < dst_cols) {
        float fc = (dc + 0.5f) * inverse_col_scale - 0.5f;
        float fr = (dr + 0.5f) * inverse_row_scale - 0.5f;
        int sc = __float2int_rd(fc);
        int sr = __float2int_rd(fr);

        fc -= sc;
        fr -= sr;

        if (sc < 0) {
            sc = 0;
            fc = 0;
        }
        if (sr < 0) {
            sr = 0;
            fr = 0;
        }

        if (sc + 1 >= src_cols) {
            sc = src_cols - 2;
            fc = 1;
        }

        if (sr + 1 >= src_rows) {
            sr = src_rows - 2;
            fr = 1;
        }

        int src_address = sr * src_step + sc * CH;

        // if the type is uchar, use sr and sc to donate fx * (1 << SCALE)
        float dst_data[CH] = {0};
#pragma unroll
        for (int ch = 0; ch < CH; ch++) {
            float pcrsc00 = src[src_address + ch];
            float pcrsc01 = src[src_address + CH + ch];
            float pcrsc10 = src[src_address + src_step + ch];
            float pcrsc11 = src[src_address + src_step + CH + ch];
            dst_data[ch] = fr * (pcrsc11 * fc + pcrsc10 * (1 - fc)) +
                           (1 - fr) * (pcrsc01 * fc + pcrsc00 * (1 - fc));
        }
        int dst_address = dr * dst_step + dc * CH;
#pragma unroll
        for (int ch = 0; ch < CH; ch++)
            dst[dst_address++] = (T)(dst_data[ch]);
    }
}

template <typename T, size_t CH>
__global__ void resize_linear_vector_kernel(
        const T* src, T* dst, const size_t src_rows, const size_t src_cols,
        const size_t dst_rows, const size_t dst_cols, const size_t src_step,
        const size_t dst_step, const float row_scale, const float col_scale,
        const float inverse_row_scale, const float inverse_col_scale) {
    size_t dc = blockIdx.x * blockDim.x + threadIdx.x;
    size_t dr = blockIdx.y * blockDim.y * ELEMENTS_PER_THREADS + threadIdx.y;

    if (dr < dst_rows && dc < dst_cols) {
        float fc = (dc + 0.5f) * inverse_col_scale - 0.5f;
        int sc = __float2int_rd(fc);
        fc -= sc;
        if (sc < 0) {
            sc = 0;
            fc = 0;
        }

        if (sc + 1 >= src_cols) {
            sc = src_cols - 2;
            fc = 1;
        }
        for (int i = 0; i < ELEMENTS_PER_THREADS; i++) {
            if (dr >= dst_rows)
                return;

            float fr = (dr + 0.5f) * inverse_row_scale - 0.5f;
            int sr = __float2int_rd(fr);
            fr -= sr;

            if (sr < 0) {
                sr = 0;
                fr = 0;
            }
            if (sr + 1 >= src_rows) {
                sr = src_rows - 2;
                fr = 1;
            }
            int src_address = sr * src_step + sc * CH;
            float dst_data[CH] = {0};
#pragma unroll
            for (int ch = 0; ch < CH; ch++) {
                float pcrsc00 = src[src_address + ch];
                float pcrsc01 = src[src_address + CH + ch];
                float pcrsc10 = src[src_address + src_step + ch];
                float pcrsc11 = src[src_address + src_step + CH + ch];
                dst_data[ch] = fr * (pcrsc11 * fc + pcrsc10 * (1 - fc)) +
                               (1 - fr) * (pcrsc01 * fc + pcrsc00 * (1 - fc));
            }

            int dst_address = dr * dst_step + dc * CH;
#pragma unroll
            for (int ch = 0; ch < CH; ch++)
                dst[dst_address++] = (T)(dst_data[ch]);

            dr += blockDim.y;
        }
    }
}

template <typename T, size_t CH>
void resize_area_proxy(
        const T*, T*, size_t, size_t, size_t, size_t, size_t, size_t, void*,
        hipStream_t);

template <typename T, size_t CH>
void resize_linear_proxy(
        const T* src, T* dst, const size_t src_rows, const size_t src_cols,
        const size_t dst_rows, const size_t dst_cols, const size_t src_step,
        const size_t dst_step, void* workspace, hipStream_t stream) {
    if (src_rows == dst_rows * 2 && src_cols == dst_cols * 2) {
        resize_area_proxy<T, CH>(
                src, dst, src_rows, src_cols, dst_rows, dst_cols, src_step, dst_step,
                workspace, stream);
        return;
    }

    dim3 THREADS(32, 8, 1);

    float row_scale = (float)dst_rows / src_rows;
    float col_scale = (float)dst_cols / src_cols;

    if (CH == 3 && (dst_rows < src_rows && dst_cols < src_cols)) {
        dim3 BLOCKS(DIVUP(dst_cols, THREADS.x), DIVUP(dst_rows, THREADS.y));

        hipDeviceSetCacheConfig(hipFuncCachePreferL1);

        resize_linear_Restric_kernel<T, CH><<<BLOCKS, THREADS, 0, stream>>>(
                src, dst, src_rows, src_cols, dst_rows, dst_cols, src_step, dst_step,
                row_scale, col_scale, 1 / row_scale, 1 / col_scale);

    } else {
        dim3 BLOCKS(
                DIVUP(dst_cols, THREADS.x),
                DIVUP(dst_rows, THREADS.y * ELEMENTS_PER_THREADS));

        hipDeviceSetCacheConfig(hipFuncCachePreferL1);

        resize_linear_vector_kernel<T, CH><<<BLOCKS, THREADS, 0, stream>>>(
                src, dst, src_rows, src_cols, dst_rows, dst_cols, src_step, dst_step,
                row_scale, col_scale, 1 / row_scale, 1 / col_scale);
    }
}

template <size_t CH>
__global__ void resize_cubic_32f_kernel_vector(
        const float* __restrict__ src, float* dst, const size_t src_rows,
        const size_t src_cols, const size_t dst_rows, const size_t dst_cols,
        const size_t src_step, const size_t dst_step, const float row_scale,
        const float col_scale) {
    size_t dc = blockIdx.x * blockDim.x + threadIdx.x;
    size_t dr = blockIdx.y * blockDim.y * ELEMENTS_PER_THREADS + threadIdx.y;
    if (dr < dst_rows && dc < dst_cols) {
        float fc = ((float)dc + 0.5) * col_scale - 0.5;
        int sc = floor(fc);
        fc -= sc;
        float coef_col[4];
        interpolate_cubic(fc, coef_col);

        for (int i = 0; i < ELEMENTS_PER_THREADS; i++) {
            if (dr >= dst_rows)
                return;
            float fr = ((float)dr + 0.5) * row_scale - 0.5;
            int sr = floor(fr);
            fr -= sr;
            float coef_row[4];
            interpolate_cubic(fr, coef_row);
            float dst_data[CH] = {0};
#pragma unroll
            for (int offset_r = 0; offset_r < 4; ++offset_r) {
                int tr_step =
                        saturate(sr + offset_r - 1, 0, (int)src_rows - 1) * src_step;
#pragma unroll
                for (int offset_c = 0; offset_c < 4; ++offset_c) {
                    int tc_step =
                            saturate(sc + offset_c - 1, 0, (int)src_cols - 1) * CH;
                    int src_address = tr_step + tc_step;
#pragma unroll
                    for (size_t ch = 0; ch < CH; ++ch) {
                        dst_data[ch] += coef_row[offset_r] * coef_col[offset_c] *
                                        src[src_address++];
                    }
                }
            }
            int dst_address = dr * dst_step + dc * CH;
#pragma unroll
            for (int i = 0; i < CH; i++)
                dst[dst_address++] = dst_data[i];
            dr += blockDim.y;
        }
    }
}

template <size_t CH>
__global__ void resize_cubic_8u_kernel_vector(
        const uchar* __restrict__ src, uchar* dst, const size_t src_rows,
        const size_t src_cols, const size_t dst_rows, const size_t dst_cols,
        const size_t src_step, const size_t dst_step, const float row_scale,
        const float col_scale) {
    size_t dc = blockIdx.x * blockDim.x + threadIdx.x;
    size_t dr = blockIdx.y * blockDim.y * ELEMENTS_PER_THREADS + threadIdx.y;
    if (dr < dst_rows && dc < dst_cols) {
        float fc = ((float)dc + 0.5) * col_scale - 0.5;
        int sc = __float2int_rd(fc);
        fc -= sc;
        short icoef_col[4] = {0};

        float coef_col[4];
        interpolate_cubic(fc, coef_col);
#pragma unroll
        for (int i = 0; i < 4; i++) {
            icoef_col[i] = (short)(coef_col[i] * ONE);
        }

        for (int i = 0; i < ELEMENTS_PER_THREADS; i++) {
            if (dr >= dst_rows)
                return;
            float fr = ((float)dr + 0.5) * row_scale - 0.5;
            int sr = __float2int_rd(fr);
            fr -= sr;
            short icoef_row[4];
            float coef_row[4];
            interpolate_cubic(fr, coef_row);
#pragma unroll
            for (int i = 0; i < 4; i++) {
                icoef_row[i] = (short)(coef_row[i] * ONE);
            }

            int dst_data[CH] = {0};
#pragma unroll
            for (int offset_r = 0; offset_r < 4; ++offset_r) {
                int tr_step =
                        saturate(sr + offset_r - 1, 0, (int)src_rows - 1) * src_step;
#pragma unroll
                for (int offset_c = 0; offset_c < 4; ++offset_c) {
                    int tc_step =
                            saturate(sc + offset_c - 1, 0, (int)src_cols - 1) * CH;
                    int src_address = tr_step + tc_step;
#pragma unroll
                    for (size_t ch = 0; ch < CH; ++ch) {
                        dst_data[ch] += icoef_row[offset_r] * icoef_col[offset_c] *
                                        src[src_address++];
                    }
                }
            }
            int dst_address = dr * dst_step + dc * CH;
#pragma unroll
            for (int i = 0; i < CH; i++)
                dst[dst_address++] = saturate(dst_data[i] >> (SCALE + SCALE), 0, 255);
            dr += blockDim.y;
        }
    }
}

template <size_t CH>
__global__ void resize_cubic_32f_kernel_cacheToGlobal(
        const float* src, float* dst, const size_t src_rows, const size_t src_cols,
        const size_t dst_rows, const size_t dst_cols, const size_t src_step,
        const size_t dst_step, const float* gl_coef_row, const float* gl_coef_col,
        const int* gl_sr, const int* gl_sc) {
    size_t dc = blockIdx.x * blockDim.x + threadIdx.x;
    size_t dr = blockIdx.y * blockDim.y * ELEMENTS_PER_THREADS + threadIdx.y;

    if (dr < dst_rows && dc < dst_cols) {
        int sc = gl_sc[dc];
        float coef_col[4];
#pragma unroll
        for (int i = 0, index = dc; i < 4; i++, index += dst_cols)
            coef_col[i] = gl_coef_col[index];

        for (int i = 0; i < ELEMENTS_PER_THREADS; i++) {
            if (dr >= dst_rows)
                return;
            int sr = gl_sr[dr];
            float coef_row[4];
#pragma unroll
            for (int i = 0, index = dr; i < 4; i++, index += dst_rows)
                coef_row[i] = gl_coef_row[index];

            float dst_data[CH] = {0};
#pragma unroll
            for (int offset_r = 0; offset_r < 4; ++offset_r) {
                int tr_step =
                        saturate(sr + offset_r - 1, 0, (int)src_rows - 1) * src_step;
#pragma unroll
                for (int offset_c = 0; offset_c < 4; ++offset_c) {
                    int tc_step =
                            saturate(sc + offset_c - 1, 0, (int)src_cols - 1) * CH;
                    int src_address = tr_step + tc_step;
#pragma unroll
                    for (size_t ch = 0; ch < CH; ++ch) {
                        dst_data[ch] += coef_row[offset_r] * coef_col[offset_c] *
                                        src[src_address++];
                    }
                }
            }
            int dst_address = dr * dst_step + dc * CH;
#pragma unroll
            for (int i = 0; i < CH; i++)
                dst[dst_address++] = dst_data[i];

            dr += blockDim.y;
        }
    }
}

template <size_t CH>
__global__ void resize_cubic_8u_kernel_cacheToGlobal(
        const uchar* src, uchar* dst, const size_t src_rows, const size_t src_cols,
        const size_t dst_rows, const size_t dst_cols, const size_t src_step,
        const size_t dst_step, const short* gl_icoef_row, const short* gl_icoef_col,
        const int* gl_sr, const int* gl_sc) {
    size_t dc = blockIdx.x * blockDim.x + threadIdx.x;
    size_t dr = blockIdx.y * blockDim.y * ELEMENTS_PER_THREADS + threadIdx.y;

    if (dr < dst_rows && dc < dst_cols) {
        int sc = gl_sc[dc];
        short icoef_col[4];
#pragma unroll
        for (int i = 0, index = dc; i < 4; i++, index += dst_cols)
            icoef_col[i] = gl_icoef_col[index];

        for (int i = 0; i < ELEMENTS_PER_THREADS; i++) {
            if (dr >= dst_rows)
                return;
            int sr = gl_sr[dr];
            short icoef_row[4];
#pragma unroll
            for (int i = 0, index = dr; i < 4; i++, index += dst_rows)
                icoef_row[i] = gl_icoef_row[index];

            int dst_data[CH] = {0};
#pragma unroll
            for (int offset_r = 0; offset_r < 4; ++offset_r) {
                int tr_step =
                        saturate(sr + offset_r - 1, 0, (int)src_rows - 1) * src_step;
#pragma unroll
                for (int offset_c = 0; offset_c < 4; ++offset_c) {
                    int tc_step =
                            saturate(sc + offset_c - 1, 0, (int)src_cols - 1) * CH;
                    int src_address = tr_step + tc_step;
#pragma unroll
                    for (size_t ch = 0; ch < CH; ++ch) {
                        dst_data[ch] += icoef_row[offset_r] * icoef_col[offset_c] *
                                        src[src_address++];
                    }
                }
            }
            int dst_address = dr * dst_step + dc * CH;
#pragma unroll
            for (int i = 0; i < CH; i++)
                dst[dst_address++] = saturate(dst_data[i] >> (SCALE + SCALE), 0, 255);

            dr += blockDim.y;
        }
    }
}

template <typename T, size_t CH>
void resize_cubic_proxy(
        const T* src, T* dst, const size_t src_rows, const size_t src_cols,
        const size_t dst_rows, const size_t dst_cols, const size_t src_step,
        const size_t dst_step, void* workspace, hipStream_t stream) {
    dim3 THREADS(32, 8, 1);
    float row_scale = (float)src_rows / dst_rows;
    float col_scale = (float)src_cols / dst_cols;

    size_t dst_area_size = dst_rows * dst_cols;
    size_t src_area_size = src_rows * src_cols;

    bool enlarge = dst_area_size > src_area_size;
    bool shrink = dst_area_size <= src_area_size;
    bool U8 = sizeof(T) == sizeof(uchar);
    bool F32_1 = sizeof(T) == sizeof(float) && CH == 1;
    bool F32_3 = sizeof(T) == sizeof(float) && CH == 3;

    bool use_vector = (enlarge && (dst_area_size <= 500 * 500)) ||
                      (shrink && (F32_3 || (U8 && dst_area_size <= 500 * 500) ||
                                  (F32_1 && dst_area_size <= 1000 * 1000)));

    if (use_vector) {
        dim3 BLOCKS(
                DIVUP(dst_cols, THREADS.x),
                DIVUP(dst_rows, THREADS.y * ELEMENTS_PER_THREADS));

        if (sizeof(T) == sizeof(float)) {
            resize_cubic_32f_kernel_vector<CH><<<BLOCKS, THREADS, 0, stream>>>(
                    (const float*)src, (float*)dst, src_rows, src_cols, dst_rows,
                    dst_cols, src_step, dst_step, row_scale, col_scale);
        } else {
            resize_cubic_8u_kernel_vector<CH><<<BLOCKS, THREADS, 0, stream>>>(
                    (const uchar*)src, (uchar*)dst, src_rows, src_cols, dst_rows,
                    dst_cols, src_step, dst_step, row_scale, col_scale);
        }

    } else {
        dim3 BLOCKS(
                DIVUP(dst_cols, THREADS.x),
                DIVUP(dst_rows, THREADS.y * ELEMENTS_PER_THREADS));

        hipDeviceSetCacheConfig(hipFuncCachePreferL1);

        if (sizeof(T) == sizeof(float)) {
            float* dev_coef_row = static_cast<float*>(workspace);
            int* dev_sr = reinterpret_cast<int*>(dev_coef_row + dst_rows * 4);
            float* dev_coef_col = reinterpret_cast<float*>(dev_sr + dst_rows);
            int* dev_sc = reinterpret_cast<int*>(dev_coef_col + dst_cols * 4);

            precompute_cubic_coef_f32<<<DIVUP(dst_rows, 128), 128, 0, stream>>>(
                    dev_coef_row, row_scale, dst_rows);
            precompute_cubic_coef_f32<<<DIVUP(dst_cols, 128), 128, 0, stream>>>(
                    dev_coef_col, col_scale, dst_cols);

            resize_cubic_32f_kernel_cacheToGlobal<CH><<<BLOCKS, THREADS, 0, stream>>>(
                    (const float*)src, (float*)dst, src_rows, src_cols, dst_rows,
                    dst_cols, src_step, dst_step, dev_coef_row, dev_coef_col, dev_sr,
                    dev_sc);

        } else {
            short* dev_coef_row = static_cast<short*>(workspace);
            int* dev_sr = reinterpret_cast<int*>(dev_coef_row + dst_rows * 4);
            short* dev_coef_col = reinterpret_cast<short*>(dev_sr + dst_rows);
            int* dev_sc = reinterpret_cast<int*>(dev_coef_col + dst_cols * 4);

            precompute_cubic_coef_u8<<<DIVUP(dst_rows, 128), 128, 0, stream>>>(
                    dev_coef_row, row_scale, dst_rows);
            precompute_cubic_coef_u8<<<DIVUP(dst_cols, 128), 128, 0, stream>>>(
                    dev_coef_col, col_scale, dst_cols);

            resize_cubic_8u_kernel_cacheToGlobal<CH><<<BLOCKS, THREADS, 0, stream>>>(
                    (const uchar*)src, (uchar*)dst, src_rows, src_cols, dst_rows,
                    dst_cols, src_step, dst_step, dev_coef_row, dev_coef_col, dev_sr,
                    dev_sc);
        }
    }
}

template <size_t CH>
__global__ void resize_lanczos4_32f_kernel_vector(
        const float* __restrict__ src, float* dst, const size_t src_rows,
        const size_t src_cols, const size_t dst_rows, const size_t dst_cols,
        const size_t src_step, const size_t dst_step, const float row_scale,
        const float col_scale) {
    size_t dc = blockIdx.x * blockDim.x + threadIdx.x;
    size_t dr = blockIdx.y * blockDim.y * ELEMENTS_PER_THREADS + threadIdx.y;
    if (dr < dst_rows && dc < dst_cols) {
        float fc = ((float)dc + 0.5) * col_scale - 0.5;
        int sc = floor(fc);
        fc -= sc;
        float coef_col[8];
        interpolate_lanczos4_coefs(fc, coef_col);

        for (int i = 0; i < ELEMENTS_PER_THREADS; i++) {
            if (dr >= dst_rows)
                return;
            float fr = ((float)dr + 0.5) * row_scale - 0.5;
            int sr = floor(fr);
            fr -= sr;
            float coef_row[8];
            interpolate_lanczos4_coefs(fr, coef_row);
            float dst_data[CH] = {0};
#pragma unroll
            for (int offset_r = 0; offset_r < 8; ++offset_r) {
                int tr_step =
                        saturate(sr + offset_r - 3, 0, (int)src_rows - 1) * src_step;
#pragma unroll
                for (int offset_c = 0; offset_c < 8; ++offset_c) {
                    int tc_step =
                            saturate(sc + offset_c - 3, 0, (int)src_cols - 1) * CH;
                    int src_address = tr_step + tc_step;
#pragma unroll
                    for (size_t ch = 0; ch < CH; ++ch) {
                        dst_data[ch] += coef_row[offset_r] * coef_col[offset_c] *
                                        src[src_address++];
                    }
                }
            }
            int dst_address = dr * dst_step + dc * CH;
#pragma unroll
            for (int i = 0; i < CH; i++)
                dst[dst_address++] = dst_data[i];
            dr += blockDim.y;
        }
    }
}

template <size_t CH>
__global__ void resize_lanczos4_8u_kernel_vector(
        const uchar* __restrict__ src, uchar* dst, const size_t src_rows,
        const size_t src_cols, const size_t dst_rows, const size_t dst_cols,
        const size_t src_step, const size_t dst_step, const float row_scale,
        const float col_scale) {
    size_t dc = blockIdx.x * blockDim.x + threadIdx.x;
    size_t dr = blockIdx.y * blockDim.y * ELEMENTS_PER_THREADS + threadIdx.y;
    if (dr < dst_rows && dc < dst_cols) {
        float fc = ((float)dc + 0.5) * col_scale - 0.5;
        int sc = floor(fc);
        fc -= sc;
        short icoef_col[8] = {0};
        const float s45 = 0.70710678118654752440084436210485;
        const float cs[][2] = {{1, 0},  {-s45, -s45}, {0, 1},  {s45, -s45},
                               {-1, 0}, {s45, s45},   {0, -1}, {-s45, s45}};
        const float MEGCV_PI = 3.1415926536;

        {
            if (fc < FLT_EPSILON)
                icoef_col[3] = ONE;
            else {
                float coef_col[8];
                float sum = 0;
                float y0 = -(fc + 3) * MEGCV_PI * 0.25, s0 = sin(y0), c0 = cos(y0);
#pragma unroll
                for (int i = 0; i < 8; i++) {
                    float y = -(fc + 3 - i) * MEGCV_PI * 0.25;
                    coef_col[i] = (float)((cs[i][0] * s0 + cs[i][1] * c0) / (y * y));
                    sum += coef_col[i];
                }

                sum = 1.f / sum;
#pragma unroll
                for (int i = 0; i < 8; i++) {
                    coef_col[i] *= sum;
                    icoef_col[i] = (short)(coef_col[i] * ONE);
                }
            }
        }

        for (int i = 0; i < ELEMENTS_PER_THREADS; i++) {
            if (dr >= dst_rows)
                return;
            float fr = ((float)dr + 0.5) * row_scale - 0.5;
            int sr = floor(fr);
            fr -= sr;
            short icoef_row[8] = {0};
            {
                if (fr < FLT_EPSILON)
                    icoef_row[3] = ONE;
                else {
                    float coef_row[8];
                    float sum = 0;
                    float y0 = -(fr + 3) * MEGCV_PI * 0.25, s0 = sin(y0), c0 = cos(y0);
#pragma unroll
                    for (int i = 0; i < 8; i++) {
                        float y = -(fr + 3 - i) * MEGCV_PI * 0.25;
                        coef_row[i] =
                                (float)((cs[i][0] * s0 + cs[i][1] * c0) / (y * y));
                        sum += coef_row[i];
                    }

                    sum = 1.f / sum;
#pragma unroll
                    for (int i = 0; i < 8; i++) {
                        coef_row[i] *= sum;
                        icoef_row[i] = (short)(coef_row[i] * ONE);
                    }
                }
            }

            int dst_data[CH] = {0};
#pragma unroll
            for (int offset_r = 0; offset_r < 8; ++offset_r) {
                int tr_step =
                        saturate(sr + offset_r - 3, 0, (int)src_rows - 1) * src_step;
#pragma unroll
                for (int offset_c = 0; offset_c < 8; ++offset_c) {
                    int tc_step =
                            saturate(sc + offset_c - 3, 0, (int)src_cols - 1) * CH;
                    int src_address = tr_step + tc_step;
#pragma unroll
                    for (size_t ch = 0; ch < CH; ++ch) {
                        dst_data[ch] += icoef_row[offset_r] * icoef_col[offset_c] *
                                        src[src_address++];
                    }
                }
            }

            int dst_address = dr * dst_step + dc * CH;
            for (int ch = 0; ch < CH; ch++)
                dst[dst_address++] = saturate(dst_data[ch] >> (SCALE + SCALE), 0, 255);
            dr += blockDim.y;
        }
    }
}

template <size_t CH>
__global__ void resize_lanczos4_32f_kernel_cacheToGlobal(
        const float* src, float* dst, const size_t src_rows, const size_t src_cols,
        const size_t dst_rows, const size_t dst_cols, const size_t src_step,
        const size_t dst_step, const float* gl_coef_row, const float* gl_coef_col,
        const int* gl_sr, const int* gl_sc) {
    size_t dc = blockIdx.x * blockDim.x + threadIdx.x;
    size_t dr = blockIdx.y * blockDim.y * ELEMENTS_PER_THREADS + threadIdx.y;

    if (dr < dst_rows && dc < dst_cols) {
        int sc = gl_sc[dc];
        float coef_col[8];
#pragma unroll
        for (int i = 0, index = dc; i < 8; i++, index += dst_cols)
            coef_col[i] = gl_coef_col[index];

        for (int i = 0; i < ELEMENTS_PER_THREADS; i++) {
            if (dr >= dst_rows)
                return;
            int sr = gl_sr[dr];
            float coef_row[8];
#pragma unroll
            for (int i = 0, index = dr; i < 8; i++, index += dst_rows)
                coef_row[i] = gl_coef_row[index];

            float dst_data[CH] = {0};
#pragma unroll
            for (int offset_r = 0; offset_r < 8; ++offset_r) {
                int tr_step =
                        saturate(sr + offset_r - 3, 0, (int)src_rows - 1) * src_step;
#pragma unroll
                for (int offset_c = 0; offset_c < 8; ++offset_c) {
                    int tc_step =
                            saturate(sc + offset_c - 3, 0, (int)src_cols - 1) * CH;
                    int src_address = tr_step + tc_step;
#pragma unroll
                    for (size_t ch = 0; ch < CH; ++ch) {
                        dst_data[ch] += coef_row[offset_r] * coef_col[offset_c] *
                                        src[src_address++];
                    }
                }
            }
            int dst_address = dr * dst_step + dc * CH;
#pragma unroll
            for (int i = 0; i < CH; i++)
                dst[dst_address++] = dst_data[i];

            dr += blockDim.y;
        }
    }
}

template <size_t CH>
__global__ void resize_lanczos4_8u_kernel_cacheToGlobal(
        const uchar* src, uchar* dst, const size_t src_rows, const size_t src_cols,
        const size_t dst_rows, const size_t dst_cols, const size_t src_step,
        const size_t dst_step, const short* gl_icoef_row, const short* gl_icoef_col,
        const int* gl_sr, const int* gl_sc) {
    size_t dc = blockIdx.x * blockDim.x + threadIdx.x;
    size_t dr = blockIdx.y * blockDim.y * ELEMENTS_PER_THREADS + threadIdx.y;

    if (dr < dst_rows && dc < dst_cols) {
        int sc = gl_sc[dc];
        short icoef_col[8];
#pragma unroll
        for (int i = 0, index = dc; i < 8; i++, index += dst_cols)
            icoef_col[i] = gl_icoef_col[index];

        for (int i = 0; i < ELEMENTS_PER_THREADS; i++) {
            if (dr >= dst_rows)
                return;
            int sr = gl_sr[dr];
            short icoef_row[8];
#pragma unroll
            for (int i = 0, index = dr; i < 8; i++, index += dst_rows)
                icoef_row[i] = gl_icoef_row[index];

            int dst_data[CH] = {0};
#pragma unroll
            for (int offset_r = 0; offset_r < 8; ++offset_r) {
                int tr_step =
                        saturate(sr + offset_r - 3, 0, (int)src_rows - 1) * src_step;
#pragma unroll
                for (int offset_c = 0; offset_c < 8; ++offset_c) {
                    int tc_step =
                            saturate(sc + offset_c - 3, 0, (int)src_cols - 1) * CH;
                    int src_address = tr_step + tc_step;
#pragma unroll
                    for (size_t ch = 0; ch < CH; ++ch) {
                        dst_data[ch] += icoef_row[offset_r] * icoef_col[offset_c] *
                                        src[src_address++];
                    }
                }
            }
            int dst_address = dr * dst_step + dc * CH;
#pragma unroll
            for (int i = 0; i < CH; i++)
                dst[dst_address++] = saturate(dst_data[i] >> (SCALE + SCALE), 0, 255);

            dr += blockDim.y;
        }
    }
}

template <typename T, size_t CH>
void resize_lanczos4_proxy(
        const T* src, T* dst, const size_t src_rows, const size_t src_cols,
        const size_t dst_rows, const size_t dst_cols, const size_t src_step,
        const size_t dst_step, void* workspace, hipStream_t stream) {
    dim3 THREADS(16, 16, 1);

    float row_scale = (float)src_rows / dst_rows;
    float col_scale = (float)src_cols / dst_cols;

    size_t dst_area_size = dst_rows * dst_cols;
    size_t src_area_size = src_rows * src_cols;

    bool enlarge = dst_area_size > src_area_size;
    bool shrink = dst_area_size <= src_area_size;
    bool U8 = sizeof(T) == sizeof(uchar);
    bool F32_1 = sizeof(T) == sizeof(float) && CH == 1;
    bool F32_3 = sizeof(T) == sizeof(float) && CH == 3;

    bool use_vector = (enlarge && (dst_area_size <= 500 * 500)) ||
                      (shrink && (F32_3 || (U8 && dst_area_size <= 500 * 500) ||
                                  (F32_1 && dst_area_size <= 1000 * 1000)));

    if (use_vector) {
        dim3 BLOCKS(
                DIVUP(dst_cols, THREADS.x),
                DIVUP(dst_rows, THREADS.y * ELEMENTS_PER_THREADS));

        if (sizeof(T) == sizeof(float)) {
            resize_lanczos4_32f_kernel_vector<CH><<<BLOCKS, THREADS, 0, stream>>>(
                    (const float*)src, (float*)dst, src_rows, src_cols, dst_rows,
                    dst_cols, src_step, dst_step, row_scale, col_scale);
        } else {
            resize_lanczos4_8u_kernel_vector<CH><<<BLOCKS, THREADS, 0, stream>>>(
                    (const uchar*)src, (uchar*)dst, src_rows, src_cols, dst_rows,
                    dst_cols, src_step, dst_step, row_scale, col_scale);
        }

    } else {
        dim3 BLOCKS(
                DIVUP(dst_cols, THREADS.x),
                DIVUP(dst_rows, THREADS.y * ELEMENTS_PER_THREADS));

        hipDeviceSetCacheConfig(hipFuncCachePreferL1);

        if (sizeof(T) == sizeof(float)) {
            float* dev_coef_row = static_cast<float*>(workspace);
            int* dev_sr = reinterpret_cast<int*>(dev_coef_row + dst_rows * 8);
            float* dev_coef_col = reinterpret_cast<float*>(dev_sr + dst_rows);
            int* dev_sc = reinterpret_cast<int*>(dev_coef_col + dst_cols * 8);

            precompute_lanczos4_coef_f32<<<DIVUP(dst_rows, 128), 128, 0, stream>>>(
                    dev_coef_row, row_scale, dst_rows);
            precompute_lanczos4_coef_f32<<<DIVUP(dst_cols, 128), 128, 0, stream>>>(
                    dev_coef_col, col_scale, dst_cols);
            resize_lanczos4_32f_kernel_cacheToGlobal<CH>
                    <<<BLOCKS, THREADS, 0, stream>>>(
                            (const float*)src, (float*)dst, src_rows, src_cols,
                            dst_rows, dst_cols, src_step, dst_step, dev_coef_row,
                            dev_coef_col, dev_sr, dev_sc);

        } else {
            short* dev_coef_row = static_cast<short*>(workspace);
            int* dev_sr = reinterpret_cast<int*>(dev_coef_row + dst_rows * 8);
            short* dev_coef_col = reinterpret_cast<short*>(dev_sr + dst_rows);
            int* dev_sc = reinterpret_cast<int*>(dev_coef_col + dst_cols * 8);

            precompute_lanczos4_coef_u8<<<DIVUP(dst_rows, 128), 128, 0, stream>>>(
                    dev_coef_row, row_scale, dst_rows);
            precompute_lanczos4_coef_u8<<<DIVUP(dst_cols, 128), 128, 0, stream>>>(
                    dev_coef_col, col_scale, dst_cols);

            resize_lanczos4_8u_kernel_cacheToGlobal<CH><<<BLOCKS, THREADS, 0, stream>>>(
                    (const uchar*)src, (uchar*)dst, src_rows, src_cols, dst_rows,
                    dst_cols, src_step, dst_step, dev_coef_row, dev_coef_col, dev_sr,
                    dev_sc);
        }
    }
}

template <size_t CH>
__global__ void resize_area_version1_shrink_32f_kernel(
        const float* src, float* dst, const size_t src_rows, const size_t src_cols,
        const size_t dst_rows, const size_t dst_cols, const size_t src_step,
        const size_t dst_step, const float row_scale, const float col_scale,
        const float _row_scale, const float _col_scale) {
    size_t dc = blockIdx.x * blockDim.x + threadIdx.x;
    size_t dr = blockIdx.y * blockDim.y + threadIdx.y;
    if (dr < dst_rows && dc < dst_cols) {
        float fsr1 = (float)dr * row_scale;
        float fsr2 = (float)(dr + 1) * row_scale;
        int sr1 = floor(fsr1);
        int sr2 = ceil(fsr2);

        float fsc1 = (float)dc * col_scale;
        float fsc2 = (float)(dc + 1) * col_scale;
        int sc1 = floor(fsc1);
        int sc2 = ceil(fsc2);

        float dst_data[CH] = {0};

        {
            float coefr = (float)(sr1 + 1 - fsr1) * _row_scale;
            {
                float coefc = (float)(sc1 + 1 - fsc1) * _col_scale;
                for (size_t ch = 0; ch < CH; ++ch) {
                    dst_data[ch] += coefr * coefc * at(src, sr1, sc1, ch);
                }
            }
            for (int sc = sc1 + 1; sc < sc2 - 1; ++sc) {
                float coefc = _col_scale;
                for (size_t ch = 0; ch < CH; ++ch) {
                    dst_data[ch] += coefr * coefc * at(src, sr1, sc, ch);
                }
            }
            {
                float coefc = (float)(fsc2 - (sc2 - 1)) * _col_scale;
                for (size_t ch = 0; ch < CH; ++ch) {
                    dst_data[ch] += coefr * coefc * at(src, sr1, sc2 - 1, ch);
                }
            }
        }

        for (int sr = sr1 + 1; sr < sr2 - 1; ++sr) {
            float coefr = 1.0f * _row_scale;
            {
                float coefc = (float)(sc1 + 1 - fsc1) * _col_scale;
                for (size_t ch = 0; ch < CH; ++ch) {
                    dst_data[ch] += coefr * coefc * at(src, sr, sc1, ch);
                }
            }
            for (int sc = sc1 + 1; sc < sc2 - 1; ++sc) {
                float coefc = _col_scale;
                for (size_t ch = 0; ch < CH; ++ch) {
                    dst_data[ch] += coefr * coefc * at(src, sr, sc, ch);
                }
            }
            {
                float coefc = (float)(fsc2 - (sc2 - 1)) * _col_scale;
                for (size_t ch = 0; ch < CH; ++ch) {
                    dst_data[ch] += coefr * coefc * at(src, sr, sc2 - 1, ch);
                }
            }
        }

        {
            float coefr = (float)(fsr2 - (sr2 - 1)) * _row_scale;
            {
                float coefc = (float)(sc1 + 1 - fsc1) * _col_scale;
                for (size_t ch = 0; ch < CH; ++ch) {
                    dst_data[ch] += coefr * coefc * at(src, sr2 - 1, sc1, ch);
                }
            }
            for (int sc = sc1 + 1; sc < sc2 - 1; ++sc) {
                float coefc = _col_scale;
                for (size_t ch = 0; ch < CH; ++ch) {
                    dst_data[ch] += coefr * coefc * at(src, sr2 - 1, sc, ch);
                }
            }
            {
                float coefc = (float)(fsc2 - (sc2 - 1)) * _col_scale;
                for (size_t ch = 0; ch < CH; ++ch) {
                    dst_data[ch] += coefr * coefc * at(src, sr2 - 1, sc2 - 1, ch);
                }
            }
        }

        for (size_t ch = 0; ch < CH; ++ch)
            at(dst, dr, dc, ch) = dst_data[ch];
    }
}

template <size_t CH>
__global__ void resize_area_version1_shrink_8u_kernel(
        const uchar* src, uchar* dst, const size_t src_rows, const size_t src_cols,
        const size_t dst_rows, const size_t dst_cols, const size_t src_step,
        const size_t dst_step, const float row_scale, const float col_scale,
        const float _row_scale, const float _col_scale) {
    size_t dc = blockIdx.x * blockDim.x + threadIdx.x;
    size_t dr = blockIdx.y * blockDim.y + threadIdx.y;
    if (dr < dst_rows && dc < dst_cols) {
        float fsr1 = (float)dr * row_scale;
        float fsr2 = (float)(dr + 1) * row_scale;
        int sr1 = floor(fsr1);
        int sr2 = ceil(fsr2);

        float fsc1 = (float)dc * col_scale;
        float fsc2 = (float)(dc + 1) * col_scale;
        int sc1 = floor(fsc1);
        int sc2 = ceil(fsc2);
        float dst_data[CH] = {0};

        {
            float coefr = (float)(sr1 + 1 - fsr1) * _row_scale;
            {
                float coefc = (float)(sc1 + 1 - fsc1) * _col_scale;
                for (size_t ch = 0; ch < CH; ++ch) {
                    dst_data[ch] += coefr * coefc * at(src, sr1, sc1, ch);
                }
            }
            for (int sc = sc1 + 1; sc < sc2 - 1; ++sc) {
                float coefc = _col_scale;
                for (size_t ch = 0; ch < CH; ++ch) {
                    dst_data[ch] += coefr * coefc * at(src, sr1, sc, ch);
                }
            }
            {
                float coefc = (float)(fsc2 - (sc2 - 1)) * _col_scale;
                for (size_t ch = 0; ch < CH; ++ch) {
                    dst_data[ch] += coefr * coefc * at(src, sr1, sc2 - 1, ch);
                }
            }
        }
        for (int sr = sr1 + 1; sr < sr2 - 1; ++sr) {
            float coefr = 1.0f * _row_scale;
            {
                float coefc = (float)(sc1 + 1 - fsc1) * _col_scale;
                for (size_t ch = 0; ch < CH; ++ch) {
                    dst_data[ch] += coefr * coefc * at(src, sr, sc1, ch);
                }
            }
            for (int sc = sc1 + 1; sc < sc2 - 1; ++sc) {
                float coefc = _col_scale;
                for (size_t ch = 0; ch < CH; ++ch) {
                    dst_data[ch] += coefr * coefc * at(src, sr, sc, ch);
                }
            }
            {
                float coefc = (float)(fsc2 - (sc2 - 1)) * _col_scale;
                for (size_t ch = 0; ch < CH; ++ch) {
                    dst_data[ch] += coefr * coefc * at(src, sr, sc2 - 1, ch);
                }
            }
        }

        {
            float coefr = (float)(fsr2 - (sr2 - 1)) * _row_scale;
            {
                float coefc = (float)(sc1 + 1 - fsc1) * _col_scale;
                for (size_t ch = 0; ch < CH; ++ch) {
                    dst_data[ch] += coefr * coefc * at(src, sr2 - 1, sc1, ch);
                }
            }
            for (int sc = sc1 + 1; sc < sc2 - 1; ++sc) {
                float coefc = _col_scale;
                for (size_t ch = 0; ch < CH; ++ch) {
                    dst_data[ch] += coefr * coefc * at(src, sr2 - 1, sc, ch);
                }
            }
            {
                float coefc = (float)(fsc2 - (sc2 - 1)) * _col_scale;
                for (size_t ch = 0; ch < CH; ++ch) {
                    dst_data[ch] += coefr * coefc * at(src, sr2 - 1, sc2 - 1, ch);
                }
            }
        }

        for (size_t ch = 0; ch < CH; ++ch)
            at(dst, dr, dc, ch) = saturate((int)dst_data[ch], 0, 255);
    }
}

template <size_t CH>
__global__ void resize_area_version2_shrink_32f_kernel(
        const float* src, float* dst, const size_t src_rows, const size_t src_cols,
        const size_t dst_rows, const size_t dst_cols, const size_t src_step,
        const size_t dst_step, const float row_scale, const float col_scale,
        const float _row_scale, const float _col_scale) {
    size_t dc0 = blockIdx.x * blockDim.x;
    size_t dr = blockIdx.y * blockDim.y + threadIdx.y;
    if (dr < dst_rows && dc0 < dst_cols) {
        __shared__ float lc_dst_data[THREADS_Y][THREADS_X * CH];

        size_t dc = dc0 + threadIdx.x;

        float fsr1 = (float)dr * row_scale;
        float fsr2 = (float)(dr + 1) * row_scale;
        int sr1 = floor(fsr1);
        int sr2 = ceil(fsr2);

        float fsc1 = (float)dc0 * col_scale;
        float fsc2 = (float)(dc0 + blockDim.x) * col_scale;
        int sc1 = floor(fsc1);
        int sc2 = ceil(fsc2);

        for (size_t ch = 0; ch < CH; ch++)
            lc_dst_data[threadIdx.y][threadIdx.x * CH + ch] = 0;

        __syncthreads();

        size_t min_col_edge = min((int)src_cols, sc2) * CH;
        for (int sc_address = sc1 * CH + threadIdx.x; sc_address < min_col_edge;
             sc_address += blockDim.x) {
            float sum = 0;
            {
                float coefr = (float)(sr1 + 1 - fsr1) * _row_scale;
                sum += coefr * src[sr1 * src_step + sc_address];
            }
            float coefr = _row_scale;
            for (int sr = sr1 + 1; sr < sr2 - 1; ++sr) {
                sum += coefr * src[sr * src_step + sc_address];
            }
            {
                float coefr = (float)(fsr2 - (sr2 - 1)) * _row_scale;
                sum += coefr * src[(sr2 - 1) * src_step + sc_address];
            }

            size_t multi = floor(((sc_address / CH) + 1) * _col_scale);
            float x = ((sc_address / CH) + 1) - multi * col_scale;
            if (x >= 1) {
                atomicAdd(
                        &(lc_dst_data[threadIdx.y]
                                     [(multi - dc0) * CH + sc_address % CH]),
                        sum * _col_scale);
            } else {
                if (multi < dc0 + blockDim.x)
                    atomicAdd(
                            &(lc_dst_data[threadIdx.y]
                                         [(multi - dc0) * CH + sc_address % CH]),
                            sum * (x * _col_scale));
                if (multi - 1 >= dc0)
                    atomicAdd(
                            &(lc_dst_data[threadIdx.y]
                                         [(multi - 1 - dc0) * CH + sc_address % CH]),
                            sum * ((1 - x) * _col_scale));
            }
        }

        __syncthreads();

        if (dc < dst_cols) {
            for (size_t ch = 0; ch < CH; ++ch)
                at(dst, dr, dc, ch) = lc_dst_data[threadIdx.y][(threadIdx.x) * CH + ch];
        }
    }
}

template <size_t CH>
__global__ void resize_area_version2_shrink_8u_kernel(
        const uchar* src, uchar* dst, const size_t src_rows, const size_t src_cols,
        const size_t dst_rows, const size_t dst_cols, const size_t src_step,
        const size_t dst_step, const float row_scale, const float col_scale,
        const float _row_scale, const float _col_scale) {
    size_t dc0 = blockIdx.x * blockDim.x;
    size_t dr = blockIdx.y * blockDim.y + threadIdx.y;
    if (dr < dst_rows && dc0 < dst_cols) {
        __shared__ float lc_dst_data[THREADS_Y][THREADS_X * CH];

        size_t dc = dc0 + threadIdx.x;

        float fsr1 = (float)dr * row_scale;
        float fsr2 = (float)(dr + 1) * row_scale;
        int sr1 = floor(fsr1);
        int sr2 = ceil(fsr2);

        float fsc1 = (float)dc0 * col_scale;
        float fsc2 = (float)(dc0 + blockDim.x) * col_scale;
        int sc1 = floor(fsc1);
        int sc2 = ceil(fsc2);

        for (size_t ch = 0; ch < CH; ch++)
            lc_dst_data[threadIdx.y][threadIdx.x * CH + ch] = 0;

        __syncthreads();

        size_t min_col_edge = min((int)src_cols, sc2) * CH;
        for (int sc_address = sc1 * CH + threadIdx.x; sc_address < min_col_edge;
             sc_address += blockDim.x) {
            float sum = 0;
            {
                float coefr = (float)(sr1 + 1 - fsr1) * _row_scale;
                sum += coefr * src[sr1 * src_step + sc_address];
            }
            float coefr = _row_scale;
            for (int sr = sr1 + 1; sr < sr2 - 1; ++sr) {
                sum += coefr * src[sr * src_step + sc_address];
            }
            {
                float coefr = (float)(fsr2 - (sr2 - 1)) * _row_scale;
                sum += coefr * src[(sr2 - 1) * src_step + sc_address];
            }

            size_t multi = floor(((sc_address / CH) + 1) * _col_scale);
            float x = ((sc_address / CH) + 1) - multi * col_scale;
            if (x >= 1) {
                atomicAdd(
                        &(lc_dst_data[threadIdx.y]
                                     [(multi - dc0) * CH + sc_address % CH]),
                        sum * _col_scale);
            } else {
                if (multi < dc0 + blockDim.x)
                    atomicAdd(
                            &(lc_dst_data[threadIdx.y]
                                         [(multi - dc0) * CH + sc_address % CH]),
                            sum * (x * _col_scale));
                if (multi - 1 >= dc0)
                    atomicAdd(
                            &(lc_dst_data[threadIdx.y]
                                         [(multi - 1 - dc0) * CH + sc_address % CH]),
                            sum * ((1 - x) * _col_scale));
            }
        }

        __syncthreads();

        if (dc < dst_cols) {
            for (size_t ch = 0; ch < CH; ++ch)
                at(dst, dr, dc, ch) = saturate(
                        (int)lc_dst_data[threadIdx.y][(threadIdx.x) * CH + ch], 0, 255);
        }
    }
}

template <size_t CH>
__global__ void resize_area_version1_shrink_fast_32f_kernel(
        const float* __restrict__ src, float* dst, const size_t src_rows,
        const size_t src_cols, const size_t dst_rows, const size_t dst_cols,
        const size_t src_step, const size_t dst_step, const size_t cell_rows,
        const size_t cell_cols, const float _cell_rows, const float _cell_cols) {
    size_t dc = blockIdx.x * blockDim.x + threadIdx.x;
    size_t dr = blockIdx.y * blockDim.y + threadIdx.y;
    if (dr < dst_rows && dc < dst_cols) {
        int sr0 = dr * cell_rows;
        int sc0 = dc * cell_cols;
        float dst_data[CH] = {0};
        for (int sr = sr0; sr < cell_rows + sr0; ++sr) {
            for (int sc = sc0; sc < cell_cols + sc0; ++sc) {
                for (size_t ch = 0; ch < CH; ++ch) {
                    dst_data[ch] += at(src, sr, sc, ch);
                }
            }
        }

        for (size_t ch = 0; ch < CH; ++ch)
            at(dst, dr, dc, ch) = dst_data[ch] * _cell_rows * _cell_cols;
    }
}

template <size_t CH>
__global__ void resize_area_version1_shrink_fast_8u_kernel(
        const uchar* __restrict__ src, uchar* dst, const size_t src_rows,
        const size_t src_cols, const size_t dst_rows, const size_t dst_cols,
        const size_t src_step, const size_t dst_step, const size_t cell_rows,
        const size_t cell_cols, const float _cell_rows, const float _cell_cols) {
    size_t dc = blockIdx.x * blockDim.x + threadIdx.x;
    size_t dr = blockIdx.y * blockDim.y + threadIdx.y;
    if (dr < dst_rows && dc < dst_cols) {
        int sr0 = dr * cell_rows;
        int sc0 = dc * cell_cols;
        int dst_data[CH] = {0};
        for (int sr = sr0; sr < cell_rows + sr0; ++sr) {
            for (int sc = sc0; sc < cell_cols + sc0; ++sc) {
                for (size_t ch = 0; ch < CH; ++ch) {
                    dst_data[ch] += at(src, sr, sc, ch);
                }
            }
        }

        for (size_t ch = 0; ch < CH; ++ch) {
            at(dst, dr, dc, ch) = (uchar)(dst_data[ch] * _cell_rows * _cell_cols);
        }
    }
}

template <size_t CH>
__global__ void resize_area_version2_shrink_fast_32f_kernel(
        const float* __restrict__ src, float* dst, const size_t src_rows,
        const size_t src_cols, const size_t dst_rows, const size_t dst_cols,
        const size_t src_step, const size_t dst_step, const size_t cell_rows,
        const size_t cell_cols, const float _cell_rows, const float _cell_cols) {
    size_t dc0 = blockIdx.x * blockDim.x;
    size_t dr = blockIdx.y * blockDim.y + threadIdx.y;
    if (dr < dst_rows && dc0 < dst_cols) {
        __shared__ float lc_dst_data[THREADS_Y][THREADS_X * CH];
        int sc0 = dc0 * cell_cols * CH;
        int sr0 = dr * cell_rows;

        for (size_t ch = 0; ch < CH; ch++)
            lc_dst_data[threadIdx.y][threadIdx.x * CH + ch] = 0;

        __syncthreads();

        size_t block_cell_width = cell_cols * CH * blockDim.x;
        for (int i = threadIdx.x, sc = sc0 + threadIdx.x;
             i < block_cell_width && sc < src_cols * CH;
             i += blockDim.x, sc += blockDim.x) {
            float sum = 0;
            for (int j = 0, sr = sr0 * src_step; j < cell_rows; j++, sr += src_step)
                sum += src[sr + sc];
            atomicAdd(
                    &(lc_dst_data[threadIdx.y][(i / (cell_cols * CH)) * CH + i % CH]),
                    sum);
        }

        __syncthreads();

        size_t dc = dc0 + threadIdx.x;
        if (dc < dst_cols) {
            for (size_t ch = 0; ch < CH; ++ch)
                at(dst, dr, dc, ch) = lc_dst_data[threadIdx.y][threadIdx.x * CH + ch] *
                                      _cell_rows * _cell_cols;
        }
    }
}

template <size_t CH>
__global__ void resize_area_version2_shrink_fast_8u_kernel(
        const uchar* __restrict__ src, uchar* dst, const size_t src_rows,
        const size_t src_cols, const size_t dst_rows, const size_t dst_cols,
        const size_t src_step, const size_t dst_step, const size_t cell_rows,
        const size_t cell_cols, const float _cell_rows, const float _cell_cols) {
    size_t dc0 = blockIdx.x * blockDim.x;
    size_t dr = blockIdx.y * blockDim.y + threadIdx.y;
    if (dr < dst_rows && dc0 < dst_cols) {
        __shared__ int lc_dst_data[THREADS_Y][THREADS_X * CH];
        int sc0 = dc0 * cell_cols * CH;
        int sr0 = dr * cell_rows;

        for (size_t ch = 0; ch < CH; ch++)
            lc_dst_data[threadIdx.y][threadIdx.x * CH + ch] = 0;

        __syncthreads();

        size_t block_cell_width = cell_cols * CH * blockDim.x;
        for (int i = threadIdx.x, sc = sc0 + threadIdx.x;
             i < block_cell_width && sc < src_cols * CH;
             i += blockDim.x, sc += blockDim.x) {
            int sum = 0;
            for (int j = 0, sr = sr0 * src_step; j < cell_rows; j++, sr += src_step)
                sum += src[sr + sc];
            atomicAdd(
                    &(lc_dst_data[threadIdx.y][(i / (cell_cols * CH)) * CH + i % CH]),
                    sum);
        }

        __syncthreads();

        size_t dc = dc0 + threadIdx.x;
        if (dc < dst_cols) {
            for (size_t ch = 0; ch < CH; ++ch)
                at(dst, dr, dc, ch) =
                        (uchar)(lc_dst_data[threadIdx.y][threadIdx.x * CH + ch] *
                                _cell_rows * _cell_cols);
        }
    }
}

template <typename T, size_t CH>
void resize_area_proxy(
        const T* src, T* dst, const size_t src_rows, const size_t src_cols,
        const size_t dst_rows, const size_t dst_cols, const size_t src_step,
        const size_t dst_step, void* workspace, hipStream_t stream) {
    dim3 THREADS(THREADS_X, THREADS_Y, 1);

    float row_scale = (float)src_rows / dst_rows;
    float col_scale = (float)src_cols / dst_cols;

    if (src_rows > dst_rows && src_cols > dst_cols) {
        if (src_rows % dst_rows == 0 && src_cols % dst_cols == 0) {
            dim3 BLOCKS(DIVUP(dst_cols, THREADS.x), DIVUP(dst_rows, THREADS.y));

            if (sizeof(T) == sizeof(float)) {
                if ((CH == 1 && (sizeof(T) * CH * col_scale <= 24)) ||
                    (CH == 3 && (sizeof(T) * CH * col_scale <= 36))) {
                    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
                    resize_area_version1_shrink_fast_32f_kernel<CH>
                            <<<BLOCKS, THREADS, 0, stream>>>(
                                    (const float*)src, (float*)dst, src_rows, src_cols,
                                    dst_rows, dst_cols, src_step, dst_step,
                                    (size_t)row_scale, (size_t)col_scale,
                                    (float)1 / row_scale, (float)1 / col_scale);
                } else {
                    hipDeviceSetCacheConfig(hipFuncCachePreferShared);
                    resize_area_version2_shrink_fast_32f_kernel<CH>
                            <<<BLOCKS, THREADS, 0, stream>>>(
                                    (const float*)src, (float*)dst, src_rows, src_cols,
                                    dst_rows, dst_cols, src_step, dst_step,
                                    (size_t)row_scale, (size_t)col_scale,
                                    (float)1 / row_scale, (float)1 / col_scale);
                }

            } else {
                if (sizeof(T) * CH * col_scale <= 24) {
                    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
                    resize_area_version1_shrink_fast_8u_kernel<CH>
                            <<<BLOCKS, THREADS, 0, stream>>>(
                                    (const uchar*)src, (uchar*)dst, src_rows, src_cols,
                                    dst_rows, dst_cols, src_step, dst_step,
                                    (size_t)row_scale, (size_t)col_scale,
                                    (float)1 / row_scale, (float)1 / col_scale);
                } else {
                    hipDeviceSetCacheConfig(hipFuncCachePreferShared);
                    resize_area_version2_shrink_fast_8u_kernel<CH>
                            <<<BLOCKS, THREADS, 0, stream>>>(
                                    (const uchar*)src, (uchar*)dst, src_rows, src_cols,
                                    dst_rows, dst_cols, src_step, dst_step,
                                    (size_t)row_scale, (size_t)col_scale,
                                    (float)1 / row_scale, (float)1 / col_scale);
                }
            }

        } else {
            size_t access_step = (int)(sizeof(T) * CH * col_scale);
            if (access_step <= 24) {
                dim3 BLOCKS(DIVUP(dst_cols, THREADS.x), DIVUP(dst_rows, THREADS.y));

                hipDeviceSetCacheConfig(hipFuncCachePreferL1);

                if (sizeof(T) == sizeof(float)) {
                    resize_area_version1_shrink_32f_kernel<CH>
                            <<<BLOCKS, THREADS, 0, stream>>>(
                                    (const float*)src, (float*)dst, src_rows, src_cols,
                                    dst_rows, dst_cols, src_step, dst_step, row_scale,
                                    col_scale, (float)1 / row_scale,
                                    (float)1 / col_scale);
                } else {
                    resize_area_version1_shrink_8u_kernel<CH>
                            <<<BLOCKS, THREADS, 0, stream>>>(
                                    (const uchar*)src, (uchar*)dst, src_rows, src_cols,
                                    dst_rows, dst_cols, src_step, dst_step, row_scale,
                                    col_scale, (float)1 / row_scale,
                                    (float)1 / col_scale);
                }

            } else if (access_step > 24) {
                dim3 BLOCKS(DIVUP(dst_cols, THREADS.x), DIVUP(dst_rows, THREADS.y));

                hipDeviceSetCacheConfig(hipFuncCachePreferNone);

                if (sizeof(T) == sizeof(float)) {
                    resize_area_version2_shrink_32f_kernel<CH>
                            <<<BLOCKS, THREADS, 0, stream>>>(
                                    (const float*)src, (float*)dst, src_rows, src_cols,
                                    dst_rows, dst_cols, src_step, dst_step, row_scale,
                                    col_scale, (float)1 / row_scale,
                                    (float)1 / col_scale);
                } else {
                    resize_area_version2_shrink_8u_kernel<CH>
                            <<<BLOCKS, THREADS, 0, stream>>>(
                                    (const uchar*)src, (uchar*)dst, src_rows, src_cols,
                                    dst_rows, dst_cols, src_step, dst_step, row_scale,
                                    col_scale, (float)1 / row_scale,
                                    (float)1 / col_scale);
                }
            }
        }
    } else {
        resize_linear_proxy<T, CH>(
                src, dst, src_rows, src_cols, dst_rows, dst_cols, src_step, dst_step,
                workspace, stream);
    }
}

}  // anonymous namespace

template <typename T>
void megdnn::cuda::resize::resize_cv(
        const T* src, T* dst, const size_t src_rows, const size_t src_cols,
        const size_t dst_rows, const size_t dst_cols, const size_t src_step,
        const size_t dst_step, size_t ch, InterpolationMode imode, void* workspace,
        hipStream_t stream) {
    megdnn_assert(ch == 1 || ch == 3);
#define cb(_mode, _MODE)                                                        \
    case INTER_##_MODE: {                                                       \
        if (ch == 1) {                                                          \
            resize_##_mode##_proxy<T, 1>(                                       \
                    src, dst, src_rows, src_cols, dst_rows, dst_cols, src_step, \
                    dst_step, workspace, stream);                               \
        } else {                                                                \
            resize_##_mode##_proxy<T, 3>(                                       \
                    src, dst, src_rows, src_cols, dst_rows, dst_cols, src_step, \
                    dst_step, workspace, stream);                               \
        }                                                                       \
        break;                                                                  \
    }

    switch (imode) {
        cb(nearest, NEAREST);
        cb(linear, LINEAR);
        cb(cubic, CUBIC);
        cb(lanczos4, LANCZOS4);
        cb(area, AREA);
        default:
            megdnn_throw("unsupported interpolation mode");
            break;
    }
#undef cb
}

#define INST(_type)                                                              \
    template void megdnn::cuda::resize::resize_cv<_type>(                        \
            const _type* src, _type* dst, const size_t src_rows,                 \
            const size_t src_cols, const size_t dst_rows, const size_t dst_cols, \
            const size_t src_step, const size_t dst_step, size_t ch,             \
            InterpolationMode imode, void* workspace, hipStream_t stream);

INST(float);
INST(uchar);

#undef cb

// vim: syntax=cpp.doxygen
