#include "hip/hip_runtime.h"
/**
 * \file src/cuda/convolution/backward_data/deconv_int8_helper.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or
 * implied.
 */

#include "src/cuda/convolution/backward_data/deconv_int8_helper.cuh"
#include "src/cuda/transpose_utils.cuh"

using namespace megdnn;
using namespace cuda;
using namespace deconv;

#define BLOCKSIZE_X 16
#define BLOCKSIZE_Y 16

namespace {

__global__ void reorder_filter_nc4hw4_to_n4hwc4_kernel(
        int8_t* __restrict__ dst, const int8_t* __restrict__ src, uint32_t OC,
        uint32_t IC, uint32_t FHFW) {
    const int32_t ocb = blockIdx.z;
    const int32_t icb = blockIdx.y * BLOCKSIZE_X + threadIdx.y;
    const int32_t fhfw = blockIdx.x * BLOCKSIZE_Y + threadIdx.x;

    if (fhfw < FHFW && icb < IC / 4) {
        array_wrapper<int, 4> src_value;
        int dst_value[4];
#pragma unroll
        for (int i = 0; i < 4; i++) {
            src_value[i] = *reinterpret_cast<const int*>(
                    src + (ocb * 4 + i) * IC * FHFW + (icb * FHFW + fhfw) * 4);
        }

        // transpose 4x4
        auto trans = transpose_int8_interleavedx4<4, int>();
        trans(src_value, dst_value);

#pragma unroll
        for (int i = 0; i < 4; i++) {
            *reinterpret_cast<int*>(
                    dst + (ocb * FHFW * IC + fhfw * IC + icb * 4 + i) * 4) =
                    dst_value[i];
        }
    }
}

template <uint32_t interleaved, typename vec_type>
__global__ void reorder_filter_nhwc_to_cnxhwx_kernel(
        int8_t* __restrict__ dst, const int8_t* __restrict__ src, uint32_t OC,
        uint32_t IC, uint32_t FHFW) {
    uint32_t lane = threadIdx.x + blockIdx.x * blockDim.x;
    const int32_t ocb = lane / (FHFW * IC / 4);
    const int32_t fhfw_icb = lane % (FHFW * IC / 4);
    const int32_t fhfw = fhfw_icb / (IC / 4);
    const int32_t icb = fhfw_icb % (IC / 4);

    if (ocb < OC / interleaved && fhfw < FHFW) {
        array_wrapper<int, interleaved> src_value;
        vec_type dst_value[4];

#pragma unroll
        for (int i = 0; i < interleaved; i++) {
            src_value[i] = *reinterpret_cast<const int*>(
                    src + (ocb * interleaved + i) * FHFW * IC + fhfw * IC + icb * 4);
        }

        auto trans = transpose_int8_interleavedx4<interleaved, vec_type>();
        trans(src_value, dst_value);

#pragma unroll
        for (int i = 0; i < 4; i++) {
            *reinterpret_cast<vec_type*>(
                    dst + (icb * 4 + i) * FHFW * OC +
                    (ocb * FHFW + fhfw) * interleaved) = dst_value[i];
        }
    }
}

}  // namespace

void megdnn::cuda::deconv::reorder_filter_nc4hw4_to_n4hwc4(
        int8_t* dst, const int8_t* src, uint32_t OC, uint32_t IC, uint32_t FH,
        uint32_t FW, hipStream_t stream) {
    dim3 threads(BLOCKSIZE_X, BLOCKSIZE_Y, 1);
    dim3 blocks(DIVUP(FH * FW, BLOCKSIZE_X), DIVUP(IC / 4, BLOCKSIZE_Y), OC / 4);

    reorder_filter_nc4hw4_to_n4hwc4_kernel<<<blocks, threads, 0, stream>>>(
            dst, src, OC, IC, FH * FW);
    after_kernel_launch();
}

void megdnn::cuda::deconv::reorder_filter_nhwc_to_cnxhwx(
        int8_t* dst, const int8_t* src, uint32_t OC, uint32_t IC, uint32_t FH,
        uint32_t FW, uint32_t interleaved, hipStream_t stream) {
    int32_t vthreads = OC / interleaved * IC / 4 * FH * FW;
    int32_t nr_threads = std::min(256, vthreads);
    int32_t nr_blocks = DIVUP(vthreads, nr_threads);

    if (interleaved == 4) {
        reorder_filter_nhwc_to_cnxhwx_kernel<4, int>
                <<<nr_blocks, nr_threads, 0, stream>>>(dst, src, OC, IC, FH * FW);
    } else if (interleaved == 8) {
        reorder_filter_nhwc_to_cnxhwx_kernel<8, int2>
                <<<nr_blocks, nr_threads, 0, stream>>>(dst, src, OC, IC, FH * FW);
    } else {
        reorder_filter_nhwc_to_cnxhwx_kernel<16, int4>
                <<<nr_blocks, nr_threads, 0, stream>>>(dst, src, OC, IC, FH * FW);
    }
    after_kernel_launch();
}

// vim: syntax=cuda.doxygen
