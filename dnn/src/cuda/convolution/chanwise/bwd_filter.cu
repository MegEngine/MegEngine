#include "hip/hip_runtime.h"
/**
 * \file dnn/src/cuda/convolution/chanwise/bwd_filter.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2020 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 */

#include "./kern.cuh"
#include "./kern_helper.cuh"
#include "src/cuda/cub/util_ptx.cuh"
#include "hip/hip_fp16.h"
#include "src/cuda/fp16_help.cuh"

const uint32_t WARP_SIZE = 32, BATCH_UNROLL = 4;

using namespace megdnn;
using namespace cuda;
using namespace convolution;
using namespace chanwise;

namespace {

/*!
 * \brief compute grad w.r.t. filter
 *
 * block dim: out_id * kern_id
 * threads with the same out_id computes grad for corresponding kernel element
 * \tparam nr_thpf number of threads for one element in the filter; must be
 *      power of 2;
 */
template<typename T, uint32_t nr_thpf>
__global__ void kern_bwd_filter_float(
        T* flt_grad, const T* src, const T* dst_grad, Param param) {

    const uint32_t
        N = param.batch, IC = param.src_chl, IH = param.src_h, IW = param.src_w,
        CHL_MUL = param.chl_mul,
        FH = param.flt_h, FW = param.flt_w,
        PH = param.pad_h, PW = param.pad_w,
        SH = param.stride_h, SW = param.stride_w,
        OH = param.out_h, OW = param.out_w,
        SRC_BATCH_STRIDE = IC * IH * IW,
        DST_BATCH_STRIDE = IC * CHL_MUL * OH * OW,
        BLKDIM_X = blockDim.x / nr_thpf,
        THREADID_X = threadIdx.x / nr_thpf,
        OUT_IDX = blockIdx.x * BLKDIM_X + THREADID_X;

    uint32_t ic, chl_mul, fh, fw;
    {
        uint32_t i = OUT_IDX;
        i = div_mod(i, FW, fw);
        i = div_mod(i, FH, fh);
        i = div_mod(i, CHL_MUL, chl_mul);
        ic = i;
    }
    if (ic >= IC) {
        return;
    }
    src += ic * IH * IW;
    dst_grad += (ic * CHL_MUL + chl_mul) * OH * OW;

    const uint32_t
        oh_lo = max(int32_t(PH - fh + SH - 1), 0) / SH,
        oh_hi = min((IH - 1 + PH - fh) / SH + 1, OH),
        ow_lo = max(int32_t(PW - fw + SW - 1), 0) / SW,
        ow_hi = min((IW - 1 + PW - fw) / SW + 1, OW),
        oblk_h = oh_hi - oh_lo,
        oblk_w = ow_hi - ow_lo,
        oblk_tot = oblk_h * oblk_w * ((N + BATCH_UNROLL - 1) / BATCH_UNROLL),
        tid = threadIdx.x % nr_thpf;

    if (IH + PH < fh + 1 || oh_lo >= oh_hi ||
            IW + PW < fw + 1 || ow_lo >= ow_hi) {
        if (!tid)
            flt_grad[OUT_IDX] = 0;
        return;
    }

    T sum(0);
    for (uint32_t oblk_idx = tid; oblk_idx < oblk_tot; oblk_idx += nr_thpf) {
        uint32_t n, oh, ow;
        n = div_mod(div_mod(oblk_idx, oblk_w, ow), oblk_h, oh) * BATCH_UNROLL;
        oh += oh_lo;
        ow += ow_lo;
        uint32_t ih = oh * SH - PH + fh,
                 iw = ow * SW - PW + fw,
                 soff = ih * IW + iw + n * SRC_BATCH_STRIDE,
                 doff = oh * OW + ow + n * DST_BATCH_STRIDE;
#pragma unroll
        for (uint32_t i = 0; i < BATCH_UNROLL; ++ i) {
            if (!i || n + i < N) {
                sum += src[soff] * dst_grad[doff];
            }
            soff += SRC_BATCH_STRIDE;
            doff += DST_BATCH_STRIDE;
        }
    }

    if (nr_thpf == 1) {
        flt_grad[OUT_IDX] = sum;
    } else {
        // reduce all sums in a block
        extern __shared__ uint8_t shared_storage[];
        volatile T* thread_sum = reinterpret_cast<T*>(shared_storage);
        thread_sum += THREADID_X * nr_thpf;
        thread_sum[tid] = sum;
#pragma unroll
        for (uint32_t i = nr_thpf / 2; i; i >>= 1) {
            bool cond = nr_thpf >= i * 2 && tid < i;
            if (i >= WARP_SIZE) {
                __syncthreads();
            } else {
                hipcub::WARP_SYNC(0xffffffff);
            }
            if (cond) {
                T v0 = thread_sum[tid], v1 = v0 + thread_sum[tid + i];
                thread_sum[tid] = v1;
            }
        }

        if (!tid) {
            flt_grad[OUT_IDX] = thread_sum[0];
        }
    }
}

#if CUDA_VERSION >= 9000
template<typename T, uint32_t nr_thpf>
__global__ void kern_bwd_filter_hf(
		__half* flt_grad, const __half* src, const __half* dst_grad, Param param) {
	const uint32_t
		N = param.batch, IC = param.src_chl, IH = param.src_h, IW = param.src_w,
		CHL_MUL = param.chl_mul,
		FH = param.flt_h, FW = param.flt_w,
		PH = param.pad_h, PW = param.pad_w,
		SH = param.stride_h, SW = param.stride_w,
		OH = param.out_h, OW = param.out_w,
		SRC_BATCH_STRIDE = IC * IH * IW,
		DST_BATCH_STRIDE = IC * CHL_MUL * OH * OW,
		BLKDIM_X = (blockDim.x / nr_thpf) * 2,
		THREADID_X = (threadIdx.x / nr_thpf) * 2,
		OUT_IDX = blockIdx.x * BLKDIM_X + THREADID_X,
        LAST_IDX = FH * FW * CHL_MUL * IC,
        tid = threadIdx.x % nr_thpf;
    __half2 sum2{0.0, 0.0};

	if (OUT_IDX % FW != FW - 1) {
		uint32_t ic, chl_mul, fh, fw;
		{
			uint32_t i = OUT_IDX;
			i = div_mod(i, FW, fw);
			i = div_mod(i, FH, fh);
			i = div_mod(i, CHL_MUL, chl_mul);
			ic = i;
		}
		if (ic >= IC) {
			return;
		}
		src += ic * IH * IW;
		dst_grad += (ic * CHL_MUL + chl_mul) * OH * OW;

		const uint32_t
			oh_lo = max(int32_t(PH - fh + SH - 1), 0) / SH,
			oh_hi = min((IH - 1 + PH - fh) / SH + 1, OH),
			ow_lox = max(int32_t(PW - fw + SW - 1), 0) / SW,
			ow_loy = max(int32_t(PW - fw + SW - 2), 0) / SW,
			ow_hix = min((IW - 1 + PW - fw) / SW + 1, OW),
			ow_hiy = min((IW - 2 + PW - fw) / SW + 1, OW),
			oblk_h = oh_hi - oh_lo,
			oblk_wx = ow_hix - ow_lox,
			oblk_wy = ow_hiy - ow_loy;
        if (IH + PH < fh + 1 || oh_lo >= oh_hi || IW + PW < fw + 1) {
            if (!tid) {
                flt_grad[OUT_IDX] = 0;
                flt_grad[OUT_IDX + 1] = 0;
            }
            return;
        }
	
		if (ow_lox >= ow_hix) {
			if (!tid)
				flt_grad[OUT_IDX] = 0;
		}

		if (IW + PW < fw + 2 || ow_loy >= ow_hiy) {
			if (!tid)
				flt_grad[OUT_IDX + 1] = 0;
            if (ow_lox >= ow_hix)
                return;
		}

		sum2.x = 0.0;
		sum2.y = 0.0;
		__half2 src2{0.0, 0.0};
		__half2 dst2{0.0, 0.0};

		const uint32_t
			oblk_w = max(ow_hix, ow_hiy) - min(ow_lox, ow_loy),
			oblk_tot = oblk_h * oblk_w * ((N + BATCH_UNROLL - 1) / BATCH_UNROLL);

		for (uint32_t oblk_idx = tid; oblk_idx < oblk_tot; oblk_idx += nr_thpf) {
			uint32_t n_x, n_y, oh, ow_x, ow_y;
			n_x = div_mod(div_mod(oblk_idx, oblk_wx, ow_x), oblk_h, oh) * BATCH_UNROLL;
			n_y = div_mod(div_mod(oblk_idx, oblk_wy, ow_y), oblk_h, oh) * BATCH_UNROLL;
			oh += oh_lo;
			ow_x += ow_lox;
			ow_y += ow_loy;
			uint32_t ih = oh * SH - PH + fh,
					 iw_x = ow_x * SW - PW + fw,
					 iw_y = ow_y * SW - PW + fw + 1,
					 soff_x = ih * IW + iw_x + n_x * SRC_BATCH_STRIDE,
					 soff_y = ih * IW + iw_y + n_y * SRC_BATCH_STRIDE,
					 doff_x = oh * OW + ow_x + n_x * DST_BATCH_STRIDE,
					 doff_y = oh * OW + ow_y + n_y * DST_BATCH_STRIDE;
#pragma unroll
			for (uint32_t i = 0; i < BATCH_UNROLL; ++ i) {
				if (!i || n_x + i < N || n_y + i < N) {
					src2.x = 0.0;
					src2.y = 0.0;
					dst2.x = 0.0;
					dst2.y = 0.0;
					if (n_x + i < N && ow_x < ow_hix) {
						src2.x = src[soff_x];
						dst2.x = dst_grad[doff_x];
					}
					if (n_y + i < N && ow_y < ow_hiy) {
						src2.y = src[soff_y];
						dst2.y = dst_grad[doff_y];
					}
					sum2 = fma2(src2, dst2, sum2);
				}
				soff_x += SRC_BATCH_STRIDE;
				soff_y += SRC_BATCH_STRIDE;
				doff_x += DST_BATCH_STRIDE;
				doff_y += DST_BATCH_STRIDE;
			}
		}
	} else {
		for (size_t offset = 0; offset < 2; ++ offset) {
			uint32_t ic, chl_mul, fh, fw;
			{
				uint32_t i = OUT_IDX + offset;
				i = div_mod(i, FW, fw);
				i = div_mod(i, FH, fh);
				i = div_mod(i, CHL_MUL, chl_mul);
				ic = i;
			}
			if (ic >= IC) {
				if (offset == 0)
                    return;
                else
                    break;
			}
			const uint32_t
				oh_lo = max(int32_t(PH - fh + SH - 1), 0) / SH,
				oh_hi = min((IH - 1 + PH - fh) / SH + 1, OH),
				ow_lo = max(int32_t(PW - fw + SW - 1), 0) / SW,
				ow_hi = min((IW - 1 + PW - fw) / SW + 1, OW),
				oblk_h = oh_hi - oh_lo,
				oblk_w = ow_hi - ow_lo,
				oblk_tot = oblk_h * oblk_w * ((N + BATCH_UNROLL - 1) / BATCH_UNROLL);

			if (IH + PH < fh + 1 || oh_lo >= oh_hi ||
					IW + PW < fw + 1 || ow_lo >= ow_hi) {
				if (!tid)
					flt_grad[OUT_IDX + offset] = 0;
				continue;
			}

			__half sum(0.0);

			for (uint32_t oblk_idx = tid; oblk_idx < oblk_tot; oblk_idx += nr_thpf) {
				uint32_t n, oh, ow;
				n = div_mod(div_mod(oblk_idx, oblk_w, ow), oblk_h, oh) * BATCH_UNROLL;
				oh += oh_lo;
				ow += ow_lo;
				uint32_t ih = oh * SH - PH + fh,
						 iw = ow * SW - PW + fw,
						 soff = ic * IH * IW + ih * IW + iw + n * SRC_BATCH_STRIDE,
						 doff = (ic * CHL_MUL + chl_mul) * OH * OW + oh * OW + ow + n * DST_BATCH_STRIDE;
#pragma unroll
				for (uint32_t i = 0; i < BATCH_UNROLL; ++ i) {
					if (!i || n + i < N) {
						sum = fma(src[soff], dst_grad[doff], sum);
					}
					soff += SRC_BATCH_STRIDE;
					doff += DST_BATCH_STRIDE;
				}
			}
            if (!offset)
                sum2.x = sum;
            if (offset)
                sum2.y = sum;
		}
	}

    if (nr_thpf == 1) {
        flt_grad[OUT_IDX] = sum2.x;
        if (OUT_IDX != LAST_IDX)
            flt_grad[OUT_IDX + 1] = sum2.y;
    } else {
        extern __shared__ uint8_t shared_storage[];
        __half2* thread_sum = reinterpret_cast<__half2*>(shared_storage);
        thread_sum += THREADID_X * nr_thpf / 2;
        thread_sum[tid] = sum2;
#pragma unroll
        for (uint32_t i = nr_thpf / 2; i; i >>= 1) {
            bool cond = nr_thpf >= i * 2 && tid < i;
            if (i >= WARP_SIZE) {
                __syncthreads();
            } else {
                hipcub::WARP_SYNC(0xffffffff);
            }
            if (cond) {
                __half2 one = {1.0, 1.0};
                __half2 v0 = thread_sum[tid], v1 = fma2(v0, one, thread_sum[tid + i]);
                thread_sum[tid] = v1;
            }
        }

        if (!tid) {
            flt_grad[OUT_IDX] = thread_sum[0].x;
            if (OUT_IDX != LAST_IDX)
                flt_grad[OUT_IDX + 1] = thread_sum[0].y;
        }
    }
}
#endif

#define GET_KERN(func, type)                                    \
    FixFunction<type> f_struct;                                 \
    switch (_p) {                                               \
        case 1 << 10:                                           \
            f_struct.f = func<type, 1 << 10>;                   \
            break;                                              \
        case 1 << 9:                                            \
            f_struct.f = func<type, 1 << 9>;                    \
            break;                                              \
        case 1 << 8:                                            \
            f_struct.f = func<type, 1 << 8>;                    \
            break;                                              \
        case 1 << 7:                                            \
            f_struct.f = func<type, 1 << 7>;                    \
            break;                                              \
        case 1 << 6:                                            \
            f_struct.f = func<type, 1 << 6>;                    \
            break;                                              \
        case 1 << 5:                                            \
            f_struct.f = func<type, 1 << 5>;                    \
            break;                                              \
        case 1 << 4:                                            \
            f_struct.f = func<type, 1 << 4>;                    \
            break;                                              \
        case 1 << 3:                                            \
            f_struct.f = func<type, 1 << 3>;                    \
            break;                                              \
        case 1 << 2:                                            \
            f_struct.f = func<type, 1 << 2>;                    \
            break;                                              \
        case 1 << 1:                                            \
            f_struct.f = func<type, 1 << 1>;                    \
            break;                                              \
        case 1 << 0:                                            \
            f_struct.f = func<type, 1 << 0>;                    \
            break;                                              \
        default:                                                \
            megdnn_assert(false, "DO NOT IMP CASE FUNCTION!!"); \
    }                                                           \
    return f_struct;

template <typename T>
struct FixFunction {
    void (*f)(T*, const T*, const T*, Param);
};

template <typename T>
FixFunction<T> get_kern(const uint32_t& _p);

template <>
FixFunction<float> get_kern<float>(const uint32_t& _p) {
    GET_KERN(kern_bwd_filter_float, float);
}

#if CUDA_VERSION >= 9000
template <>
FixFunction<__half> get_kern<__half>(const uint32_t& _p) {
    GET_KERN(kern_bwd_filter_hf, __half);
}
#endif

template <>
FixFunction<dt_float16> get_kern<dt_float16>(const uint32_t& _p) {
    GET_KERN(kern_bwd_filter_float, dt_float16);
}

#undef GET_KERN
}  // anonymous namespace

namespace megdnn {
namespace cuda {
namespace convolution {
namespace chanwise {
template <typename T>
void run_bwd_filter(T *filter_grad, const T *src, const T *dst_grad,
		const Param &param, hipStream_t stream) {
	void (*kern)(T*, const T*, const T*, Param) = NULL;
	uint32_t                                           
		nr_thread = query_blocksize_for_kernel(get_kern<T>(1024).f),
		nr_thpf = std::min(nr_thread,                  
        	std::max<uint32_t>(                    
				1,                                 
				param.out_h * param.out_w * param.batch /
				(BATCH_UNROLL * 16)));
	// find nearest power-of-2 of nr_thpf
	do {
#define CK(_n) \
		if (nr_thpf >= _n) { \
			kern = get_kern<T>(_n).f; \
			nr_thpf = _n; \
			break; \
		}
		CK(1<<10);
		CK(1<<9);
		CK(1<<8);
		CK(1<<7);
		CK(1<<6);
		CK(1<<5);
		CK(1<<4);
		CK(1<<3);
		CK(1<<2);
		CK(1<<1);
		CK(1<<0);
#undef CK
	} while(0);

	megdnn_assert(kern);
	nr_thread = query_blocksize_for_kernel(kern);

	uint32_t nr_flt_per_blk = nr_thread / nr_thpf;
	while (nr_flt_per_blk * nr_thpf % WARP_SIZE)
		--nr_flt_per_blk;
	megdnn_assert(nr_flt_per_blk);

	int nr_block = DIVUP(
		param.flt_h * param.flt_w * param.src_chl * param.chl_mul,
		nr_flt_per_blk);
	nr_thread = nr_flt_per_blk * nr_thpf;
	uint32_t shared = nr_thread * 2 * sizeof(T);
	kern <<< nr_block, nr_thread, shared, stream >>> (
		filter_grad, src, dst_grad, param);
	after_kernel_launch();
}

template void run_bwd_filter(float*, const float*, const float*, const Param&,
                             hipStream_t);

#if CUDA_VERSION >= 9000
template void run_bwd_filter(__half*, const __half*, const __half*, const Param&,
                             hipStream_t);
#endif

template void run_bwd_filter(dt_float16*, const dt_float16*, const dt_float16*,
                             const Param&, hipStream_t);

} // namespace chanwise
} // namespace convolution
} // namespace cuda
} // namespace megdnn


// vim: syntax=cuda.doxygen

