#include "hip/hip_runtime.h"
/**
 * \file dnn/src/cuda/batch_conv_bias/helper.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 */
#include "src/cuda/batch_conv_bias/helper.cuh"
#include "src/cuda/query_blocksize.cuh"
#include "src/cuda/utils.cuh"

using namespace megdnn;
using namespace cuda;
using namespace batch_conv_bias;

namespace {
__global__ void kern_compute_offset(
        int* __restrict__ offset, const convolution::ConvParam param) {
    const int tid = threadIdx.x + blockDim.x * blockIdx.x;
    const int img_pixels = param.ho * param.wo;
    const int img_pixels_ru128 = DIVUP(img_pixels, 128) * 128;
    const int filter_pixels = param.fh * param.fw;
    if (tid >= img_pixels_ru128 * filter_pixels)
        return;
    const int filter_idx = tid / img_pixels;
    const int img_idx = tid - img_pixels * filter_idx;
    const int oh = img_idx / param.wo;
    const int ow = img_idx - oh * param.wo;
    const int kh = filter_idx / param.fw;
    const int kw = filter_idx - param.fw * kh;
    const int ih = param.sh * oh - param.ph + kh;
    const int iw = param.sw * ow - param.pw + kw;
    if (img_idx < img_pixels && ih >= 0 && ih < param.hi && iw >= 0 && iw < param.wi) {
        offset[tid] = ih * param.wi + iw;
    } else {
        offset[tid] = -1;
    }
}
}  // namespace

void megdnn::cuda::batch_conv_bias::compute_offset(
        int* offset, const convolution::ConvParam& param, hipStream_t stream) {
    uint32_t nr_threads = query_blocksize_for_kernel(
            reinterpret_cast<const void*>(kern_compute_offset));
    uint32_t img_pixels = param.ho * param.wo;
    uint32_t img_pixels_ru128 = DIVUP(img_pixels, 128) * 128;
    uint32_t filter_pixels = param.fh * param.fw;
    uint32_t vthreads = img_pixels_ru128 * filter_pixels;
    uint32_t nr_blocks = DIVUP(vthreads, nr_threads);
    kern_compute_offset<<<nr_blocks, nr_threads, 0, stream>>>(offset, param);
    after_kernel_launch();
}

// vim: ft=cpp syntax=cuda.doxygen foldmethod=marker foldmarker=f{{{,f}}}
