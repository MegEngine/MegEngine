#include "hip/hip_runtime.h"
/**
 * \file dnn/src/cuda/relayout_format/relayout_format_nchw_nhwc.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or
 * implied.
 */

#include "src/cuda/query_blocksize.cuh"
#include "src/cuda/relayout_format/relayout_format_kern.cuh"

using namespace megdnn;
using namespace cuda;
using namespace relayout_format;
using namespace internal;

namespace {
template <int pack_w>
struct rwtype_helper;

template <>
struct rwtype_helper<2> {
    using InnerDtype = char;
};

template <>
struct rwtype_helper<8> {
    using InnerDtype = unsigned;
};
}  // namespace

void relayout_format::relayout_format_cuda_nchw_nhwc(
        const TensorND& src, const TensorND& dst, const hipStream_t& stream,
        const float src_scale, const float dst_scale, const uint8_t src_zero_point,
        const uint8_t dst_zero_point) {
    auto&& stype = src.layout.dtype;
    auto&& dtype = dst.layout.dtype;
    auto& src_layout = src.layout;
    auto& dst_layout = dst.layout;
    int n = src.layout[0];
    int ic = src.layout[1];
    int h = src.layout[2];
    int w = src.layout[3];
    int w_pad = DIVUP(w, 2) * 2;
    int hw = h * w_pad;
    int n_stride_src = src_layout.stride[0];
    int ic_stride = src_layout.stride[1];
    int n_stride_dst = dst_layout.stride[0];
    int hw_stride = dst_layout.stride[2];
    static constexpr int chan_blk = 8;
    static constexpr int pack_oc = 8;
    int problem_size = n * DIVUP(ic, chan_blk) * hw;
    int oc = dst.layout[3];

    bool same_scale = src_scale == dst_scale;
    bool padding = w % 2 != 0;
#define DISPATCH_RAW(                                                               \
        _padding, _same_scale, _pack_w, _src_type, _dst_type, _src_c_type,          \
        _dst_c_type, _size_nbits)                                                   \
    if (padding == _padding && same_scale == _same_scale && hw % _pack_w == 0 &&    \
        stype.enumv().ev == DTypeEnum::Ev::_src_type &&                             \
        dtype.enumv().ev == DTypeEnum::Ev::_dst_type) {                             \
        using InnerDtype_ = typename rwtype_helper<_pack_w>::InnerDtype;            \
        using SrcIterator_ = TensorIteratorOverChannel<                             \
                InnerDtype_, 1, chan_blk, _pack_w, _size_nbits>;                    \
        using DstIterator_ = typename TensorIteratorPolicy<                         \
                _padding, _dst_c_type, pack_oc, chan_blk, _pack_w, _size_nbits,     \
                LayoutType::NHWC>::TensorIterator;                                  \
        using CudaPostProcess_ =                                                    \
                CudaPostProcess<dtype::_src_type, dtype::_dst_type, _same_scale>;   \
        using Transpose_ = Translayout<                                             \
                _pack_w, chan_blk, InnerDtype_, dtype::_src_type, dtype::_dst_type, \
                _same_scale>;                                                       \
        using RelayoutProblem_ = RelayoutProblem<                                   \
                SrcIterator_, DstIterator_, Transpose_, CudaPostProcess_>;          \
        n_stride_src = n_stride_src * _size_nbits / (8 * sizeof(InnerDtype_));      \
        ic_stride = ic_stride * _size_nbits / (8 * sizeof(InnerDtype_));            \
        n_stride_dst = n_stride_dst * _size_nbits / (8 * sizeof(_dst_c_type));      \
        hw_stride = hw_stride * _size_nbits / (8 * sizeof(_dst_c_type));            \
        typename RelayoutProblem_::Param param{                                     \
                SrcIterator_{(InnerDtype_*)src.raw_ptr, ic_stride, ic, w, w_pad},   \
                DstIterator_{(_dst_c_type*)dst.raw_ptr, hw_stride, oc, w, w_pad},   \
                CudaPostProcess_{                                                   \
                        src_scale, src_zero_point, dst_scale, dst_zero_point},      \
                n_stride_src,                                                       \
                n_stride_dst,                                                       \
                n,                                                                  \
                ic,                                                                 \
                hw,                                                                 \
                src_zero_point};                                                    \
        auto kernel = relayout_kern<RelayoutProblem_>;                              \
        int nr_threads = query_blocksize_for_kernel(kernel);                        \
        nr_threads = std::min(nr_threads, DIVUP(problem_size, _pack_w));            \
        const dim3 block_dim(DIVUP(problem_size, nr_threads* _pack_w));             \
        const dim3 thread_dim(nr_threads);                                          \
        return kernel<<<block_dim, thread_dim, 0, stream>>>(param);                 \
    }
#define DISPATCH_4BITS(_src_type, _dst_type)                            \
    DISPATCH_RAW(true, true, 8, _src_type, _dst_type, char, char, 4);   \
    DISPATCH_RAW(true, false, 8, _src_type, _dst_type, char, char, 4);  \
    DISPATCH_RAW(true, true, 2, _src_type, _dst_type, char, char, 4);   \
    DISPATCH_RAW(true, false, 2, _src_type, _dst_type, char, char, 4);  \
    DISPATCH_RAW(false, true, 8, _src_type, _dst_type, char, char, 4);  \
    DISPATCH_RAW(false, false, 8, _src_type, _dst_type, char, char, 4); \
    DISPATCH_RAW(false, true, 2, _src_type, _dst_type, char, char, 4);  \
    DISPATCH_RAW(false, false, 2, _src_type, _dst_type, char, char, 4);
    DISPATCH_4BITS(QuantizedS4, QuantizedS4);
    DISPATCH_4BITS(Quantized4Asymm, Quantized4Asymm);
#undef DISPATCH_4BITS
#undef DISPATCH_RAW
    megdnn_assert(
            false, "Unsupported data type(src:%s, dst:%s) or image size(%dx%d).",
            stype.name(), dtype.name(), h, w);
}

void relayout_format::relayout_format_cuda_nhwc_nchw(
        const TensorND& src, const TensorND& dst, const hipStream_t& stream,
        const float src_scale, const float dst_scale, const uint8_t src_zero_point,
        const uint8_t dst_zero_point) {
    auto&& stype = src.layout.dtype;
    auto&& dtype = dst.layout.dtype;
    auto& src_layout = src.layout;
    auto& dst_layout = dst.layout;

    int n = src.layout[0];
    int h = src.layout[1];
    int w = src.layout[2];
    int ic = src.layout[3];
    int w_pad = DIVUP(w, 2) * 2;
    int hw = h * w_pad;
    int n_stride_src = src_layout.stride[0];
    int hw_stride = src_layout.stride[2];
    int n_stride_dst = dst_layout.stride[0];
    int oc_stride = dst_layout.stride[1];
    static constexpr int chan_blk = 8;
    static constexpr int pack_oc = 8;
    int problem_size = n * DIVUP(ic, chan_blk) * hw;
    int oc = dst.layout[1];

    bool same_scale = src_scale == dst_scale;
    bool padding = w % 2 != 0;
#define DISPATCH_RAW(                                                               \
        _padding, _same_scale, _pack_w, _src_type, _dst_type, _src_c_type,          \
        _dst_c_type, _size_nbits)                                                   \
    if (padding == _padding && same_scale == _same_scale && hw % _pack_w == 0 &&    \
        stype.enumv().ev == DTypeEnum::Ev::_src_type &&                             \
        dtype.enumv().ev == DTypeEnum::Ev::_dst_type) {                             \
        using SrcIterator_ = typename TensorIteratorPolicy<                         \
                _padding, _src_c_type, pack_oc, chan_blk, _pack_w, _size_nbits,     \
                LayoutType::NHWC>::TensorIterator;                                  \
        using InnerDtype_ = typename rwtype_helper<_pack_w>::InnerDtype;            \
        using DstIterator_ = TensorIteratorOverChannel<                             \
                InnerDtype_, 1, chan_blk, _pack_w, _size_nbits>;                    \
        using CudaPostProcess_ =                                                    \
                CudaPostProcess<dtype::_src_type, dtype::_dst_type, _same_scale>;   \
        using Transpose_ = Translayout<                                             \
                chan_blk, _pack_w, _src_c_type, dtype::_src_type, dtype::_dst_type, \
                _same_scale>;                                                       \
        using RelayoutProblem_ = RelayoutProblem<                                   \
                SrcIterator_, DstIterator_, Transpose_, CudaPostProcess_>;          \
        n_stride_src = n_stride_src * _size_nbits / (8 * sizeof(_src_c_type));      \
        hw_stride = hw_stride * _size_nbits / (8 * sizeof(_src_c_type));            \
        n_stride_dst = n_stride_dst * _size_nbits / (8 * sizeof(InnerDtype_));      \
        oc_stride = oc_stride * _size_nbits / (8 * sizeof(InnerDtype_));            \
        typename RelayoutProblem_::Param param{                                     \
                SrcIterator_{(_src_c_type*)src.raw_ptr, hw_stride, ic, w, w_pad},   \
                DstIterator_{(InnerDtype_*)dst.raw_ptr, oc_stride, oc, w, w_pad},   \
                CudaPostProcess_{                                                   \
                        src_scale, src_zero_point, dst_scale, dst_zero_point},      \
                n_stride_src,                                                       \
                n_stride_dst,                                                       \
                n,                                                                  \
                ic,                                                                 \
                hw,                                                                 \
                src_zero_point};                                                    \
        auto kernel = relayout_kern<RelayoutProblem_>;                              \
        int nr_threads = query_blocksize_for_kernel(kernel);                        \
        nr_threads = std::min(nr_threads, DIVUP(problem_size, _pack_w));            \
        const dim3 block_dim(DIVUP(problem_size, nr_threads* _pack_w));             \
        const dim3 thread_dim(nr_threads);                                          \
        return kernel<<<block_dim, thread_dim, 0, stream>>>(param);                 \
    }
#define DISPATCH_4BITS(_src_type, _dst_type)                            \
    DISPATCH_RAW(true, true, 8, _src_type, _dst_type, char, char, 4);   \
    DISPATCH_RAW(true, false, 8, _src_type, _dst_type, char, char, 4);  \
    DISPATCH_RAW(true, true, 2, _src_type, _dst_type, char, char, 4);   \
    DISPATCH_RAW(true, false, 2, _src_type, _dst_type, char, char, 4);  \
    DISPATCH_RAW(false, true, 8, _src_type, _dst_type, char, char, 4);  \
    DISPATCH_RAW(false, false, 8, _src_type, _dst_type, char, char, 4); \
    DISPATCH_RAW(false, true, 2, _src_type, _dst_type, char, char, 4);  \
    DISPATCH_RAW(false, false, 2, _src_type, _dst_type, char, char, 4);
    DISPATCH_4BITS(QuantizedS4, QuantizedS4);
    DISPATCH_4BITS(Quantized4Asymm, Quantized4Asymm);
#undef DISPATCH_4BITS
#undef DISPATCH_RAW
    megdnn_assert(
            false, "Unsupported data type(src:%s, dst:%s) or image size(%dx%d).",
            stype.name(), dtype.name(), h, w);
}
