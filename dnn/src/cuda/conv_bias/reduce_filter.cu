/***************************************************************************************************
 * Copyright (c) 2017-2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 *modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice,
 *this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *notice, this list of conditions and the following disclaimer in the
 *documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its
 *contributors may be used to endorse or promote products derived from this
 *software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 *AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 *IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 *DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY DIRECT,
 *INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 *DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 *OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TOR (INCLUDING
 *NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 *EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/**
 * \file dnn/src/cuda/conv_bias/reduce_filter.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or
 * implied.
 */

#include "./reduce_filter.cuh"
#include "src/cuda/utils.cuh"

#include "src/cuda/integer_subbyte_utils.cuh"
#include "src/cuda/reduce_helper.cuh"

using namespace megdnn;
using namespace cuda;

namespace {

template <bool signedness>
struct ReduceWithScaleInt4Op {
    typedef int32_t wtype;
    const uint8_t* src;
    int32_t* dst;
    int32_t scale;
    static const wtype INIT = 0;

#if MEGDNN_CC_CUDA
    __host__ __device__ void write(uint32_t idx, wtype val) { dst[idx] = val * scale; }

    __host__ __device__ static wtype apply(wtype a, wtype b) { return a + b; }

    __device__ wtype read(uint32_t idx) {
        constexpr uint32_t subbytes_per_pixel = 8;
        const uint32_t* sptr = (const uint32_t*)(src + subbytes_per_pixel * idx / 2);
        uint32_t val = *sptr;
        int32_t ret = 0;
#pragma unroll
        for (int j = 0; j < 8; j++) {
            ret += integer_subbyte::unpack_integer_4bits<signedness>(val, (j << 2));
        }
        return ret;
    }
#endif
};

template <bool signedness>
struct ReduceUpdateBiasInt4Op {
    typedef int32_t wtype;
    const uint8_t* filter;
    const int32_t* src_bias;
    int32_t* dst_bias;
    int32_t zero_point;
    static const wtype INIT = 0;

#if MEGDNN_CC_CUDA
    __host__ __device__ void write(uint32_t idx, wtype val) {
        dst_bias[idx] = src_bias[idx] - val * zero_point;
    }

    __host__ __device__ static wtype apply(wtype a, wtype b) { return a + b; }

    __device__ wtype read(uint32_t idx) {
        constexpr uint32_t subbytes_per_pixel = 8;
        const uint32_t* fptr = (const uint32_t*)(filter + subbytes_per_pixel * idx / 2);
        uint32_t val = *fptr;
        int32_t ret = 0;
#pragma unroll
        for (int j = 0; j < 8; j++) {
            ret += integer_subbyte::unpack_integer_4bits<signedness>(val, (j << 2));
        }
        return ret;
    }
#endif
};

}  // namespace

template <bool signedness>
void megdnn::cuda::do_dispatch_reduce_with_scale_filter_4bit(
        const uint8_t* src, int32_t scale, uint32_t rows, uint32_t cols, int32_t* dst,
        hipStream_t stream) {
    // rows = OC
    // cols is measured in pixels, i.e. IC * FH * FW / 8, a pixel consists of 8
    // subbyte data,
    ReduceWithScaleInt4Op<signedness> op;
    op.src = src;
    op.scale = scale;
    op.dst = dst;
    static_cast<void>(op);
    static_cast<void>(stream);
    static_cast<void>(rows);
    static_cast<void>(cols);
    run_reduce<ReduceWithScaleInt4Op<signedness>, false>(
            dst + rows, rows, cols, 1, stream, op);
}

#define INST(signedness)                                                               \
    template void megdnn::cuda::do_dispatch_reduce_with_scale_filter_4bit<signedness>( \
            const uint8_t* src, int32_t scale, uint32_t rows, uint32_t cols,           \
            int32_t* dst, hipStream_t stream)
INST(false);
INST(true);
#undef INST

template <bool signedness>
void megdnn::cuda::do_dispatch_reduce_filter_and_update_bias_4bit(
        const uint8_t* filter, const int32_t* src_bias, uint32_t rows, uint32_t cols,
        int32_t* dst_bias, int32_t* workspace, int32_t zero_point,
        hipStream_t stream) {
    ReduceUpdateBiasInt4Op<signedness> op;
    op.filter = filter;
    op.src_bias = src_bias;
    op.dst_bias = dst_bias;
    op.zero_point = zero_point;
    run_reduce<ReduceUpdateBiasInt4Op<signedness>, false>(
            workspace, rows, cols, 1, stream, op);
}

#define INST(signedness)                                                              \
    template void                                                                     \
    megdnn::cuda::do_dispatch_reduce_filter_and_update_bias_4bit<signedness>(         \
            const uint8_t* filter, const int32_t* src_bias, uint32_t rows,            \
            uint32_t cols, int32_t* dst_bias, int32_t* workspace, int32_t zero_point, \
            hipStream_t stream)

INST(false);
INST(true);
#undef INST

size_t megdnn::cuda::do_dispatch_reduce_workspace_in_bytes(
        size_t A, size_t B, size_t C) {
    return get_reduce_workspace_in_bytes<ReduceWithScaleInt4Op<false>>(A, B, C);
}

// vim: ft=cpp syntax=cuda.doxygen
