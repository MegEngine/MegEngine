#include "hip/hip_runtime.h"
/**
 * \file dnn/src/cuda/remap/backward_data.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or
 * implied.
 */
#include <hip/hip_runtime.h>
#include "src/common/rounding_converter.cuh"
#include "src/cuda/cv/kernel_common.cuh"
#include "src/cuda/remap/common.h"
#include "src/cuda/utils.cuh"

using namespace megdnn;
using namespace cuda;
using namespace remap;
using namespace rounding;

namespace {

template <const uint32_t format>
__device__ inline int get_offset(
        int height, int width, int channel, int h, int w, int c);

template <>
__device__ inline int get_offset<param_enumv::Remap::Format::NCHW>(
        int height, int width, int channel, int h, int w, int c) {
    return channel * h * w + height * w + width;
}

template <typename ctype, const uint32_t format, ::BorderMode bmode>
struct GetSrcData {
    __device__ static inline int get_index(
            int height, int width, int channel, int h, int w, int c) {
        height = megcv::border_interpolate<bmode>(height, h);
        width = megcv::border_interpolate<bmode>(width, w);
        return get_offset<format>(height, width, channel, h, w, c);
    }
};

template <typename ctype, const uint32_t format>
struct GetSrcData<ctype, format, ::BorderMode::BORDER_CONSTANT> {
    __device__ static inline int get_index(
            int height, int width, int channel, int h, int w, int c) {
        return (height >= 0 && height < h && width >= 0 && width < w)
                     ? get_offset<format>(height, width, channel, h, w, c)
                     : -1;
    }
};

template <typename ctype, const uint32_t format, ::BorderMode bmode>
__global__ void kern_general(
        ctype* __restrict grad, const float* map_xy, const ctype* diff, int C, int IH,
        int IW, int OH, int OW) {
    int ow = blockIdx.x * blockDim.x + threadIdx.x;
    int oh = blockIdx.y * blockDim.y + threadIdx.y;
    grad += blockIdx.z * C * IH * IW;
    diff += blockIdx.z * C * OH * OW;
    map_xy += blockIdx.z * 2 * OH * OW;
    RoundingConverter<ctype> round_converter;

    if (ow < OW && oh < OH) {
        float index_col = map_xy[oh * OW * 2 + ow * 2 + 0];
        float index_row = map_xy[oh * OW * 2 + ow * 2 + 1];
        int col = static_cast<int>(floor(index_col));
        int row = static_cast<int>(floor(index_row));
        float v = index_col - col;  // alphah
        float u = index_row - row;  // alphaw
        const float one = 1.f;
        for (int c = 0; c < C; ++c) {
            float hidden =
                    static_cast<float>(diff[get_offset<format>(oh, ow, c, OH, OW, C)]);

            int a00 = GetSrcData<ctype, format, bmode>::get_index(
                    row + 0, col + 0, c, IH, IW, C);
            if (a00 != -1) {
                atomic_add(grad + a00, round_converter((one - u) * (one - v) * hidden));
            }

            int a01 = GetSrcData<ctype, format, bmode>::get_index(
                    row + 0, col + 1, c, IH, IW, C);
            if (a01 != -1) {
                atomic_add(grad + a01, round_converter((one - u) * v * hidden));
            }

            int a10 = GetSrcData<ctype, format, bmode>::get_index(
                    row + 1, col + 0, c, IH, IW, C);
            if (a10 != -1) {
                atomic_add(grad + a10, round_converter(u * (one - v) * hidden));
            }

            int a11 = GetSrcData<ctype, param_enumv::Remap::Format::NCHW, bmode>::
                    get_index(row + 1, col + 1, c, IH, IW, C);
            if (a11 != -1) {
                atomic_add(grad + a11, round_converter(u * v * hidden));
            }
        }
    }
}

template <typename ctype, const uint32_t format, ::BorderMode bmode>
void dispatch_backwarddata(
        ctype* grad, const float* map_xy, const ctype* diff, int N, int C, int IH,
        int IW, int OH, int OW, hipStream_t stream) {
    const int BX = 32, BY = 16;
    const int max_batch_size = 65535;
    while (N) {
        size_t curr_batch_size = N < max_batch_size ? N : max_batch_size;
        dim3 threads(BX, BY);
        dim3 blocks((OW + BX - 1) / BX, (OH + BY - 1) / BY, curr_batch_size);

        cuda_check(hipMemsetAsync(
                grad, 0, sizeof(ctype) * curr_batch_size * C * IH * IW, stream));
        kern_general<ctype, format, bmode>
                <<<blocks, threads, 0, stream>>>(grad, map_xy, diff, C, IH, IW, OH, OW);

        N -= curr_batch_size;
        grad += curr_batch_size * C * IH * IW;
        diff += curr_batch_size * C * OH * OW;
        map_xy += curr_batch_size * 2 * OH * OW;
    }
}

}  // anonymous namespace

namespace megdnn {
namespace cuda {
namespace remap {

template <typename ctype, const uint32_t format, ::BorderMode bmode>
void backwarddata_proxy(
        ctype* grad, const float* map_xy, const ctype* diff, int N, int C, int IH,
        int IW, int OH, int OW, hipStream_t stream) {
    dispatch_backwarddata<ctype, format, bmode>(
            grad, map_xy, diff, N, C, IH, IW, OH, OW, stream);
    after_kernel_launch();
}

#define INST(ctype, format, bmode)                                            \
    template void backwarddata_proxy<                                         \
            ctype, param_enumv::Remap::Format::format, ::BorderMode::bmode>(  \
            ctype*, const float*, const ctype*, int, int, int, int, int, int, \
            hipStream_t);

#define FOR_FORMAT_BMODE(ctype)           \
    INST(ctype, NCHW, BORDER_CONSTANT)    \
    INST(ctype, NCHW, BORDER_REPLICATE)   \
    INST(ctype, NCHW, BORDER_REFLECT)     \
    INST(ctype, NCHW, BORDER_REFLECT_101) \
    INST(ctype, NCHW, BORDER_WRAP)

FOR_FORMAT_BMODE(float)
DNN_INC_FLOAT16(FOR_FORMAT_BMODE(dt_bfloat16))

#undef FOR_FORMAT_BMODE
#undef INST

}  // namespace remap
}  // namespace cuda
}  // namespace megdnn

// vim: syntax=cpp.doxygen
