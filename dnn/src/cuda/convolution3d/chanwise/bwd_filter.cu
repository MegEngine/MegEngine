#include "hip/hip_runtime.h"
/**
 * \file dnn/src/cuda/convolution3d/chanwise/bwd_filter.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 */

#include "./kern.cuh"
#include "./kern_helper.cuh"

const uint32_t WARP_SIZE = 32, BATCH_UNROLL = 4;

using namespace megdnn;
using namespace cuda;
using namespace convolution3d;
using namespace chanwise;

namespace {

template <typename T, uint32_t nr_thpf>
__global__ void kern_bwd_filter(
        T* flt_grad, const T* src, const T* dst_grad, Param param) {
    const uint32_t N = param.batch, IC = param.src_chl, ID = param.src_d,
                   IH = param.src_h, IW = param.src_w, CHL_MUL = param.chl_mul,
                   FD = param.flt_d, FH = param.flt_h, FW = param.flt_w,
                   PD = param.pad_d, PH = param.pad_h, PW = param.pad_w,
                   SD = param.stride_d, SH = param.stride_h, SW = param.stride_w,
                   OD = param.out_d, OH = param.out_h, OW = param.out_w,
                   SRC_BATCH_STRIDE = IC * ID * IH * IW,
                   DST_BATCH_STRIDE = IC * CHL_MUL * OD * OH * OW,
                   BLKDIM_X = blockDim.x / nr_thpf, THREADID_X = threadIdx.x / nr_thpf,
                   OUT_IDX = blockIdx.x * BLKDIM_X + THREADID_X;

    uint32_t ic, chl_mul, fd, fh, fw;
    {
        uint32_t i = OUT_IDX;
        i = div_mod(i, FW, fw);
        i = div_mod(i, FH, fh);
        i = div_mod(i, FD, fd);
        i = div_mod(i, CHL_MUL, chl_mul);
        ic = i;
    }
    if (ic >= IC) {
        return;
    }
    src += ic * ID * IH * IW;
    dst_grad += (ic * CHL_MUL + chl_mul) * OD * OH * OW;

    const uint32_t od_lo = max(int32_t(PD - fd + SD - 1), 0) / SD,
                   od_hi = min((ID - 1 + PD - fd) / SD + 1, OD),
                   oh_lo = max(int32_t(PH - fh + SH - 1), 0) / SH,
                   oh_hi = min((IH - 1 + PH - fh) / SH + 1, OH),
                   ow_lo = max(int32_t(PW - fw + SW - 1), 0) / SW,
                   ow_hi = min((IW - 1 + PW - fw) / SW + 1, OW), oblk_d = od_hi - od_lo,
                   oblk_h = oh_hi - oh_lo, oblk_w = ow_hi - ow_lo,
                   oblk_tot = oblk_d * oblk_h * oblk_w *
                              ((N + BATCH_UNROLL - 1) / BATCH_UNROLL),
                   tid = threadIdx.x % nr_thpf;

    if (ID + PD < fd + 1 || od_lo >= od_hi || IH + PH < fh + 1 || oh_lo >= oh_hi ||
        IW + PW < fw + 1 || ow_lo >= ow_hi) {
        if (!tid)
            flt_grad[OUT_IDX] = 0;
        return;
    }

    T sum(0);
    for (uint32_t oblk_idx = tid; oblk_idx < oblk_tot; oblk_idx += nr_thpf) {
        uint32_t n, oh, ow, od;
        n = div_mod(div_mod(div_mod(oblk_idx, oblk_w, ow), oblk_h, oh), oblk_d, od) *
            BATCH_UNROLL;
        od += od_lo;
        oh += oh_lo;
        ow += ow_lo;
        uint32_t id = od * SD - PD + fd, ih = oh * SH - PH + fh, iw = ow * SW - PW + fw,
                 soff = id * IH * IW + ih * IW + iw + n * SRC_BATCH_STRIDE,
                 doff = od * OH * OW + oh * OW + ow + n * DST_BATCH_STRIDE;
#pragma unroll
        for (uint32_t i = 0; i < BATCH_UNROLL; ++i) {
            if (!i || n + i < N) {
                sum += src[soff] * dst_grad[doff];
            }
            soff += SRC_BATCH_STRIDE;
            doff += DST_BATCH_STRIDE;
        }
    }

    if (nr_thpf == 1) {
        flt_grad[OUT_IDX] = sum;
    } else {
        // reduce all sums in a block
        extern __shared__ uint8_t shared_storage[];
        volatile T* thread_sum = reinterpret_cast<T*>(shared_storage);
        thread_sum += THREADID_X * nr_thpf;
        thread_sum[tid] = sum;
#pragma unroll
        for (uint32_t i = nr_thpf / 2; i; i >>= 1) {
            bool cond = nr_thpf >= i * 2 && tid < i;
            if (i >= WARP_SIZE) {
                __syncthreads();
            }
            T v0 = thread_sum[tid], v1 = v0 + thread_sum[tid + i];
            thread_sum[tid] = cond ? v1 : v0;
        }

        if (!tid)
            flt_grad[OUT_IDX] = thread_sum[0];
    }
}

}  // anonymous namespace

template <typename T>
void convolution3d::chanwise::run_bwd_filter(
        T* filter_grad, const T* src, const T* dst_grad, const Param& param,
        hipStream_t stream) {
    void (*kern)(T*, const T*, const T*, Param) = NULL;
    uint32_t nr_thread = query_blocksize_for_kernel(kern_bwd_filter<T, 1024>),
             nr_thpf = std::min(
                     nr_thread, std::max<uint32_t>(
                                        1, param.out_d * param.out_h * param.out_w *
                                                   param.batch / (BATCH_UNROLL * 16)));

    // find nearest power-of-2 of nr_thpf
    do {
#define CK(_n)                         \
    if (nr_thpf >= _n) {               \
        kern = kern_bwd_filter<T, _n>; \
        nr_thpf = _n;                  \
        break;                         \
    }
        CK(1 << 10);
        CK(1 << 9);
        CK(1 << 8);
        CK(1 << 7);
        CK(1 << 6);
        CK(1 << 5);
        CK(1 << 4);
        CK(1 << 3);
        CK(1 << 2);
        CK(1 << 1);
        CK(1 << 0);
#undef CK
    } while (0);

    megdnn_assert(kern);
    nr_thread = query_blocksize_for_kernel(kern);

    uint32_t nr_flt_per_blk = nr_thread / nr_thpf;
    while (nr_flt_per_blk * nr_thpf % WARP_SIZE)
        --nr_flt_per_blk;
    megdnn_assert(nr_flt_per_blk);

    int nr_block = DIVUP(
            param.flt_d * param.flt_h * param.flt_w * param.src_chl * param.chl_mul,
            nr_flt_per_blk);
    nr_thread = nr_flt_per_blk * nr_thpf;
    uint32_t shared = nr_thread * 2 * sizeof(T);
    kern<<<nr_block, nr_thread, shared, stream>>>(filter_grad, src, dst_grad, param);
    after_kernel_launch();
}

namespace megdnn {
namespace cuda {
namespace convolution3d {
namespace chanwise {

#define DO_INST(_ct)              \
    template void run_bwd_filter( \
            _ct*, const _ct*, const _ct*, const Param&, hipStream_t);
#define INST(_dt) DO_INST(DTypeTrait<_dt>::ctype)

MEGDNN_FOREACH_COMPUTING_DTYPE_FLOAT(INST)

#undef INST
#undef DO_INST

}  // namespace chanwise
}  // namespace convolution3d
}  // namespace cuda
}  // namespace megdnn

// vim: syntax=cuda.doxygen
