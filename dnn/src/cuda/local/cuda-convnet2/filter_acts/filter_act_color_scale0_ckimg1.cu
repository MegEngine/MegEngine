/**
 * \file dnn/src/cuda/local/cuda-convnet2/filter_acts/filter_act_color_scale0_ckimg1.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 */
/**
 * Copyright 2014 Google Inc. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *    http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 * --------------------------------------------------------------------------
 * * This file has been modified by Megvii ("Megvii Modifications").
 * * All Megvii Modifications are Copyright (C) 2014-2021 Megvii Inc. All rights
 * reserved.
 * --------------------------------------------------------------------------
 */
#include "filter_act_color.cuh"
namespace megdnn {
namespace cuda {

FILTER_COLOR(false, true)
}  // namespace cuda
}  // namespace megdnn
