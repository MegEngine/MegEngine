#include "hip/hip_runtime.h"
/**
 * \file dnn/src/cuda/roi_align/roi_align.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 */
#include "src/cuda/roi_align/roi_align.cuh"

#include <cfloat>
#include "megdnn/dtype.h"
#include "src/common/roi_align_helper.h"
#include "src/cuda/query_blocksize.cuh"
#include "src/cuda/utils.cuh"

using namespace megdnn;
using namespace roi_align;

namespace megdnn {
namespace cuda {
namespace roi_align {

#define CUDA_KERNEL_LOOP(vtid, vthreads)                                    \
    for (int vtid = blockIdx.x * blockDim.x + threadIdx.x; vtid < vthreads; \
         vtid += blockDim.x * gridDim.x)

template <typename T, typename Pooler>
__global__ void forward_kernel(
        const int nthreads, const T* bottom_data, const float spatial_scale,
        const float offset, const int channels, const int height, const int width,
        const int pooled_height, const int pooled_width, const int sample_height,
        const int sample_width, const T* bottom_rois, T* top_data, int* argmax_data) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        // (n, c, ph, pw) is an element in the pooled output
        int pw = index % pooled_width;
        int ph = (index / pooled_width) % pooled_height;
        int c = (index / pooled_width / pooled_height) % channels;
        int n = index / pooled_width / pooled_height / channels;

        bottom_rois += n * 5;
        int roi_batch_ind = bottom_rois[0];
        float roi_start_w = bottom_rois[1] * spatial_scale - offset;
        float roi_start_h = bottom_rois[2] * spatial_scale - offset;
        float roi_end_w = bottom_rois[3] * spatial_scale - offset;
        float roi_end_h = bottom_rois[4] * spatial_scale - offset;

        // Force malformed ROIs to be 1x1
        float roi_width = max(roi_end_w - roi_start_w, ((float)(0.0)));
        float roi_height = max(roi_end_h - roi_start_h, ((float)(0.0)));
        float bin_size_h =
                static_cast<float>(roi_height) / static_cast<float>(pooled_height);
        float bin_size_w =
                static_cast<float>(roi_width) / static_cast<float>(pooled_width);

        // regularly sample from a sample_height * sample_width grid
        bottom_data += (roi_batch_ind * channels + c) * height * width;
        float sample_h_rate = 1.0f / float(sample_height);
        float sample_w_rate = 1.0f / float(sample_width);
        float hcenter;
        float wcenter;

        Pooler pooler;
        for (int h_iter = 0; h_iter < sample_height; ++h_iter) {
            for (int w_iter = 0; w_iter < sample_width; ++w_iter) {
                hcenter = roi_start_h +
                          bin_size_h * (ph + sample_h_rate * (h_iter + 0.5f));
                wcenter = roi_start_w +
                          bin_size_w * (pw + sample_w_rate * (w_iter + 0.5f));
                T val = bilinear_interp(bottom_data, hcenter, wcenter, height, width);
                int idx = h_iter * sample_width + w_iter;
                pooler.feed(val, idx);
            }
        }
        pooler.writeback_val(top_data[index]);
        pooler.writeback_idx(argmax_data[index]);
    }
}

template <typename T, typename BwdPooler>
__global__ void backward_kernel(
        const int nthreads, const T* top_diff, const T* bottom_rois,
        const int* argmax_data, const float spatial_scale, const float offset,
        const int channels, const int height, const int width, const int pooled_height,
        const int pooled_width, const int sample_height, const int sample_width,
        T* bottom_diff) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        // (n, c, ph, pw) is an element in the pooled output
        int pw = index % pooled_width;
        int ph = (index / pooled_width) % pooled_height;
        int c = (index / pooled_width / pooled_height) % channels;
        int n = index / pooled_width / pooled_height / channels;

        bottom_rois += n * 5;
        int roi_batch_ind = bottom_rois[0];
        float roi_start_w = bottom_rois[1] * spatial_scale - offset;
        float roi_start_h = bottom_rois[2] * spatial_scale - offset;
        float roi_end_w = bottom_rois[3] * spatial_scale - offset;
        float roi_end_h = bottom_rois[4] * spatial_scale - offset;

        // Force malformed ROIs to be 1x1
        float roi_width = max(roi_end_w - roi_start_w, ((float)(0.0)));
        float roi_height = max(roi_end_h - roi_start_h, ((float)(0.0)));
        float bin_size_h =
                static_cast<float>(roi_height) / static_cast<float>(pooled_height);
        float bin_size_w =
                static_cast<float>(roi_width) / static_cast<float>(pooled_width);

        // regularly sample from a sample_height * sample_width grid
        bottom_diff += (roi_batch_ind * channels + c) * height * width;
        BwdPooler pooler{ph,    pw,          sample_height, sample_width, height,
                         width, roi_start_h, roi_start_w,   bin_size_h,   bin_size_w};
        pooler.update(index, top_diff, argmax_data, bottom_diff);
    }
}

template <typename T, typename Pooler>
void forward_proxy(
        const int nthreads, const T* bottom_data, const float spatial_scale,
        const float offset, const int channels, const int height, const int width,
        const int pooled_height, const int pooled_width, const int sample_height,
        const int sample_width, const T* bottom_rois, T* top_data, int* argmax_data,
        hipStream_t stream) {
    int threads_block = query_blocksize_for_kernel(forward_kernel<T, Pooler>);
    forward_kernel<T, Pooler>
            <<<DIVUP(nthreads, threads_block), threads_block, 0, stream>>>(
                    nthreads, bottom_data, spatial_scale, offset, channels, height,
                    width, pooled_height, pooled_width, sample_height, sample_width,
                    bottom_rois, top_data, argmax_data);
    after_kernel_launch();
}

template <typename T, typename BwdPooler>
void backward_proxy(
        const int nthreads, const T* top_diff, const int* argmax_data,
        const float spatial_scale, const float offset, const int channels,
        const int height, const int width, const int pooled_height,
        const int pooled_width, const int sample_height, const int sample_width,
        const T* bottom_rois, T* bottom_diff, hipStream_t stream) {
    int threads_block = query_blocksize_for_kernel(backward_kernel<T, BwdPooler>);
    backward_kernel<T, BwdPooler>
            <<<DIVUP(nthreads, threads_block), threads_block, 0, stream>>>(
                    nthreads, top_diff, bottom_rois, argmax_data, spatial_scale, offset,
                    channels, height, width, pooled_height, pooled_width, sample_height,
                    sample_width, bottom_diff);
    after_kernel_launch();
}

#define INST(T)                                                                        \
    template void forward_proxy<T, ::megdnn::roi_align::MaxPooler<T>>(                 \
            const int, const T*, const float, const float, const int, const int,       \
            const int, const int, const int, const int, const int, const T*, T*, int*, \
            hipStream_t);                                                             \
    template void forward_proxy<T, ::megdnn::roi_align::AveragePooler<T>>(             \
            const int, const T*, const float, const float, const int, const int,       \
            const int, const int, const int, const int, const int, const T*, T*, int*, \
            hipStream_t);                                                             \
    template void backward_proxy<T, ::megdnn::roi_align::BwdMaxPooler<T>>(             \
            const int, const T*, const int*, const float, const float, const int,      \
            const int, const int, const int, const int, const int, const int,          \
            const T*, T*, hipStream_t);                                               \
    template void backward_proxy<T, ::megdnn::roi_align::BwdAveragePooler<T>>(         \
            const int, const T*, const int*, const float, const float, const int,      \
            const int, const int, const int, const int, const int, const int,          \
            const T*, T*, hipStream_t);
INST(dt_float32)
INST(dt_float16)
INST(dt_bfloat16)
#undef INST

}  // namespace roi_align
}  // namespace cuda
}  // namespace megdnn

// vim: syntax=cpp.doxygen
