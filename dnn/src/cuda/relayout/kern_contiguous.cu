/**
 * \file dnn/src/cuda/relayout/kern_contiguous.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 */

#include "src/cuda/elemwise_helper.cuh"
#include "src/cuda/query_blocksize.cuh"
#include "src/cuda/relayout/kern_contiguous.cuh"
#include "src/cuda/utils.cuh"

namespace megdnn {
namespace cuda {

// dst is contiguous
void copy_last_contiguous(
        const TensorND& dst, const TensorND& src, size_t contiguous_size,
        hipStream_t stream) {
    ElemwiseOpParamN<2> param;
    param[0] = dst;
    param[1] = src;

#define RUN(_dt)                                                                  \
    do {                                                                          \
        typedef DTypeTrait<dtype::_dt>::ctype ctype;                              \
        param[0].layout.dtype = param[1].layout.dtype = dtype::_dt();             \
        param.init_from_given_tensor();                                           \
        param.assert_initialized();                                               \
        contiguous_intl::UserOpInvoker<ctype, 2>(param, stream, contiguous_size); \
        return;                                                                   \
    } while (0)

    switch (dst.layout.dtype.size()) {
        case 1:
            RUN(Byte);
        case 2:
            RUN(Float16);
        case 4:
            RUN(Int32);
    }
    megdnn_assert(0, "bad dtype size");
}

}  // namespace cuda
}  // namespace megdnn

// vim: syntax=cpp.doxygen
