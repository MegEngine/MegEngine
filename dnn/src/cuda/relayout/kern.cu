/**
 * \file dnn/src/cuda/relayout/kern.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 */

#include "megdnn/basic_types.h"
#include "src/cuda/elemwise_helper.cuh"
#include "src/cuda/relayout/kern.cuh"
#include "src/cuda/utils.cuh"

namespace megdnn {
namespace cuda {

void copy_noncontig_general(
        const TensorND& dst, const TensorND& src, hipStream_t stream) {
    ElemwiseOpParamN<2> param;
    param[0] = dst;
    param[1] = src;

#define RUN(_dt, body)                                                  \
    do {                                                                \
        typedef DTypeTrait<dtype::_dt>::ctype ctype;                    \
        body;                                                           \
        param.init_from_given_tensor();                                 \
        param.assert_initialized();                                     \
        noncontig_general_intl::UserOpInvoker<ctype, 2>(param, stream); \
        return;                                                         \
    } while (0)

    if (!dst.layout.dtype.is_low_bit()) {
        switch (dst.layout.dtype.size()) {
            case 1:
                RUN(Byte,
                    param[0].layout.dtype = param[1].layout.dtype = dtype::Byte());
            case 2:
                RUN(Float16,
                    param[0].layout.dtype = param[1].layout.dtype = dtype::Float16());
            case 4:
                RUN(Int32,
                    param[0].layout.dtype = param[1].layout.dtype = dtype::Int32());
        }
    } else {
        RUN(Quantized4Asymm, );
    }
    megdnn_assert(0, "bad dtype size");
}

}  // namespace cuda
}  // namespace megdnn

// vim: ft=cpp syntax=cpp.doxygen
