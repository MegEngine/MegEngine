#include "hip/hip_runtime.h"
/**
 * \file dnn/src/cuda/matrix_inverse/helper.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 */

#include "./helper.cuh"
#include "src/cuda/error_info.cuh"

using namespace megdnn;
using namespace cuda;
using namespace matrix_inverse;

namespace {

__global__ void kern_check_error(
        const int* src_info, uint32_t n, megcore::AsyncErrorInfo* dst_info,
        void* tracker) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n && src_info[i]) {
        set_async_error_info(
                dst_info, tracker,
                "The U is exactly singular and the inversion "
                "failed on %d-th input matrix (U(%d, %d) = 0)",
                i, src_info[i], src_info[i]);
    }
}

}  // anonymous namespace

void matrix_inverse::check_error(
        const int* src_info, uint32_t n, megcore::AsyncErrorInfo* dst_info,
        void* tracker, hipStream_t stream) {
    if (!dst_info) {
        return;
    }
    uint32_t threads = NR_THREADS;
    uint32_t blocks = DIVUP(n, threads);
    kern_check_error<<<blocks, threads, 0, stream>>>(src_info, n, dst_info, tracker);
    after_kernel_launch();
}

// vim: syntax=cpp.doxygen
